#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/ModelTrainerExperimentalGpu.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/io/Parameters.h>
#include <SmartPeak/simulator/ChromatogramSimulator.h>

#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

/**
Application designed to train a network to accurately integrate and identify peaks

Features:
- de-noises the chromatogram for more accurate peak area calculation
- determines the best left, right, and inner points for each peak as probabilities

Input:
- vector of time/mz and intensity pairs

Data pre-processing:
- each time/mz and intensity pair is binned into equally spaced time steps
- intensities are normalized to the range 0 to 1

Output:
- vector of intensity bins
- vector of logits of peak probabilities (peak threshold > 0.75)

Post-processing:
- integration of peaks based on binned intensity, average distance between time-steps, and logit peak probability pairs

*/

// Extended 
template<typename TensorT>
class ModelTrainerExt : public ModelTrainerExperimentalGpu<TensorT>
{
public:
  /*
  @brief Denoising Auto Encoder that takes a segment of a raw chromatogram
    and returns a smoothed and denoised version of the same chromatogram
  */
  void makeDenoisingAE(Model<TensorT>& model, int n_inputs = 512, int n_encodings = 32,
    int n_hidden_0 = 512, int n_hidden_1 = 256, int n_hidden_2 = 64, bool specify_layers = true) {
    model.setId(0);
    model.setName("DenoisingAE");
    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Define the activation
    auto activation = std::make_shared<LeakyReLUOp<TensorT>>(LeakyReLUOp<TensorT>());
    auto activation_grad = std::make_shared<LeakyReLUGradOp<TensorT>>(LeakyReLUGradOp<TensorT>());

    // Define the node integration
    auto integration_op = std::make_shared<SumOp<TensorT>>(SumOp<TensorT>());
    auto integration_error_op = std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>());
    auto integration_weight_grad_op = std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>());

    // Define the solver
    auto solver_op = std::make_shared<AdamOp<TensorT>>(AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8, 10));

    // Add the Encoder FC layers
    if (n_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_0", "EN_Intensity_0", node_names, n_hidden_0,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_0) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_1", "EN_Intensity_1", node_names, n_hidden_1,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_1) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_2", "EN_Intensity_2", node_names, n_hidden_2,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_2) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }

    // Add the encoding layers for Intensity
    std::vector<std::string> node_names_encoding = model_builder.addFullyConnected(model, "Encoding_Intensity", "Encoding_Intensity", node_names, n_encodings,
      activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      solver_op, 0.0f, 0.0f, false, specify_layers);

    // Add the Decoder FC layers
    node_names = node_names_encoding;
    if (n_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE_Intensity_2", "DE_Intensity_2", node_names, n_hidden_2,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_2) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE_Intensity_1", "DE_Intensity_1", node_names, n_hidden_1,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_1) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE_Intensity_0", "DE_Intensity_0", node_names, n_hidden_0,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_0) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }

    // Add the output nodes
    node_names = model_builder.addFullyConnected(model, "DE_Intensity_Out", "DE_Intensity_Out", node_names, n_inputs,
      //std::make_shared<SigmoidOp<TensorT>>(SigmoidOp<TensorT>()),
      //std::make_shared<SigmoidGradOp<TensorT>>(SigmoidGradOp<TensorT>()),
      std::make_shared<LeakyReLUOp<TensorT>>(LeakyReLUOp<TensorT>()),
      std::make_shared<LeakyReLUGradOp<TensorT>>(LeakyReLUGradOp<TensorT>()),
      integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>(node_names.size(), 1)),
      solver_op, 0.0f, 0.0f, false, specify_layers);
    node_names = model_builder.addSinglyConnected(model, "Output", "Output", node_names, n_inputs,
      std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()),
      std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1)),
      std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()), 0.0f, 0.0f, false, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names) {
      model.nodes_.at(node_name)->setType(NodeType::output);
    }

    model.setInputAndOutputNodes();

    //if (!model.checkCompleteInputToOutput())
    //  std::cout << "Model input and output are not fully connected!" << std::endl;
  }
  /*
  @brief An encoder that approximates the h, mu, tau, and sigma parameters of the EMG distribution
  */
  void makeEncoderEMG(Model<TensorT>& model, int n_inputs = 512, int n_outputs = 32,
    int n_hidden_0 = 512, int n_hidden_1 = 256, int n_hidden_2 = 64, bool specify_layers = true) {
    model.setId(0);
    model.setName("EncoderEMG");
    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Define the activation
    auto activation = std::make_shared<LeakyReLUOp<TensorT>>(LeakyReLUOp<TensorT>());
    auto activation_grad = std::make_shared<LeakyReLUGradOp<TensorT>>(LeakyReLUGradOp<TensorT>());

    // Define the node integration
    auto integration_op = std::make_shared<SumOp<TensorT>>(SumOp<TensorT>());
    auto integration_error_op = std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>());
    auto integration_weight_grad_op = std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>());

    // Define the solver
    auto solver_op = std::make_shared<AdamOp<TensorT>>(AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8, 10));

    // Add the Encoder FC layers
    if (n_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_0", "EN_Intensity_0", node_names, n_hidden_0,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_0) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_1", "EN_Intensity_1", node_names, n_hidden_1,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_1) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }
    if (n_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN_Intensity_2", "EN_Intensity_2", node_names, n_hidden_2,
        activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
        std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names.size() + n_hidden_2) / 2, 1)),
        solver_op, 0.0f, 0.0f, false, specify_layers);
    }

    // Add the output nodes
    node_names = model_builder.addFullyConnected(model, "DE_Intensity_Out", "DE_Intensity_Out", node_names, n_outputs,
      //std::make_shared<SigmoidOp<TensorT>>(SigmoidOp<TensorT>()),
      //std::make_shared<SigmoidGradOp<TensorT>>(SigmoidGradOp<TensorT>()),
      std::make_shared<LeakyReLUOp<TensorT>>(LeakyReLUOp<TensorT>()),
      std::make_shared<LeakyReLUGradOp<TensorT>>(LeakyReLUGradOp<TensorT>()),
      integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>(node_names.size(), 1)),
      solver_op, 0.0f, 0.0f, false, specify_layers);
    node_names = model_builder.addSinglyConnected(model, "Output", "Output", node_names, n_outputs,
      std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()),
      std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1)),
      std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()), 0.0f, 0.0f, false, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names) {
      model.nodes_.at(node_name)->setType(NodeType::output);
    }

    model.setInputAndOutputNodes();

    //if (!model.checkCompleteInputToOutput())
    //  std::cout << "Model input and output are not fully connected!" << std::endl;
  }
};

template<typename TensorT>
class DataSimulatorExt : public ChromatogramSimulator<TensorT>
{
public:
  void simulateChromData_(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = loss_output_data.dimension(2);
    const int n_metric_nodes = metric_output_data.dimension(2);
    input_data.setZero();
    loss_output_data.setZero();
    metric_output_data.setZero();

    if (this->output_data_type_ == "EMG") {
      assert(n_output_nodes == 4);
      assert(n_metric_nodes == 4);
    }
    else {
      assert(n_output_nodes == n_input_nodes);
      assert(n_metric_nodes == n_input_nodes);
    }

    // Reformat the Chromatogram for training
    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        std::vector<TensorT> chrom_time, chrom_intensity, chrom_time_test, chrom_intensity_test;
        std::vector<std::pair<TensorT, TensorT>> best_lr;
        std::vector<TensorT> peak_apices;
        std::vector<EMGModel<TensorT>> emgs;

        // make the chrom and noisy chrom
        this->simulateChromatogram(chrom_time_test, chrom_intensity_test, chrom_time, chrom_intensity, best_lr, peak_apices, emgs,
          step_size_mu_, step_size_sigma_, chrom_window_size_,
          noise_mu_, noise_sigma_, baseline_height_,
          n_peaks_, emg_h_, emg_tau_, emg_mu_offset_, emg_sigma_);

        for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
          input_data(batch_iter, memory_iter, nodes_iter) = chrom_intensity.at(nodes_iter);  //intensity
          if (this->output_data_type_ == "Points") {
            loss_output_data(batch_iter, memory_iter, nodes_iter) = chrom_intensity_test.at(nodes_iter);  //intensity
            metric_output_data(batch_iter, memory_iter, nodes_iter) = chrom_intensity_test.at(nodes_iter);  //intensity
          }
          else if (this->output_data_type_ == "IsApex") {
            TensorT isPeakApex = 0.0;
            for (const TensorT& peak_apex : peak_apices) {
              if (abs(chrom_time_test.at(nodes_iter) - peak_apex) < 1e-6) {
                isPeakApex = 1.0;
              }
            }
            loss_output_data(batch_iter, memory_iter, nodes_iter) = isPeakApex;  //IsPeakApex
            metric_output_data(batch_iter, memory_iter, nodes_iter) = isPeakApex;  //IsPeakApex
          }
          else if (this->output_data_type_ == "IsPeak") {
            TensorT isPeak = 0.0;
            for (const std::pair<TensorT, TensorT>& lr : best_lr) {
              if (chrom_time_test.at(nodes_iter) >= lr.first && chrom_time_test.at(nodes_iter) <= lr.second) {
                isPeak = 1.0;
              }
            }
            loss_output_data(batch_iter, memory_iter, nodes_iter) = isPeak;  //IsPeak
            metric_output_data(batch_iter, memory_iter, nodes_iter) = isPeak;  //IsPeak
          }
        }
        if (this->output_data_type_ == "EMG") {
          for (int i = 0; i < emgs.size(); ++i) {
            loss_output_data(batch_iter, memory_iter, i * 4) = emgs.at(i).getH();
            metric_output_data(batch_iter, memory_iter, i * 4) = emgs.at(i).getH();
            loss_output_data(batch_iter, memory_iter, i * 4 + 1) = emgs.at(i).getTau();
            metric_output_data(batch_iter, memory_iter, i * 4 + 1) = emgs.at(i).getTau();
            loss_output_data(batch_iter, memory_iter, i * 4 + 2) = emgs.at(i).getMu() / chrom_window_size_.first;
            metric_output_data(batch_iter, memory_iter, i * 4 + 2) = emgs.at(i).getMu() / chrom_window_size_.first;
            loss_output_data(batch_iter, memory_iter, i * 4 + 3) = emgs.at(i).getSigma() / chrom_window_size_.first;
            metric_output_data(batch_iter, memory_iter, i * 4 + 3) = emgs.at(i).getSigma() / chrom_window_size_.first;
          }
        }
      }
    }
    time_steps.setConstant(1.0f);
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override {
    simulateChromData_(input_data, loss_output_data, metric_output_data, time_steps);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override
  {
    simulateChromData_(input_data, loss_output_data, metric_output_data, time_steps);
  }

  /// public members that are passed to simulate methods
  std::pair<TensorT, TensorT> step_size_mu_ = std::make_pair(1, 1);
  std::pair<TensorT, TensorT> step_size_sigma_ = std::make_pair(0, 0);
  std::pair<TensorT, TensorT> chrom_window_size_ = std::make_pair(500, 500);
  std::pair<TensorT, TensorT> noise_mu_ = std::make_pair(0, 0);
  std::pair<TensorT, TensorT> noise_sigma_ = std::make_pair(0, 0.05);
  std::pair<TensorT, TensorT> baseline_height_ = std::make_pair(0, 0);
  std::pair<TensorT, TensorT> n_peaks_ = std::make_pair(10, 20);
  std::pair<TensorT, TensorT> emg_h_ = std::make_pair(0.1, 1.0);
  std::pair<TensorT, TensorT> emg_tau_ = std::make_pair(0, 1);
  std::pair<TensorT, TensorT> emg_mu_offset_ = std::make_pair(-10, 10);
  std::pair<TensorT, TensorT> emg_sigma_ = std::make_pair(0.1, 0.3);
  std::string output_data_type_ = "Points"; // "IsApex", "isPeak", "EMG"
  int encoding_size_ = 64;
};

template<class ...ParameterTypes>
void main_(const ParameterTypes& ...args) {
  auto parameters = std::make_tuple(args...);

  // define the model logger
  ModelLogger<float> model_logger(true, true, true, false, false, false, false);

  // define the data simulator
  std::size_t input_size;
  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos) {
    input_size = 64;
  }
  else {
    input_size = 512;
  }
  const std::size_t encoding_size = input_size / 8;
  DataSimulatorExt<float> data_simulator;
  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Points") != std::string::npos)
    data_simulator.output_data_type_ = "Points";
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("IsPeak") != std::string::npos)
    data_simulator.output_data_type_ = "IsPeak";
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("IsApex") != std::string::npos)
    data_simulator.output_data_type_ = "IsApex";
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos)
    data_simulator.output_data_type_ = "EMG";
  data_simulator.encoding_size_ = encoding_size;

  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos) {
    if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Hard") != std::string::npos) {
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0.5);
      data_simulator.baseline_height_ = std::make_pair(0, 0.2);
      data_simulator.n_peaks_ = std::make_pair(1, 1);
      data_simulator.emg_h_ = std::make_pair(0.1, 1);
      data_simulator.emg_tau_ = std::make_pair(0, 0.5);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 50);
    }
    else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Medium") != std::string::npos) {
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0.2);
      data_simulator.baseline_height_ = std::make_pair(0, 0);
      data_simulator.n_peaks_ = std::make_pair(1, 1);
      data_simulator.emg_h_ = std::make_pair(0.1, 1);
      data_simulator.emg_tau_ = std::make_pair(0, 0);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 30);
    }
    else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Easy") != std::string::npos) {
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0.2);
      data_simulator.baseline_height_ = std::make_pair(0, 0);
      data_simulator.n_peaks_ = std::make_pair(1, 1);
      data_simulator.emg_h_ = std::make_pair(1, 1);
      data_simulator.emg_tau_ = std::make_pair(0, 0);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 10);
    }
  }
  else {
    if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Hard") != std::string::npos) {
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0.2);
      data_simulator.baseline_height_ = std::make_pair(0, 0);
      data_simulator.n_peaks_ = std::make_pair(0, 10);
      data_simulator.emg_h_ = std::make_pair(0.1, 1);
      data_simulator.emg_tau_ = std::make_pair(0, 0.5);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 30);
    }
    else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Medium") != std::string::npos) {
      // Some issues with the peak start/stop not touching the baseline
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0.2);
      data_simulator.baseline_height_ = std::make_pair(0, 0);
      data_simulator.n_peaks_ = std::make_pair(1, 5);
      data_simulator.emg_h_ = std::make_pair(0.1, 1.0);
      data_simulator.emg_tau_ = std::make_pair(0, 0);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 30);
    }
    else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Easy") != std::string::npos) {
      data_simulator.step_size_mu_ = std::make_pair(1, 1);
      data_simulator.step_size_sigma_ = std::make_pair(0, 0);
      data_simulator.chrom_window_size_ = std::make_pair(input_size, input_size);
      data_simulator.noise_mu_ = std::make_pair(0, 0);
      data_simulator.noise_sigma_ = std::make_pair(0, 0);
      data_simulator.baseline_height_ = std::make_pair(0, 0);
      data_simulator.n_peaks_ = std::make_pair(1, 2);
      data_simulator.emg_h_ = std::make_pair(1, 1);
      data_simulator.emg_tau_ = std::make_pair(0, 0);
      data_simulator.emg_mu_offset_ = std::make_pair(0, 0);
      data_simulator.emg_sigma_ = std::make_pair(10, 10);
    }
  }

  // Make the input nodes
  std::vector<std::string> input_nodes;
  for (int i = 0; i < input_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the output nodes
  std::vector<std::string> output_nodes;
  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos) {
    for (int i = 0; i < 4; ++i) {
      char name_char[512];
      sprintf(name_char, "Output_%012d", i);
      std::string name(name_char);
      output_nodes.push_back(name);
    }
  }
  else {
    for (int i = 0; i < input_size; ++i) {
      char name_char[512];
      sprintf(name_char, "Output_%012d", i);
      std::string name(name_char);
      output_nodes.push_back(name);
    }
  }

  // define the model interpreters
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  setModelInterpreterParameters(model_interpreters, args...);

  // define the model trainer
  ModelTrainerExt<float> model_trainer;
  setModelTrainerParameters(model_trainer, args...);

  std::vector<LossFunctionHelper<float>> loss_function_helpers;
  LossFunctionHelper<float> loss_function_helper1;
  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos) {
    loss_function_helper1.output_nodes_ = output_nodes;
    loss_function_helper1.loss_functions_ = { std::make_shared<MSELossOp<float>>(MSELossOp<float>(1e-6, std::get<EvoNetParameters::ModelTrainer::LossFncWeight0>(parameters).get() / float(input_size))) };
    loss_function_helper1.loss_function_grads_ = { std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>(1e-6, std::get<EvoNetParameters::ModelTrainer::LossFncWeight0>(parameters).get() / float(input_size))) };
    loss_function_helpers.push_back(loss_function_helper1);
  }
  else {
    loss_function_helper1.output_nodes_ = output_nodes;
    loss_function_helper1.loss_functions_ = { std::make_shared<BCEWithLogitsLossOp<float>>(BCEWithLogitsLossOp<float>(1e-6, std::get<EvoNetParameters::ModelTrainer::LossFncWeight0>(parameters).get() / float(input_size))) };
    loss_function_helper1.loss_function_grads_ = { std::make_shared<BCEWithLogitsLossGradOp<float>>(BCEWithLogitsLossGradOp<float>(1e-6, std::get<EvoNetParameters::ModelTrainer::LossFncWeight0>(parameters).get() / float(input_size))) };
    loss_function_helpers.push_back(loss_function_helper1);
  }
  model_trainer.setLossFunctionHelpers(loss_function_helpers);

  std::vector<MetricFunctionHelper<float>> metric_function_helpers;
  MetricFunctionHelper<float> metric_function_helper1;
  if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("Points") != std::string::npos) {
    metric_function_helper1.output_nodes_ = output_nodes;
    metric_function_helper1.metric_functions_ = { std::make_shared<MAEOp<float>>(MAEOp<float>()) };
    metric_function_helper1.metric_names_ = { "Reconstruction-MAE" };
    metric_function_helpers.push_back(metric_function_helper1);
  }
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("IsApex") != std::string::npos) {
    metric_function_helper1.output_nodes_ = output_nodes;
    metric_function_helper1.metric_functions_ = { std::make_shared<PrecisionBCOp<float>>(PrecisionBCOp<float>()) };
    metric_function_helper1.metric_names_ = { "IsPeakApex-PrecisionBC" };
    metric_function_helpers.push_back(metric_function_helper1);
  }
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("IsPeak") != std::string::npos) {
    metric_function_helper1.output_nodes_ = output_nodes;
    metric_function_helper1.metric_functions_ = { std::make_shared<PrecisionBCOp<float>>(PrecisionBCOp<float>()) };
    metric_function_helper1.metric_names_ = { "IsPeak-PrecisionBC" };
    metric_function_helpers.push_back(metric_function_helper1);
  }
  else if (std::get<EvoNetParameters::Examples::SimulationType>(parameters).get().find("EMG") != std::string::npos) {
    metric_function_helper1.output_nodes_ = output_nodes;
    metric_function_helper1.metric_functions_ = { std::make_shared<MAEOp<float>>(MAEOp<float>()) };
    metric_function_helper1.metric_names_ = { "EMGParam-MAE" };
    metric_function_helpers.push_back(metric_function_helper1);
  }
  model_trainer.setMetricFunctionHelpers(metric_function_helpers);

  // define the initial population
  Model<float> model;
  if (std::get<EvoNetParameters::Main::MakeModel>(parameters).get()) {
    std::cout << "Making the model..." << std::endl;
    if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "DenoisingAE") {
      model_trainer.makeDenoisingAE(model, input_size, encoding_size,
        std::get<EvoNetParameters::ModelTrainer::NHidden0>(parameters).get(),
        std::get<EvoNetParameters::ModelTrainer::NHidden1>(parameters).get(),
        std::get<EvoNetParameters::ModelTrainer::NHidden2>(parameters).get(),
        true);
    }
    if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "VAE") {
      // TODO
      //model_trainer.makeVAE(model, input_size, encoding_size,
      //  std::get<EvoNetParameters::ModelTrainer::NHidden0>(parameters).get(),
      //  std::get<EvoNetParameters::ModelTrainer::NHidden1>(parameters).get(),
      //  std::get<EvoNetParameters::ModelTrainer::NHidden2>(parameters).get(),
      //  true);
    }
    else if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "EncoderEMG") {
      model_trainer.makeEncoderEMG(model, input_size, 4,
        std::get<EvoNetParameters::ModelTrainer::NHidden0>(parameters).get(),
        std::get<EvoNetParameters::ModelTrainer::NHidden1>(parameters).get(),
        std::get<EvoNetParameters::ModelTrainer::NHidden2>(parameters).get(),
        true);
    }
    else if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "MPNN") {
      // TODO
    }
    model.setId(0);
  }
  else {
    ModelFile<float> model_file;
    ModelInterpreterFileGpu<float> model_interpreter_file;
    loadModelFromParameters(model, model_interpreters.at(0), model_file, model_interpreter_file, args...);
  }
  model.setName(std::get<EvoNetParameters::General::DataDir>(parameters).get() + std::get<EvoNetParameters::Main::ModelName>(parameters).get()); //So that all output will be written to a specific directory

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, data_simulator,
    input_nodes, model_logger, model_interpreters.front());
}

int main(int argc, char** argv)
{
  // Parse the user commands
  int id_int = -1;
  std::string parameters_filename = "";
  parseCommandLineArguments(argc, argv, id_int, parameters_filename);

  // Set the parameter names and defaults
  EvoNetParameters::General::ID id("id", -1);
  EvoNetParameters::General::DataDir data_dir("data_dir", std::string(""));
  EvoNetParameters::Main::DeviceId device_id("device_id", 0);
  EvoNetParameters::Main::ModelName model_name("model_name", "");
  EvoNetParameters::Main::MakeModel make_model("make_model", true);
  EvoNetParameters::Main::LoadModelCsv load_model_csv("load_model_csv", false);
  EvoNetParameters::Main::LoadModelBinary load_model_binary("load_model_binary", false);
  EvoNetParameters::Main::TrainModel train_model("train_model", true);
  EvoNetParameters::Main::EvolveModel evolve_model("evolve_model", false);
  EvoNetParameters::Main::EvaluateModel evaluate_model("evaluate_model", false);
  EvoNetParameters::Main::EvaluateModels evaluate_models("evaluate_models", false);
  EvoNetParameters::Examples::ModelType model_type("model_type", "Solution");
  EvoNetParameters::Examples::SimulationType simulation_type("simulation_type", "");
  EvoNetParameters::PopulationTrainer::PopulationName population_name("population_name", "");
  EvoNetParameters::PopulationTrainer::NGenerations n_generations("n_generations", 1);
  EvoNetParameters::PopulationTrainer::NInterpreters n_interpreters("n_interpreters", 1);
  EvoNetParameters::PopulationTrainer::PruneModelNum prune_model_num("prune_model_num", 10);
  EvoNetParameters::PopulationTrainer::RemoveIsolatedNodes remove_isolated_nodes("remove_isolated_nodes", true);
  EvoNetParameters::PopulationTrainer::CheckCompleteModelInputToOutput check_complete_model_input_to_output("check_complete_model_input_to_output", true);
  EvoNetParameters::PopulationTrainer::PopulationSize population_size("population_size", 128);
  EvoNetParameters::PopulationTrainer::NTop n_top("n_top", 8);
  EvoNetParameters::PopulationTrainer::NRandom n_random("n_random", 8);
  EvoNetParameters::PopulationTrainer::NReplicatesPerModel n_replicates_per_model("n_replicates_per_model", 1);
  EvoNetParameters::PopulationTrainer::ResetModelCopyWeights reset_model_copy_weights("reset_model_copy_weights", true);
  EvoNetParameters::PopulationTrainer::ResetModelTemplateWeights reset_model_template_weights("reset_model_template_weights", true);
  EvoNetParameters::PopulationTrainer::Logging population_logging("population_logging", true);
  EvoNetParameters::PopulationTrainer::SetPopulationSizeFixed set_population_size_fixed("set_population_size_fixed", false);
  EvoNetParameters::PopulationTrainer::SetPopulationSizeDoubling set_population_size_doubling("set_population_size_doubling", true);
  EvoNetParameters::PopulationTrainer::SetTrainingStepsByModelSize set_training_steps_by_model_size("set_training_steps_by_model_size", false);
  EvoNetParameters::ModelTrainer::BatchSize batch_size("batch_size", 32);
  EvoNetParameters::ModelTrainer::MemorySize memory_size("memory_size", 64);
  EvoNetParameters::ModelTrainer::NEpochsTraining n_epochs_training("n_epochs_training", 1000);
  EvoNetParameters::ModelTrainer::NEpochsValidation n_epochs_validation("n_epochs_validation", 25);
  EvoNetParameters::ModelTrainer::NEpochsEvaluation n_epochs_evaluation("n_epochs_evaluation", 10);
  EvoNetParameters::ModelTrainer::NTBTTSteps n_tbtt_steps("n_tbtt_steps", 64);
  EvoNetParameters::ModelTrainer::NTETTSteps n_tett_steps("n_tett_steps", 64);
  EvoNetParameters::ModelTrainer::Verbosity verbosity("verbosity", 1);
  EvoNetParameters::ModelTrainer::LoggingTraining logging_training("logging_training", true);
  EvoNetParameters::ModelTrainer::LoggingValidation logging_validation("logging_validation", false);
  EvoNetParameters::ModelTrainer::LoggingEvaluation logging_evaluation("logging_evaluation", true);
  EvoNetParameters::ModelTrainer::FindCycles find_cycles("find_cycles", true);
  EvoNetParameters::ModelTrainer::FastInterpreter fast_interpreter("fast_interpreter", true);
  EvoNetParameters::ModelTrainer::PreserveOoO preserve_ooo("preserve_ooo", true);
  EvoNetParameters::ModelTrainer::InterpretModel interpret_model("interpret_model", true);
  EvoNetParameters::ModelTrainer::ResetModel reset_model("reset_model", false);
  EvoNetParameters::ModelTrainer::NHidden0 n_hidden_0("n_hidden_0", 512);
  EvoNetParameters::ModelTrainer::NHidden1 n_hidden_1("n_hidden_1", 256);
  EvoNetParameters::ModelTrainer::NHidden2 n_hidden_2("n_hidden_2", 128);
  EvoNetParameters::ModelTrainer::LossFncWeight0 loss_fnc_weight_0("loss_fnc_weight_0", 1);
  EvoNetParameters::ModelTrainer::LossFncWeight1 loss_fnc_weight_1("loss_fnc_weight_1", 1e-6);
  EvoNetParameters::ModelTrainer::LossFncWeight2 loss_fnc_weight_2("loss_fnc_weight_2", 1e-6);
  EvoNetParameters::ModelTrainer::ResetInterpreter reset_interpreter("reset_interpreter", true);
  EvoNetParameters::ModelReplicator::NNodeDownAdditionsLB n_node_down_additions_lb("n_node_down_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeRightAdditionsLB n_node_right_additions_lb("n_node_right_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDownCopiesLB n_node_down_copies_lb("n_node_down_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeRightCopiesLB n_node_right_copies_lb("n_node_right_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkAdditionsLB n_link_additons_lb("n_link_additons_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkCopiesLB n_link_copies_lb("n_link_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDeletionsLB n_node_deletions_lb("n_node_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkDeletionsLB n_link_deletions_lb("n_link_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeActivationChangesLB n_node_activation_changes_lb("n_node_activation_changes_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeIntegrationChangesLB n_node_integration_changes_lb("n_node_integration_changes_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleAdditionsLB n_module_additions_lb("n_module_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleCopiesLB n_module_copies_lb("n_module_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleDeletionsLB n_module_deletions_lb("n_module_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDownAdditionsUB n_node_down_additions_ub("n_node_down_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeRightAdditionsUB n_node_right_additions_ub("n_node_right_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeDownCopiesUB n_node_down_copies_ub("n_node_down_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeRightCopiesUB n_node_right_copies_ub("n_node_right_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkAdditionsUB n_link_additons_ub("n_link_additons_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkCopiesUB n_link_copies_ub("n_link_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeDeletionsUB n_node_deletions_ub("n_node_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkDeletionsUB n_link_deletions_ub("n_link_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeActivationChangesUB n_node_activation_changes_ub("n_node_activation_changes_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeIntegrationChangesUB n_node_integration_changes_ub("n_node_integration_changes_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleAdditionsUB n_module_additions_ub("n_module_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleCopiesUB n_module_copies_ub("n_module_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleDeletionsUB n_module_deletions_ub("n_module_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::SetModificationRateFixed set_modification_rate_fixed("set_modification_rate_fixed", false);
  EvoNetParameters::ModelReplicator::SetModificationRateByPrevError set_modification_rate_by_prev_error("set_modification_rate_by_prev_error", false);
  auto parameters = std::make_tuple(id, data_dir,
    device_id, model_name, make_model, load_model_csv, load_model_binary, train_model, evolve_model, evaluate_model, evaluate_models,
    model_type, simulation_type,
    population_name, n_generations, n_interpreters, prune_model_num, remove_isolated_nodes, check_complete_model_input_to_output, population_size, n_top, n_random, n_replicates_per_model, reset_model_copy_weights, reset_model_template_weights, population_logging, set_population_size_fixed, set_population_size_doubling, set_training_steps_by_model_size,
    batch_size, memory_size, n_epochs_training, n_epochs_validation, n_epochs_evaluation, n_tbtt_steps, n_tett_steps, verbosity, logging_training, logging_validation, logging_evaluation, find_cycles, fast_interpreter, preserve_ooo, interpret_model, reset_model, n_hidden_0, n_hidden_1, n_hidden_2, loss_fnc_weight_0, loss_fnc_weight_1, loss_fnc_weight_2, reset_interpreter,
    n_node_down_additions_lb, n_node_right_additions_lb, n_node_down_copies_lb, n_node_right_copies_lb, n_link_additons_lb, n_link_copies_lb, n_node_deletions_lb, n_link_deletions_lb, n_node_activation_changes_lb, n_node_integration_changes_lb, n_module_additions_lb, n_module_copies_lb, n_module_deletions_lb, n_node_down_additions_ub, n_node_right_additions_ub, n_node_down_copies_ub, n_node_right_copies_ub, n_link_additons_ub, n_link_copies_ub, n_node_deletions_ub, n_link_deletions_ub, n_node_activation_changes_ub, n_node_integration_changes_ub, n_module_additions_ub, n_module_copies_ub, n_module_deletions_ub, set_modification_rate_fixed, set_modification_rate_by_prev_error);

  // Read in the parameters
  LoadParametersFromCsv loadParametersFromCsv(id_int, parameters_filename);
  parameters = SmartPeak::apply([&loadParametersFromCsv](auto&& ...args) { return loadParametersFromCsv(args...); }, parameters);

  // Run the application
  SmartPeak::apply([](auto&& ...args) { main_(args ...); }, parameters);
  return 0;
}