#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicator.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>
#include <SmartPeak/simulator/BiochemicalReaction.h>
#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

// Other extended classes
template<typename TensorT>
class ModelReplicatorExt : public ModelReplicator<TensorT>
{};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerGpu<TensorT>
{};

template<typename TensorT>
class MetDataSimClassification : public DataSimulator<TensorT>
{
public:
  void simulateData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);
    const int n_epochs = input_data.dimension(3);

    // NOTE: used for testing
    //std::string sample_group_name = sample_group_names_[0];
    //std::vector<float> mars;
    //for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
    //	float mar = calculateMAR(metabolomicsData_.at(sample_group_name),
    //		biochemicalReactions_.at(reaction_ids_[nodes_iter]));
    //	mars.push_back(mar);
    //	//std::cout << "OutputNode: "<<nodes_iter<< " = " << mar << std::endl;
    //}

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        for (int epochs_iter = 0; epochs_iter < n_epochs; ++epochs_iter) {

          // pick a random sample group name
          std::string sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);

          for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
            input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = this->model_training_.calculateMAR(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_[nodes_iter]));
            //input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = mars[nodes_iter]; // NOTE: used for testing
          }

          // convert the label to a one hot vector
          Eigen::Tensor<TensorT, 1> one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
          Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

          //// MSE + LogLoss
          //for (int nodes_iter = 0; nodes_iter < n_output_nodes/2; ++nodes_iter) {
          //	output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = one_hot_vec(nodes_iter);
          //	output_data(batch_iter, memory_iter, nodes_iter + n_output_nodes/2, epochs_iter) = one_hot_vec(nodes_iter);
          //	//output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = one_hot_vec_smoothed(nodes_iter);
          //}

          // MSE or LogLoss only
          for (int nodes_iter = 0; nodes_iter < n_output_nodes; ++nodes_iter) {
            output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = one_hot_vec(nodes_iter);
            //output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = one_hot_vec_smoothed(nodes_iter);
          }
        }
      }
    }

    // update the time_steps
    time_steps.setConstant(1.0f);
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps) {
    simulateData(input_data, output_data, time_steps);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps) {
    simulateData(input_data, output_data, time_steps);
  }
  void simulateDataClassMARs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);

    if (train)
      assert(n_input_nodes == this->model_training_.reaction_ids_.size());
    else
      assert(n_input_nodes == this->model_validation_.reaction_ids_.size());

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        //// generate the input data
        //Eigen::Tensor<TensorT, 1> conc_data(n_input_nodes);
        //for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
        //  conc_data(nodes_iter) = this->model_training_.calculateMAR(
        //    this->model_training_.metabolomicsData_.at(sample_group_name),
        //    this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_[nodes_iter]));
        //  //input_data(batch_iter, memory_iter, nodes_iter) = mars[nodes_iter]; // NOTE: used for testing
        //}

        //// pre-process the data
        //if (this->log_transform_input_) {
        //  conc_data = conc_data.log();
        //  //std::cout << "Log transformed: \n" << conc_data << std::endl;
        //}
        //if (this->linear_scale_input_) {
        //  Eigen::Tensor<TensorT, 0> min_v = conc_data.minimum();
        //  Eigen::Tensor<TensorT, 0> max_v = conc_data.maximum();
        //  conc_data = conc_data.unaryExpr(LinearScale<TensorT>(min_v(0), max_v(0), 0, 1));
        //  //std::cout << "Linear scaled: \n"<< conc_data << std::endl;
        //}
        //if (this->standardize_input_) {
        //  // Calculate the mean
        //  Eigen::Tensor<TensorT, 0> mean_v = conc_data.mean();
        //  //std::cout << "Mean" << mean_v << std::endl;
        //  // Calculate the variance
        //  auto residuals = conc_data - conc_data.constant(mean_v(0));
        //  auto ssr = residuals.pow(2).sum();
        //  Eigen::Tensor<TensorT, 0> var_v = ssr / ssr.constant(n_input_nodes - 1);
        //  //std::cout << "Var" << var_v << std::endl;
        //  // Standardize
        //  conc_data = residuals / conc_data.constant(var_v(0)).pow(0.5);
        //  //std::cout << "Standardized: \n" << conc_data << std::endl;
        //}

        // assign the input data
        for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
          //input_data(batch_iter, memory_iter, nodes_iter) = conc_data(nodes_iter);
          TensorT value;
          if (train)
            value = this->model_training_.calculateMAR(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_.at(nodes_iter)));
          else
            value = this->model_validation_.calculateMAR(
              this->model_validation_.metabolomicsData_.at(sample_group_name),
              this->model_validation_.biochemicalReactions_.at(this->model_validation_.reaction_ids_.at(nodes_iter)));
          input_data(batch_iter, memory_iter, nodes_iter) = value;
        }

        // convert the label to a one hot vector        
        Eigen::Tensor<TensorT, 1> one_hot_vec((int)this->model_training_.labels_.size());
        if (train)
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
        else
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_validation_.metaData_.at(sample_group_name).condition, this->model_validation_.labels_);
        Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

        // MSE or LogLoss only
        size_t n_labels;
        if (train)
          n_labels = this->model_training_.labels_.size();
        else
          n_labels = this->model_validation_.labels_.size();
        for (int nodes_iter = 0; nodes_iter < n_labels; ++nodes_iter) {
          loss_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec(nodes_iter);
          loss_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec(nodes_iter);
        }
      }
    }

    // update the time_steps
    time_steps.setConstant(1.0f);
  }
  void simulateDataClassSampleConcs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);

    if (train)
      assert(n_input_nodes == this->model_training_.component_group_names_.size());
    else
      assert(n_input_nodes == this->model_validation_.component_group_names_.size());

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        // assign the input data
        for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
          TensorT value;
          if (train)
            value = this->model_training_.getRandomConcentration(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.component_group_names_.at(nodes_iter));
          else
            value = this->model_validation_.getRandomConcentration(
              this->model_validation_.metabolomicsData_.at(sample_group_name),
              this->model_validation_.component_group_names_.at(nodes_iter));
          input_data(batch_iter, memory_iter, nodes_iter) = value;
        }

        // convert the label to a one hot vector      
        Eigen::Tensor<TensorT, 1> one_hot_vec((int)this->model_training_.labels_.size());
        if (train)
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
        else
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_validation_.metaData_.at(sample_group_name).condition, this->model_validation_.labels_);
        Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

        // MSE or LogLoss only
        size_t n_labels;
        if (train)
          n_labels = this->model_training_.labels_.size();
        else
          n_labels = this->model_validation_.labels_.size();
        for (int nodes_iter = 0; nodes_iter < n_labels; ++nodes_iter) {
          loss_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec_smoothed(nodes_iter);
          loss_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec_smoothed(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec_smoothed(nodes_iter);
        }
      }
    }

    // update the time_steps
    time_steps.setConstant(1.0f);
  }
  void simulateDataClassConcs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);

    if (train)
      assert(n_input_nodes == this->model_training_.component_group_names_.size());
    else
      assert(n_input_nodes == this->model_validation_.component_group_names_.size());

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        int max_replicates = 0;
        if (train) {
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
          max_replicates = this->model_training_.metabolomicsData_.at(sample_group_name).at(this->model_training_.component_group_names_.at(0)).size();
        }
        else {
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);
          max_replicates = this->model_validation_.metabolomicsData_.at(sample_group_name).at(this->model_validation_.component_group_names_.at(0)).size();
        }

        // pick a random replicate
        std::vector<int> replicates;
        for (int i = 0; i < max_replicates; ++i) {
          replicates.push_back(i);
        }
        const int replicate = selectRandomElement(replicates);

        // assign the input data
        for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
          TensorT value;
          if (train)
            value = this->model_training_.metabolomicsData_.at(sample_group_name).at(this->model_training_.component_group_names_.at(nodes_iter)).at(replicate).calculated_concentration;
          else
            value = this->model_validation_.metabolomicsData_.at(sample_group_name).at(this->model_validation_.component_group_names_.at(nodes_iter)).at(replicate).calculated_concentration;
          input_data(batch_iter, memory_iter, nodes_iter) = value;
        }

        // convert the label to a one hot vector      
        Eigen::Tensor<TensorT, 1> one_hot_vec((int)this->model_training_.labels_.size());
        if (train)
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
        else
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_validation_.metaData_.at(sample_group_name).condition, this->model_validation_.labels_);
        Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

        // MSE or LogLoss only
        size_t n_labels;
        if (train)
          n_labels = this->model_training_.labels_.size();
        else
          n_labels = this->model_validation_.labels_.size();
        for (int nodes_iter = 0; nodes_iter < n_labels; ++nodes_iter) {
          loss_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec_smoothed(nodes_iter);
          loss_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter) = one_hot_vec_smoothed(nodes_iter);
          metric_output_data(batch_iter, memory_iter, nodes_iter + (int)n_labels) = one_hot_vec_smoothed(nodes_iter);
        }
      }
    }

    // update the time_steps
    time_steps.setConstant(1.0f);
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataClassMARs(input_data, loss_output_data, metric_output_data, time_steps, true);
    else if (sample_concs_) simulateDataClassSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, true);
    else simulateDataClassConcs(input_data, loss_output_data, metric_output_data, time_steps, true);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataClassMARs(input_data, loss_output_data, metric_output_data, time_steps, false);
    else if (sample_concs_) simulateDataClassSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, false);
    else simulateDataClassConcs(input_data, loss_output_data, metric_output_data, time_steps, false);
  }

  BiochemicalReactionModel<TensorT> model_training_;
  BiochemicalReactionModel<TensorT> model_validation_;
  //bool log_transform_input_ = false;
  //bool linear_scale_input_ = false;
  //bool standardize_input_ = false;
  bool sample_concs_ = false;
  bool simulate_MARs_ = true;
};

template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  /*
  @brief Fully connected classifier
  */
  void makeModelFCClass(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, const bool& add_norm = true,
    const int& n_hidden_0 = 32, const int& n_hidden_1 = 0, const int& n_hidden_2 = 0) {
    model.setId(0);
    model.setName("Classifier");
    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names, linear_scale_input, log_transform_input, standardize_input);

    // Add the hidden layers
    if (n_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC0", "FC0", node_names, n_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC0-Norm", "FC0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC0-Norm-gain", "FC0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC1-Norm", "FC1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC1-Norm-gain", "FC1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC2", "FC2", node_names, n_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC2-Norm", "FC2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC2-Norm-gain", "FC2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief CovNet classifier
  */
  void makeModelCovNetClass(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input,
    int n_hidden_0 = 64, int n_depth_1 = 32, int n_depth_2 = 2, int n_fc = 16, bool add_norm = false, bool specify_layers = false) {
    model.setId(0);
    model.setName("CovNet");

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, specify_layers);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the hidden layers
    if (n_hidden_0 > 0) {
      node_names_input = model_builder.addFullyConnected(model, "FC0", "FC0", node_names_input, n_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_input.size() + n_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names_input = model_builder.addNormalization(model, "FC0-Norm", "FC0-Norm", node_names_input, true);
        node_names_input = model_builder.addSinglyConnected(model, "FC0-Norm-gain", "FC0-Norm-gain", node_names_input, node_names_input.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
          0.0, 0.0, true, true);
      }
    }

    // Add the first convolution -> max pool -> LeakyReLU layers
    std::vector<std::vector<std::string>> node_names_l0;
    for (size_t d = 0; d < n_depth_1; ++d) {
      std::vector<std::string> node_names;
      std::string conv_name = "Conv0-" + std::to_string(d);
      node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_input,
        sqrt(node_names_input.size()), sqrt(node_names_input.size()), 0, 0,
        2, 2, 1, 0, 0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(5, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
      if (add_norm) {
        std::string norm_name = "Norm0-" + std::to_string(d);
        node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names, true);
        std::string gain_name = "Gain0-" + std::to_string(d);
        node_names = model_builder.addSinglyConnected(model, gain_name, gain_name, node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
          0.0, 0.0, true, true);
      }
      //std::string pool_name = "Pool0-" + std::to_string(d);
      //node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
      //  sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
      //  2, 2, 2, 0, 0,
      //  std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      //  std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      //  std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
      //  std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
      //  std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
      //  std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
      //  std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
      node_names_l0.push_back(node_names);
    }

    // Add the second convolution -> max pool -> LeakyReLU layers
    std::vector<std::vector<std::string>> node_names_l1;
    int l_cnt = 0;
    for (const std::vector<std::string> &node_names_l : node_names_l0) {
      for (size_t d = 0; d < n_depth_2; ++d) {
        std::vector<std::string> node_names;
        std::string conv_name = "Conv1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_l,
          sqrt(node_names_l.size()), sqrt(node_names_l.size()), 0, 0,
          2, 2, 1, 0, 0,
          std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
          std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(5, 1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
        if (add_norm) {
          std::string norm_name = "Norm1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
          node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names, true);
          std::string gain_name = "Gain1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
          node_names = model_builder.addSinglyConnected(model, gain_name, gain_name, node_names, node_names.size(),
            std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
            std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
            std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
            std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
            std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
            std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
            std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
            0.0, 0.0, true, true);
        }
        //std::string pool_name = "Pool1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        //node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
        //  sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
        //  2, 2, 2, 0, 0,
        //  std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //  std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        //  std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
        //  std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
        //  std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
        //  std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
        //  std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
        node_names_l1.push_back(node_names);
      }
      ++l_cnt;
    }

    // Linearize the node names
    std::vector<std::string> node_names;
    if (node_names_l1.size()) {
      for (const std::vector<std::string> &node_names_l : node_names_l1) {
        for (const std::string &node_name : node_names_l) {
          node_names.push_back(node_name);
        }
      }
    }
    else {
      for (const std::vector<std::string> &node_names_l : node_names_l0) {
        for (const std::string &node_name : node_names_l) {
          node_names.push_back(node_name);
        }
      }
    }

    // Add the FC layers
    node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_fc,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size() + n_fc, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
    if (add_norm) {
      node_names = model_builder.addNormalization(model, "FC1-Norm", "FC1-Norm", node_names, true);
      node_names = model_builder.addSinglyConnected(model, "FC1-Norm-gain", "FC1-Norm-gain", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
        0.0, 0.0, true, true);
    }
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size(), 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, true);

    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief Add data preprocessing steps
  */
  void addDataPreproccessingSteps(Model<TensorT>& model, std::vector<std::string>& node_names, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input) {
    ModelBuilder<TensorT> model_builder;
    // Data pre-processing steps
    if (log_transform_input) {
      node_names = model_builder.addSinglyConnected(model, "LogScaleInput", "LogScaleInput", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LogOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LogGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, true);
    }
    if (linear_scale_input) {
      node_names = model_builder.addLinearScale(model, "LinearScaleInput", "LinearScaleInput", node_names, 0, 1, true);
    }
    if (standardize_input) {
      node_names = model_builder.addNormalization(model, "StandardizeInput", "StandardizeInput", node_names, true);
    }
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors) {
    // Check point the model every 1000 epochs
    if (n_epochs % 1000 == 0 && n_epochs != 0) {
      model_interpreter.getModelResults(model, false, true, false);
      // save the model weights
      WeightFile<float> weight_data;
      weight_data.storeWeightValuesCsv(model.getName() + "_" + std::to_string(n_epochs) + "_weights.csv", model.weights_);
      //// save the model and interpreter in binary format
      //ModelFile<TensorT> data;
      //data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
      //ModelInterpreterFileGpu<TensorT> interpreter_data;
      //interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    }
  }
  void trainingModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const TensorT & model_error_train, const TensorT & model_error_test,
    const Eigen::Tensor<TensorT, 1> & model_metrics_train, const Eigen::Tensor<TensorT, 1> & model_metrics_test)
  {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedPredictedEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 1000 == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_interpreter.getModelResults(model, true, false, false);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_train_headers = { "Train_Error" };
    std::vector<std::string> log_test_headers = { "Test_Error" };
    std::vector<TensorT> log_train_values = { model_error_train };
    std::vector<TensorT> log_test_values = { model_error_test };
    int metric_iter = 0;
    for (const std::string& metric_name : this->metric_names_) {
      log_train_headers.push_back(metric_name);
      log_test_headers.push_back(metric_name);
      log_train_values.push_back(model_metrics_train(metric_iter));
      log_test_values.push_back(model_metrics_test(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_train_headers, log_test_headers, log_train_values, log_test_values, output_nodes, expected_values);
  }
};

/// Script to run the classification network
void main_classification(const std::string& biochem_rxns_filename,
  const std::string& metabo_data_filename_train, const std::string& meta_data_filename_train,
  const std::string& metabo_data_filename_test, const std::string& meta_data_filename_test,
  bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimClassification<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data

  // Training data
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_train);
  reaction_model.readMetaData(meta_data_filename_train);
  reaction_model.findComponentGroupNames();
  if (simulate_MARs) {
    reaction_model.findMARs();
    reaction_model.findMARs(true, false);
    reaction_model.findMARs(false, true);
    reaction_model.removeRedundantMARs();
  }
  reaction_model.findLabels();
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_test);
  reaction_model.readMetaData(meta_data_filename_test);
  reaction_model.findComponentGroupNames();
  if (simulate_MARs) {
    reaction_model.findMARs();
    reaction_model.findMARs(true, false);
    reaction_model.findMARs(false, true);
    reaction_model.removeRedundantMARs();
  }
  reaction_model.findLabels();
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes = reaction_model.labels_.size();

  // define the input nodes
  std::vector<std::string> input_nodes;
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // define the output nodes
  std::vector<std::string> output_nodes;
  for (int i = 0; i < n_output_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Output_%012d", i);
    std::string name(name_char);
    output_nodes.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(64);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(10000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new CrossEntropyWithLogitsOp<float>()),
    std::shared_ptr<LossFunctionOp<float>>(new MSELossOp<float>()) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new CrossEntropyWithLogitsGradOp<float>()),
    std::shared_ptr<LossFunctionGradOp<float>>(new MSELossGradOp<float>()) });
  model_trainer.setLossOutputNodes({
    output_nodes,
    output_nodes });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new AccuracyMCMicroOp<float>()), std::shared_ptr<MetricFunctionOp<float>>(new PrecisionMCMicroOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes, output_nodes });
  model_trainer.setMetricNames({ "AccuracyMCMicro", "PrecisionMCMicro" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, false, false, false, false, 64, 64, 0); // normalization type 0
    model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, false, false, false, 64, 64, 0); // normalization type 1
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, false, true, false, 64, 64, 0); // normalization type 2
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, true, false, false, 64, 64, 0); // normalization type 3
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, true, true, false, 64, 64, 0); // normalization type 4

    //model_trainer.makeModelCovNetClass(model, n_input_nodes, n_output_nodes, true, true, false, 64, 16, 0, 32, false, true); // normalization type 3

    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

// Main
int main(int argc, char** argv)
{
  // Set the data directories
  //const std::string data_dir = "C:/Users/dmccloskey/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  const std::string data_dir = "C:/Users/domccl/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  //const std::string data_dir = "/home/user/Data/";

  // Make the filenames
  const std::string biochem_rxns_filename = data_dir + "iJO1366.csv";

  // ALEsKOs01
  const std::string metabo_data_filename_train = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  const std::string meta_data_filename_train = data_dir + "ALEsKOs01_MetaData_train.csv";
  const std::string metabo_data_filename_test = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  const std::string meta_data_filename_test = data_dir + "ALEsKOs01_MetaData_test.csv";

  //// IndustrialStrains0103
  //const std::string metabo_data_filename_train = data_dir + "IndustrialStrains0103_Metabolomics_train.csv";
  //const std::string meta_data_filename_train = data_dir + "IndustrialStrains0103_MetaData_train.csv";
  //const std::string metabo_data_filename_test = data_dir + "IndustrialStrains0103_Metabolomics_test.csv";
  //const std::string meta_data_filename_test = data_dir + "IndustrialStrains0103_MetaData_test.csv";

  main_classification(biochem_rxns_filename, metabo_data_filename_train, meta_data_filename_train,
    metabo_data_filename_test, meta_data_filename_test, true, false, true);
  return 0;
}