#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicator.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/ml/Model.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>
#include <SmartPeak/io/ModelFile.h>

#include <SmartPeak/simulator/MNISTSimulator.h>

#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

/**
 * EXAMPLES using the MNIST data set
 *
 * EXAMPLE1:
 * - classification on MNIST using DAG
 * - whole image pixels (linearized) 28x28 normalized to 0 to 1
 * - classifier (1 hot vector from 0 to 9)
 */

 // Extended classes
template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  /*
  @brief Convolution classifier

  @param n_depth_1 32 (32 filters)
  @param n_depth_2 2 (total of 64 filters)
  @param n_fc 1024
  @param add_norm Optional normalization layer after each convolution

  References:
  https://github.com/pytorch/examples/blob/master/mnist/main.py
  */
  void makeCovNet(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, int n_depth_1 = 32, int n_depth_2 = 2, int n_fc = 128, bool add_norm = false, bool specify_layers = false) {
    model.setId(0);
    model.setName("CovNet");

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, specify_layers);

    // Add the first convolution -> max pool -> ReLU layers
    std::vector<std::vector<std::string>> node_names_l0;
    for (size_t d = 0; d < n_depth_1; ++d) {
      std::vector<std::string> node_names;
      std::string conv_name = "Conv0-" + std::to_string(d);
      node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_input,
        28, 28, 0, 0,
        5, 5, 1, 0, 0,
        std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, specify_layers);
      if (add_norm) {
        std::string norm_name = "Norm0-" + std::to_string(d);
        node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names,
          std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
          std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size(), 2)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.1, 0.9, 0.999, 1e-8)), 0.0, 0.0, false, specify_layers);
      }
      std::string pool_name = "Pool0-" + std::to_string(d);
      node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
        sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
        2, 2, 2, 0, 0,
        std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
        std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
      node_names_l0.push_back(node_names);
    }

    // Add the second convolution -> max pool -> ReLU layers
    std::vector<std::vector<std::string>> node_names_l1;
    int l_cnt = 0;
    for (const std::vector<std::string> &node_names_l : node_names_l0) {
      for (size_t d = 0; d < n_depth_2; ++d) {
        std::vector<std::string> node_names;
        std::string conv_name = "Conv1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_l,
          sqrt(node_names_l.size()), sqrt(node_names_l.size()), 0, 0,
          5, 5, 1, 0, 0,
          std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
          std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, specify_layers);
        if (add_norm) {
          std::string norm_name = "Norm1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
          node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names,
            std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
            std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
            std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size(), 2)),
            std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.1, 0.9, 0.999, 1e-8)), 0.0, 0.0, false, specify_layers);
        }
        std::string pool_name = "Pool1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
          sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
          2, 2, 2, 0, 0,
          std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
          std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
          std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
        node_names_l1.push_back(node_names);
      }
      ++l_cnt;
    }

    // Linearize the node names
    std::vector<std::string> node_names;
    //for (const std::vector<std::string> &node_names_l : node_names_l0) {
    for (const std::vector<std::string> &node_names_l : node_names_l1) {
      for (const std::string &node_name : node_names_l) {
        node_names.push_back(node_name);
      }
    }

    // Add the FC layers
    //assert(node_names.size() == 320);
    node_names = model_builder.addFullyConnected(model, "FC0", "FC0", node_names, n_fc,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(180, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, specify_layers);
    if (add_norm) {
      node_names = model_builder.addNormalization(model, "NormFC0", "NormFC0", node_names,
        std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size(), 2)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.1, 0.9, 0.999, 1e-8)), 0.0, 0.0, false, specify_layers);
    }
    node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_fc, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
  }
  /*
  @brief Convolution classifier using compact convolutions
    Current work in progress because the nodes2layers algorithm
    Does not yet recognize when weights need to be allocated to a different layer
    in order to prevent over-writing the weight values

  @param n_depth_1 32 (32 filters)
  @param n_depth_2 32 (total of 64 filters)
  @param n_fc 1024
  @param add_norm Optional normalization layer after each convolution

  References:
  https://github.com/pytorch/examples/blob/master/mnist/main.py
  */
  void makeCovNetCompact(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, int n_depth_1 = 32, int n_depth_2 = 32, int n_fc = 128, int add_scalar = true) {
    model.setId(0);
    model.setName("CovNet");

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Add the first convolution -> max pool -> ReLU layers
    std::vector<std::string> node_names_conv0;
    std::string conv_name = "Conv0-" + std::to_string(0);
    node_names_conv0 = model_builder.addConvolution(model, "Conv0", conv_name, node_names_input,
      28, 28, 0, 0,
      5, 5, 1, 0, 0,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    for (size_t d = 1; d < n_depth_1; ++d) {
      std::string conv_name = "Conv0-" + std::to_string(d);
      model_builder.addConvolution(model, "Conv0", conv_name, node_names_input, node_names_conv0,
        28, 28, 0, 0,
        5, 5, 1, 0, 0,
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true);
    }
    if (add_scalar) {
      node_names_conv0 = model_builder.addScalar(model, "Scalar0", "Scalar0", node_names_conv0, 5 * n_inputs,
        std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        true);
    }

    // Add the second convolution -> max pool -> ReLU layers
    std::vector<std::string> node_names_conv1;
    conv_name = "Conv1-" + std::to_string(0);
    node_names_conv1 = model_builder.addConvolution(model, "Conv1", conv_name, node_names_conv0,
      sqrt(node_names_conv0.size()), sqrt(node_names_conv0.size()), 0, 0,
      5, 5, 1, 0, 0,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    for (size_t d = 1; d < n_depth_2; ++d) {
      std::string conv_name = "Conv1-" + std::to_string(d);
      model_builder.addConvolution(model, "Conv1", conv_name, node_names_conv0, node_names_conv1,
        sqrt(node_names_conv0.size()), sqrt(node_names_conv0.size()), 0, 0,
        5, 5, 1, 0, 0,
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_inputs, 2)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true);
    }
    if (add_scalar) {
      node_names_conv1 = model_builder.addScalar(model, "Scalar1", "Scalar1", node_names_conv1, 5 * node_names_conv0.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        true);
    }

    // Add the FC layers
    //assert(node_names.size() == 320);
    std::vector<std::string> node_names;
    node_names = model_builder.addFullyConnected(model, "FC0", "FC0", node_names_conv1, n_fc,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(180, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true, true);
    node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(n_fc, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(0.001, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true, true);

    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors) {
    //if (n_epochs = 1000) {
    //	// anneal the learning rate to 1e-4
    //}
    if (n_epochs % 999 == 0 && n_epochs != 0) {
      // save the model every 1000 epochs
      //model_interpreter.getModelResults(model, false, true, false);
      ModelFile<TensorT> data;
      //data.storeModelCsv(model.getName() + "_" + std::to_string(n_epochs) + "_nodes.csv",
      //	model.getName() + "_" + std::to_string(n_epochs) + "_links.csv",
      //	model.getName() + "_" + std::to_string(n_epochs) + "_weights.csv", model);
      data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
      ModelInterpreterFileGpu<TensorT> interpreter_data;
      interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    }
  }
  void trainingModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values,
    const std::vector<std::string>& output_nodes,
    const TensorT& model_error)
  {
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedPredictedEpoch(false);
    if (n_epochs == 0) {
      model_logger.initLogs(model);
    }
    if (n_epochs % 10 == 0) {
      if (model_logger.getLogExpectedPredictedEpoch())
        model_interpreter.getModelResults(model, true, false, false);
      model_logger.writeLogs(model, n_epochs, { "Error" }, {}, { model_error }, {}, output_nodes, expected_values);
    }
  }
  void validationModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values,
    const std::vector<std::string>& output_nodes,
    const TensorT& model_error)
  {
    model_logger.setLogTimeEpoch(false);
    model_logger.setLogTrainValMetricEpoch(false);
    model_logger.setLogExpectedPredictedEpoch(true);
    if (n_epochs == 0) {
      model_logger.initLogs(model);
    }
    if (n_epochs % 1 == 0) {
      if (model_logger.getLogExpectedPredictedEpoch())
        model_interpreter.getModelResults(model, true, false, false);
      model_logger.writeLogs(model, n_epochs, {}, { "Error" }, {}, { model_error }, output_nodes, expected_values);
    }
  }
};

template<typename TensorT>
class DataSimulatorExt : public MNISTSimulator<TensorT>
{
public:
  void simulateEvaluationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 3>& time_steps) {};
  void simulateTrainingData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);
    const int n_epochs = input_data.dimension(3);

    assert(n_output_nodes == this->validation_labels.dimension(1));
    assert(n_input_nodes == this->validation_data.dimension(1));

    // make a vector of sample_indices [BUG FREE]
    Eigen::Tensor<int, 1> sample_indices = this->getTrainingIndices(batch_size, n_epochs);

    // Reformat the input data for training [BUG FREE]
    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        for (int epochs_iter = 0; epochs_iter < n_epochs; ++epochs_iter) {
          for (int nodes_iter = 0; nodes_iter < this->training_data.dimension(1); ++nodes_iter) {
            //input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = this->training_data(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
            input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = this->training_data(sample_indices[0], nodes_iter);  // test on only 1 sample
          }
          for (int nodes_iter = 0; nodes_iter < this->training_labels.dimension(1); ++nodes_iter) {
            //output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = (TensorT)this->training_labels(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
            output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = (TensorT)this->training_labels(sample_indices[0], nodes_iter); // test on only 1 sample
            //output_data(batch_iter, memory_iter, nodes_iter + this->training_labels.dimension(1), epochs_iter) = (TensorT)this->training_labels(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
            ////output_data(batch_iter, memory_iter, nodes_iter + this->training_labels.dimension(1), epochs_iter) = (TensorT)this->training_labels(sample_indices[0], nodes_iter); // test on only 1 sample
          }
        }
      }
    }

    time_steps.setConstant(1.0f);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);
    const int n_epochs = input_data.dimension(3);

    assert(n_output_nodes == this->validation_labels.dimension(1));
    assert(n_input_nodes == this->validation_data.dimension(1));

    // make the start and end sample indices [BUG FREE]
    Eigen::Tensor<int, 1> sample_indices = this->getValidationIndices(batch_size, n_epochs);

    // Reformat the input data for validation [BUG FREE]
    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        for (int epochs_iter = 0; epochs_iter < n_epochs; ++epochs_iter) {
          for (int nodes_iter = 0; nodes_iter < this->validation_data.dimension(1); ++nodes_iter) {
            input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = this->validation_data(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
          }
          for (int nodes_iter = 0; nodes_iter < this->validation_labels.dimension(1); ++nodes_iter) {
            output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = (TensorT)this->validation_labels(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
            //output_data(batch_iter, memory_iter, nodes_iter + this->validation_labels.dimension(1), epochs_iter) = (TensorT)this->validation_labels(sample_indices[epochs_iter*batch_size + batch_iter], nodes_iter);
          }
        }
      }
    }

    time_steps.setConstant(1.0f);
  }
};

template<typename TensorT>
class ModelReplicatorExt : public ModelReplicator<TensorT>
{
public:
  void adaptiveReplicatorScheduler(
    const int& n_generations,
    std::vector<Model<TensorT>>& models,
    std::vector<std::vector<std::tuple<int, std::string, TensorT>>>& models_errors_per_generations)
  { //TODO
  }
};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerGpu<TensorT>
{
public:
  void adaptivePopulationScheduler(
    const int& n_generations,
    std::vector<Model<TensorT>>& models,
    std::vector<std::vector<std::tuple<int, std::string, TensorT>>>& models_errors_per_generations)
  {
    // Population size of 16
    if (n_generations == 0)
    {
      this->setNTop(3);
      this->setNRandom(3);
      this->setNReplicatesPerModel(15);
    }
    else
    {
      this->setNTop(3);
      this->setNRandom(3);
      this->setNReplicatesPerModel(3);
    }
  }
};

void main_CovNet() {

  const int n_hard_threads = std::thread::hardware_concurrency();
  const int n_threads = 1;

  // define the populatin trainer
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(1);
  population_trainer.setNRandom(1);
  population_trainer.setNReplicatesPerModel(1);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false, false);
  //ModelLogger<float> model_logger(true, true, true, true, true, false, true, true);

  // define the data simulator
  const std::size_t input_size = 784;
  const std::size_t training_data_size = 60000; //60000;
  const std::size_t validation_data_size = 10000; //10000;
  DataSimulatorExt<float> data_simulator;

  // read in the training data
  const std::string training_data_filename = "C:/Users/domccl/GitHub/mnist/train-images.idx3-ubyte";
  const std::string training_labels_filename = "C:/Users/domccl/GitHub/mnist/train-labels.idx1-ubyte";
  //const std::string training_data_filename = "C:/Users/dmccloskey/Documents/GitHub/mnist/train-images-idx3-ubyte";
  //const std::string training_labels_filename = "C:/Users/dmccloskey/Documents/GitHub/mnist/train-labels-idx1-ubyte";
  //const std::string training_data_filename = "/home/user/data/train-images-idx3-ubyte";
  //const std::string training_labels_filename = "/home/user/data/train-labels-idx1-ubyte";
  data_simulator.readData(training_data_filename, training_labels_filename, true, training_data_size, input_size);

  // read in the validation data
  const std::string validation_data_filename = "C:/Users/domccl/GitHub/mnist/t10k-images.idx3-ubyte";
  const std::string validation_labels_filename = "C:/Users/domccl/GitHub/mnist/t10k-labels.idx1-ubyte";
  //const std::string validation_data_filename = "C:/Users/dmccloskey/Documents/GitHub/mnist/t10k-images-idx3-ubyte";
  //const std::string validation_labels_filename = "C:/Users/dmccloskey/Documents/GitHub/mnist/t10k-labels-idx1-ubyte";
  //const std::string validation_data_filename = "/home/user/data/t10k-images-idx3-ubyte";
  //const std::string validation_labels_filename = "/home/user/data/t10k-labels-idx1-ubyte";
  data_simulator.readData(validation_data_filename, validation_labels_filename, false, validation_data_size, input_size);
  data_simulator.unitScaleData();

  // Make the input nodes
  std::vector<std::string> input_nodes;
  for (int i = 0; i < input_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the output nodes
  std::vector<std::string> output_FC_nodes;
  for (int i = 0; i < data_simulator.mnist_labels.size(); ++i) {
    char name_char[512];
    sprintf(name_char, "FC1_%012d", i);
    std::string name(name_char);
    output_FC_nodes.push_back(name);
  }

  // Make the output nodes
  std::vector<std::string> output_nodes;
  for (int i = 0; i < data_simulator.mnist_labels.size(); ++i) {
    char name_char[512];
    sprintf(name_char, "SoftMax-Out_%012d", i);
    std::string name(name_char);
    output_nodes.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(64);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(1000);
  model_trainer.setNEpochsValidation(1);
  model_trainer.setNEpochsEvaluation(100);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, true, true);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setLossFunctions({
    //std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>())//,
    std::shared_ptr<LossFunctionOp<float>>(new CrossEntropyWithLogitsOp<float>())
    });
  model_trainer.setLossFunctionGrads({
    //std::shared_ptr<LossFunctionGradOp<float>>({new MSEGradOp<float>())//,	
    std::shared_ptr<LossFunctionGradOp<float>>(new CrossEntropyWithLogitsGradOp<float>())
    });
  model_trainer.setOutputNodes({ output_FC_nodes//, output_nodes 
    });

  // define the model replicator for growth mode
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  Model<float> model;
  //model_trainer.makeCovNet(model, input_nodes.size(), output_nodes.size(), 2, 2, 32, false, true);  // Sanity test
  model_trainer.makeCovNet(model, input_nodes.size(), output_nodes.size(), 8, 2, 128, false, true);  // Minimal solving model
  //model_trainer.makeCovNet(model, input_nodes.size(), output_nodes.size(), 32, 2, 128, true, true); // Recommended model
  //model_trainer.makeCovNetCompact(model, input_nodes.size(), output_nodes.size(), 12, 12, 128);  // Test
  std::vector<Model<float>> population = { model };

  // Evolve the population
  std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
    population, model_trainer, model_interpreters, model_replicator, data_simulator, model_logger, population_logger, input_nodes);

  PopulationTrainerFile<float> population_trainer_file;
  population_trainer_file.storeModels(population, "MNIST");
  population_trainer_file.storeModelValidations("MNISTErrors.csv", models_validation_errors_per_generation);
}

int main(int argc, char** argv)
{
  // run the application
  main_CovNet();

  return 0;
}