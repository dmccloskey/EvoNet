#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicator.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>
#include <SmartPeak/simulator/BiochemicalReaction.h>
#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

// Other extended classes
template<typename TensorT>
class ModelReplicatorExt : public ModelReplicator<TensorT>
{};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerGpu<TensorT>
{};

template<typename TensorT>
class MetDataSimMultiTask : public DataSimulator<TensorT>
{
public:
  void simulateDataMARs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);
    int n_input_pixels;
    int n_classes;
    if (train) {
      n_input_pixels = this->model_training_.reaction_ids_.size();
      n_classes = this->model_training_.labels_.size();
    }
    else {
      n_input_pixels = this->model_validation_.reaction_ids_.size();
      n_classes = this->model_validation_.labels_.size();
    }

    // Assuming MSE + XEntropy classification loass, MSE reconstruction loss, and KL divergence Mu and Var losses
    assert(n_loss_output_nodes == 2 * n_classes + n_input_pixels + 2 * n_encodings_);
    assert(n_metric_output_nodes % (n_input_pixels + 2 * n_classes) == 0);
    assert(n_input_nodes == n_input_pixels + n_encodings_);

    std::random_device rd{};
    std::mt19937 gen{ rd() };
    std::normal_distribution<> d{ 0.0f, 1.0f };

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        // convert the label to a one hot vector        
        Eigen::Tensor<TensorT, 1> one_hot_vec((int)this->model_training_.labels_.size());
        if (train)
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
        else
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_validation_.metaData_.at(sample_group_name).condition, this->model_validation_.labels_);
        Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

        // assign the input, loss_output, and metric_output node values
        for (int nodes_iter = 0; nodes_iter < n_input_pixels; ++nodes_iter) {
          if (nodes_iter < n_input_pixels) {
            TensorT value;
            if (train) value = this->model_training_.calculateMAR(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_.at(nodes_iter)));
            else value = this->model_validation_.calculateMAR(
              this->model_validation_.metabolomicsData_.at(sample_group_name),
              this->model_validation_.biochemicalReactions_.at(this->model_validation_.reaction_ids_.at(nodes_iter)));
            input_data(batch_iter, memory_iter, nodes_iter) = value; // input concentration data
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // reconstruction output loss
            metric_output_data(batch_iter, memory_iter, nodes_iter) = 0; // reconstruction output metric
          }
          if (nodes_iter < n_encodings_) {
            TensorT random_value;
            if (train) random_value = d(gen);
            else random_value = 0;
            input_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = random_value; // sample from a normal distribution
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = 0; // Dummy data for KL divergence mu
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + n_encodings_) = 0; // Dummy data for KL divergence logvar
          }
          if (nodes_iter < n_classes) {
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + 2 * n_encodings_) = one_hot_vec_smoothed(nodes_iter); // classification output loss (XEntropy)
            metric_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = one_hot_vec(nodes_iter); // classification metric loss (Accuracy)
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + 2 * n_encodings_ + n_classes) = one_hot_vec(nodes_iter); // classification output loss (MSE)
            metric_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + n_classes) = one_hot_vec(nodes_iter); // classification metric loss (Precision)
          }
        }
      }
    }
  }
  void simulateDataSampleConcs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);
    int n_input_pixels;
    int n_classes;
    if (train) {
      n_input_pixels = this->model_training_.component_group_names_.size();
      n_classes = this->model_training_.labels_.size();
    }
    else {
      n_input_pixels = this->model_validation_.component_group_names_.size();
      n_classes = this->model_validation_.labels_.size();
    }

    // Assuming MSE + XEntropy classification loass, MSE reconstruction loss, and KL divergence Mu and Var losses
    assert(n_loss_output_nodes == 2 * n_classes + n_input_pixels + 2 * n_encodings_);
    assert(n_metric_output_nodes % (n_input_pixels + 2 * n_classes) == 0);
    assert(n_input_nodes == n_input_pixels + n_encodings_);

    std::random_device rd{};
    std::mt19937 gen{ rd() };
    std::normal_distribution<> d{ 0.0f, 1.0f };

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        // convert the label to a one hot vector        
        Eigen::Tensor<TensorT, 1> one_hot_vec((int)this->model_training_.labels_.size());
        if (train)
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_training_.metaData_.at(sample_group_name).condition, this->model_training_.labels_);
        else
          one_hot_vec = OneHotEncoder<std::string, TensorT>(this->model_validation_.metaData_.at(sample_group_name).condition, this->model_validation_.labels_);
        Eigen::Tensor<TensorT, 1> one_hot_vec_smoothed = one_hot_vec.unaryExpr(LabelSmoother<TensorT>(0.01, 0.01));

        // assign the input, loss_output, and metric_output node values
        for (int nodes_iter = 0; nodes_iter < n_input_pixels; ++nodes_iter) {
          if (nodes_iter < n_input_pixels) {
            TensorT value;
            if (train) value = this->model_training_.getRandomConcentration(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.component_group_names_.at(nodes_iter));
            else value = this->model_validation_.getRandomConcentration(
              this->model_validation_.metabolomicsData_.at(sample_group_name),
              this->model_validation_.component_group_names_.at(nodes_iter));
            input_data(batch_iter, memory_iter, nodes_iter) = value; // input concentration data
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // reconstruction output loss
            metric_output_data(batch_iter, memory_iter, nodes_iter) = 0; // reconstruction output metric
          }
          if (nodes_iter < n_encodings_) {
            TensorT random_value;
            if (train) random_value = d(gen);
            else random_value = 0;
            input_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = random_value; // sample from a normal distribution
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = 0; // Dummy data for KL divergence mu
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + n_encodings_) = 0; // Dummy data for KL divergence logvar
          }
          if (nodes_iter < n_classes) {
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + 2 * n_encodings_) = one_hot_vec_smoothed(nodes_iter); // classification output loss (XEntropy)
            metric_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels) = one_hot_vec(nodes_iter); // classification metric loss (Accuracy)
            loss_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + 2 * n_encodings_ + n_classes) = one_hot_vec(nodes_iter); // classification output loss (MSE)
            metric_output_data(batch_iter, memory_iter, nodes_iter + n_input_pixels + n_classes) = one_hot_vec(nodes_iter); // classification metric loss (Precision)
          }
        }
      }
    }
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataMARs(input_data, loss_output_data, metric_output_data, time_steps, true);
    else simulateDataSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, true);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataMARs(input_data, loss_output_data, metric_output_data, time_steps, false);
    else simulateDataSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, false);
  }

  BiochemicalReactionModel<TensorT> model_training_;
  BiochemicalReactionModel<TensorT> model_validation_;
  int n_encodings_;
  bool sample_concs_ = false;
  bool simulate_MARs_ = true;
};

template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  /*
  @brief Fully connected multitask model for variational reconstruction and classification
  */
  void makeModelFCMultiTask(Model<TensorT>& model, const int& n_inputs, const int& n_outputs_recon, const int& n_outputs_class, const int& n_encodings, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, bool add_norm = true) {
    model.setId(0);
    model.setName("MultiTask");
    const int n_en_hidden_0 = 64;
    const int n_en_hidden_1 = 64;
    const int n_en_hidden_2 = 0;
    const int n_de_hidden_0 = 64;
    const int n_de_hidden_1 = 64;
    const int n_de_hidden_2 = 0;
    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the encoding layers
    std::vector<std::string> node_names = node_names_input;
    if (n_en_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN0", "EN0", node_names, n_en_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN0-Norm", "EN0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN0-Norm-gain", "EN0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN1", "EN1", node_names, n_en_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN1-Norm", "EN1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN1-Norm-gain", "EN1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN2", "EN2", node_names, n_en_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN2-Norm", "EN2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN2-Norm-gain", "EN2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the mu and log var layers
    std::vector<std::string> node_names_mu = model_builder.addFullyConnected(model, "Mu", "Mu", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    std::vector<std::string> node_names_logvar = model_builder.addFullyConnected(model, "LogVar", "LogVar", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the Encoding Mu and LogVar output nodes
    for (const std::string& node_name : node_names_mu)
      model.nodes_.at(node_name)->setType(NodeType::output);
    for (const std::string& node_name : node_names_logvar)
      model.nodes_.at(node_name)->setType(NodeType::output);

    // Add the Variational Encoding layer
    std::vector<std::string> node_names_encoding = model_builder.addGaussianEncoding(model, "Encoding", "Encoding", node_names_mu, node_names_logvar, true);

    // Add the decoding layers
    if (n_de_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE0", "DE0", node_names_encoding, n_de_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_encoding.size() + n_de_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE0-Norm", "DE0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE0-Norm-gain", "DE0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE1", "DE1", node_names, n_de_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE1-Norm", "DE1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE1-Norm-gain", "DE1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE2", "DE2", node_names, n_de_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE2-Norm", "DE2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE2-Norm-gain", "DE2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the final reconstruction output layer
    node_names = model_builder.addFullyConnected(model, "Output-Recon", "Output-Recon", node_names, n_outputs_recon,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs_recon) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    // Subtract out the pre-processed input data to test against all 0's
    model_builder.addSinglyConnected(model, "Output-Recon", node_names_input, node_names,
      std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(-1)),
      std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0f, true);

    // Specify the reconstruction output node types
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);

    // Add the classification output layer
    node_names = model_builder.addFullyConnected(model, "Output-Class", "Output-Class", node_names_mu, n_outputs_class,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_mu.size() + n_outputs_class) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the classificaiton output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);

    // Set the input and output nodes
    model.setInputAndOutputNodes();

    //// Check that the model is set-up correctly
    //if (!model.checkCompleteInputToOutput())
    //  std::cout << "There is a problem with the model!" << std::endl;
  }

  /*
  @brief Add data preprocessing steps
  */
  void addDataPreproccessingSteps(Model<TensorT>& model, std::vector<std::string>& node_names, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input) {
    ModelBuilder<TensorT> model_builder;
    // Data pre-processing steps
    if (log_transform_input) {
      node_names = model_builder.addSinglyConnected(model, "LogScaleInput", "LogScaleInput", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LogOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LogGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, true, true);
    }
    if (linear_scale_input) {
      node_names = model_builder.addLinearScale(model, "LinearScaleInput", "LinearScaleInput", node_names, 0, 1, true);
    }
    if (standardize_input) {
      node_names = model_builder.addNormalization(model, "StandardizeInput", "StandardizeInput", node_names, true);
    }
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors) {
    // Check point the model every 1000 epochs
    if (n_epochs % 1000 == 0 && n_epochs != 0) {
      model_interpreter.getModelResults(model, false, true, false, false);
      ModelFile<TensorT> data;
      data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
      ModelInterpreterFileGpu<TensorT> interpreter_data;
      interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    }
  }
  void trainingModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const TensorT & model_error_train, const TensorT & model_error_test,
    const Eigen::Tensor<TensorT, 1> & model_metrics_train, const Eigen::Tensor<TensorT, 1> & model_metrics_test)
  {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedPredictedEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 1000 == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_interpreter.getModelResults(model, true, false, false);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_train_headers = { "Train_Error" };
    std::vector<std::string> log_test_headers = { "Test_Error" };
    std::vector<TensorT> log_train_values = { model_error_train };
    std::vector<TensorT> log_test_values = { model_error_test };
    int metric_iter = 0;
    for (const std::string& metric_name : this->metric_names_) {
      log_train_headers.push_back(metric_name);
      log_test_headers.push_back(metric_name);
      log_train_values.push_back(model_metrics_train(metric_iter));
      log_test_values.push_back(model_metrics_test(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_train_headers, log_test_headers, log_train_values, log_test_values, output_nodes, expected_values);
  }
};

/// Script to run the reconstruction network
void main_multiTask(const std::string& biochem_rxns_filename,
  const std::string& metabo_data_filename_train, const std::string& meta_data_filename_train,
  const std::string& metabo_data_filename_test, const std::string& meta_data_filename_test,
  bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimMultiTask<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data

  // Training data
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_train);
  reaction_model.readMetaData(meta_data_filename_train);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_test);
  reaction_model.readMetaData(meta_data_filename_test);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // Define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes_recon = n_input_nodes;
  const int n_output_nodes_class = reaction_model.labels_.size();
  const int encoding_size = 3;
  metabolomics_data.n_encodings_ = encoding_size;

  // Make the input nodes
  std::vector<std::string> input_nodes;
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the encoding nodes and add them to the input
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Encoding_%012d-Sampler", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the reconstruction nodes
  std::vector<std::string> output_nodes_recon;
  for (int i = 0; i < n_output_nodes_recon; ++i) {
    char name_char[512];
    sprintf(name_char, "Output-Recon_%012d", i);
    std::string name(name_char);
    output_nodes_recon.push_back(name);
  }

  // Make the mu nodes
  std::vector<std::string> encoding_nodes_mu;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Mu_%012d", i);
    std::string name(name_char);
    encoding_nodes_mu.push_back(name);
  }

  // Make the encoding nodes
  std::vector<std::string> encoding_nodes_logvar;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "LogVar_%012d", i);
    std::string name(name_char);
    encoding_nodes_logvar.push_back(name);
  }

  // Make the classification nodes
  std::vector<std::string> output_nodes_class;
  for (int i = 0; i < n_output_nodes_class; ++i) {
    char name_char[512];
    sprintf(name_char, "Output-Class_%012d", i);
    std::string name(name_char);
    output_nodes_class.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(16);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(100000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new MSELossOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionOp<float>>(new BCEWithLogitsLossOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceMuLossOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceLogVarLossOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionOp<float>>(new CrossEntropyWithLogitsLossOp<float>()),
    std::shared_ptr<LossFunctionOp<float>>(new MSELossOp<float>()) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new MSELossGradOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionGradOp<float>>(new BCEWithLogitsLossGradOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceMuLossGradOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceLogVarLossGradOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionGradOp<float>>(new CrossEntropyWithLogitsLossGradOp<float>()),
    std::shared_ptr<LossFunctionGradOp<float>>(new MSELossGradOp<float>()) });
  model_trainer.setLossOutputNodes({ output_nodes_recon, encoding_nodes_mu, encoding_nodes_logvar,
    output_nodes_class, output_nodes_class });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new MAEOp<float>()),
    std::shared_ptr<MetricFunctionOp<float>>(new AccuracyMCMicroOp<float>()),
    std::shared_ptr<MetricFunctionOp<float>>(new PrecisionMCMicroOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes_recon, output_nodes_class, output_nodes_class });
  model_trainer.setMetricNames({ "MAE", "AccuracyMCMicro", "PrecisionMCMicro" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    model_trainer.makeModelFCMultiTask(model, n_input_nodes, n_output_nodes_recon, n_output_nodes_class, encoding_size, true, false, false, false); // normalization type 1
    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

// Main
int main(int argc, char** argv)
{
  // Set the data directories
  //const std::string data_dir = "C:/Users/dmccloskey/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  const std::string data_dir = "C:/Users/domccl/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  //const std::string data_dir = "/home/user/Data/";

  // Make the filenames
  const std::string biochem_rxns_filename = data_dir + "iJO1366.csv";

  // ALEsKOs01
  //const std::string metabo_data_filename_train = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  //const std::string meta_data_filename_train = data_dir + "ALEsKOs01_MetaData_train.csv";
  //const std::string metabo_data_filename_test = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  //const std::string meta_data_filename_test = data_dir + "ALEsKOs01_MetaData_test.csv";

  // IndustrialStrains0103
  const std::string metabo_data_filename_train = data_dir + "IndustrialStrains0103_Metabolomics_train.csv";
  const std::string meta_data_filename_train = data_dir + "IndustrialStrains0103_MetaData_train.csv";
  const std::string metabo_data_filename_test = data_dir + "IndustrialStrains0103_Metabolomics_test.csv";
  const std::string meta_data_filename_test = data_dir + "IndustrialStrains0103_MetaData_test.csv";

  main_multiTask(biochem_rxns_filename, metabo_data_filename_train, meta_data_filename_train,
    metabo_data_filename_test, meta_data_filename_test, true, false, true);
  return 0;
}