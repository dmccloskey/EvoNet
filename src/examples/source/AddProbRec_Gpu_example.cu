/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerExperimentalGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicatorExperimental.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/ml/Model.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/simulator/AddProbSimulator.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>
#include <SmartPeak/io/Parameters.h>

#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

/*
@brief Add problem genetic + deep learning algorithm examples

Experiments:
1. addProb and single model training with the solution model initialized to the correct weights
2. addProb and single model training with solution model and weight dev from the correct weights
3. addProb and population training with the solution model as the population seed
4. addProb and population training with the minimal model as the population seed
5. addProb and single model training with the LSTM architecture
6. addProb and population training with the LSTM model as the population seed

Hyper parameters:
1. Adam solver with a learning rate of 0.001
2. Batch size of 32
3. 5000 epochs (single model training); 50 epochs (population training)
4. 25 epochs testing
*/

template<typename TensorT>
class DataSimulatorExt : public AddProbSimulator<TensorT>
{
public:
  void simulateData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);
    const int n_epochs = input_data.dimension(3);

    //// generate a new sequence 
    //// TODO: ensure that the this->sequence_length_ >= memory_size!
    //Eigen::Tensor<TensorT, 1> random_sequence(this->sequence_length_);
    //Eigen::Tensor<TensorT, 1> mask_sequence(this->sequence_length_);
    //float result = this->AddProb(random_sequence, mask_sequence, this->n_mask_);

    // Generate the input and output data for training [BUG FREE]
    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int epochs_iter = 0; epochs_iter < n_epochs; ++epochs_iter) {

        // generate a new sequence 
        // TODO: ensure that the this->sequence_length_ >= memory_size!
        Eigen::Tensor<float, 1> random_sequence(this->sequence_length_);
        Eigen::Tensor<float, 1> mask_sequence(this->sequence_length_);
        float result = this->AddProb(random_sequence, mask_sequence, this->n_mask_);
        Eigen::Tensor<float, 1> cumulative(this->sequence_length_);
        cumulative.setZero();

        float result_cumulative = 0.0;

        for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
          // determine the cumulative vector
          result_cumulative += random_sequence(memory_iter) * mask_sequence(memory_iter);
          cumulative(memory_iter) = result_cumulative;
          //std::cout << "result cumulative: " << result_cumulative << std::endl; // [TESTS: convert to a test!]
        }
        //for (int memory_iter = memory_size - 1; memory_iter >= 0; --memory_iter) {
        for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
          // assign the input sequences
          input_data(batch_iter, memory_iter, 0, epochs_iter) = random_sequence(memory_size - memory_iter - 1); // random sequence
          input_data(batch_iter, memory_iter, 1, epochs_iter) = mask_sequence(memory_size - memory_iter - 1); // mask sequence

          // assign the output
          output_data(batch_iter, memory_iter, 0, epochs_iter) = cumulative(memory_size - memory_iter - 1);
          //if (memory_iter == 0)
          //	output_data(batch_iter, memory_iter, 0, epochs_iter) = result;
          //else
          //	output_data(batch_iter, memory_iter, 0, epochs_iter) = 0.0;
        }
      }
    }
    //std::cout << "Input data: " << input_data << std::endl; // [TESTS: convert to a test!]
    //std::cout << "Output data: " << output_data << std::endl; // [TESTS: convert to a test!]

    time_steps.setConstant(1.0f);
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)	override { simulateData(input_data, output_data, time_steps); }
  void simulateValidationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)	override { simulateData(input_data, output_data, time_steps); }
  void simulateEvaluationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 3>& time_steps)override {};
  void simulateData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& output_data, Eigen::Tensor<TensorT, 3>& metric_data, Eigen::Tensor<TensorT, 2>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);

    // Generate the input and output data for training [BUG FREE]
    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {

      // generate a new sequence 
      // TODO: ensure that the this->sequence_length_ >= memory_size!
      Eigen::Tensor<float, 1> random_sequence(this->sequence_length_);
      Eigen::Tensor<float, 1> mask_sequence(this->sequence_length_);
      float result = this->AddProb(random_sequence, mask_sequence, this->n_mask_);
      Eigen::Tensor<float, 1> cumulative(this->sequence_length_);
      cumulative.setZero();

      float result_cumulative = 0.0;

      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        // determine the cumulative vector
        result_cumulative += random_sequence(memory_iter) * mask_sequence(memory_iter);
        cumulative(memory_iter) = result_cumulative;
      }
      //for (int memory_iter = memory_size - 1; memory_iter >= 0; --memory_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        // assign the input sequences
        input_data(batch_iter, memory_iter, 0) = random_sequence(memory_size - memory_iter - 1); // random sequence
        input_data(batch_iter, memory_iter, 1) = mask_sequence(memory_size - memory_iter - 1); // mask sequence

        // assign the output
        output_data(batch_iter, memory_iter, 0) = cumulative(memory_size - memory_iter - 1);
        metric_data(batch_iter, memory_iter, 0) = cumulative(memory_size - memory_iter - 1);
      }
    }

    time_steps.setConstant(1.0f);
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& output_data, Eigen::Tensor<TensorT, 3>& metric_data, Eigen::Tensor<TensorT, 2>& time_steps)override { simulateData(input_data, output_data, metric_data, time_steps); }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& output_data, Eigen::Tensor<TensorT, 3>& metric_data, Eigen::Tensor<TensorT, 2>& time_steps)override { simulateData(input_data, output_data, metric_data, time_steps); }
  void simulateEvaluationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& metric_data, Eigen::Tensor<TensorT, 2>& time_steps)override { simulateData(input_data, metric_data, Eigen::Tensor<TensorT, 3>(), time_steps); }
};

// Extended classes
template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  /*
  @brief Minimal network
  */
  void makeModelMinimal(Model<TensorT>& model)
  {
    Node<TensorT> i_rand, i_mask, h, o, output,
      h_bias, o_bias;
    Link Link_i_rand_to_h, Link_i_mask_to_h,
      Link_h_to_o, Link_o_to_output,
      Link_h_bias_to_h, Link_o_bias_to_o;
    Weight<TensorT> Weight_i_rand_to_h, Weight_i_mask_to_h,
      Weight_h_to_o, Weight_o_to_output,
      Weight_h_bias_to_h, Weight_o_bias_to_o;
    // Nodes
    i_rand = Node<TensorT>("Input_000000000000", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    i_mask = Node<TensorT>("Input_000000000001", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    h = Node<TensorT>("h", NodeType::hidden, NodeStatus::deactivated, std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()), std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    o = Node<TensorT>("o", NodeType::unmodifiable, NodeStatus::deactivated, std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()), std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    output = Node<TensorT>("Output_000000000000", NodeType::output, NodeStatus::deactivated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    h_bias = Node<TensorT>("h_bias", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    o_bias = Node<TensorT>("o_bias", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    output.setLayerName("Output");
    // weights  
    std::shared_ptr<WeightInitOp<TensorT>> weight_init = std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>(1.0));
    std::shared_ptr<SolverOp<TensorT>> solver = std::make_shared<SGDOp<TensorT>>(SGDOp<TensorT>(1e-3, 0.9, 10));
    Weight_i_rand_to_h = Weight<TensorT>("Weight_i_rand_to_h", weight_init, solver);
    Weight_i_mask_to_h = Weight<TensorT>("Weight_i_mask_to_h", weight_init, solver);
    Weight_h_to_o = Weight<TensorT>("Weight_h_to_o", weight_init, solver);
    Weight_o_to_output = Weight<TensorT>("Weight_o_to_output", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0)), std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()));
    Weight_h_bias_to_h = Weight<TensorT>("Weight_h_bias_to_h", weight_init, solver);
    Weight_o_bias_to_o = Weight<TensorT>("Weight_o_bias_to_o", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(0.0)), solver);
    weight_init.reset();
    solver.reset();
    // links
    Link_i_rand_to_h = Link("Link_i_rand_to_h", "Input_000000000000", "h", "Weight_i_rand_to_h");
    Link_i_mask_to_h = Link("Link_i_mask_to_h", "Input_000000000001", "h", "Weight_i_mask_to_h");
    Link_h_to_o = Link("Link_h_to_o", "h", "o", "Weight_h_to_o");
    Link_o_to_output = Link("Link_o_to_output", "o", "Output_000000000000", "Weight_o_to_output");
    Link_h_bias_to_h = Link("Link_h_bias_to_h", "h_bias", "h", "Weight_h_bias_to_h");
    Link_o_bias_to_o = Link("Link_o_bias_to_o", "o_bias", "o", "Weight_o_bias_to_o");
    // add nodes, links, and weights to the model
    model.setName("MemoryCell");
    model.addNodes({ i_rand, i_mask, h, o, output//, h_bias, o_bias 
      });
    model.addWeights({ Weight_i_rand_to_h, Weight_i_mask_to_h, Weight_h_to_o, Weight_o_to_output//,	Weight_h_bias_to_h, Weight_o_bias_to_o 
      });
    model.addLinks({ Link_i_rand_to_h, Link_i_mask_to_h, Link_h_to_o, Link_o_to_output//,	Link_h_bias_to_h, Link_o_bias_to_o 
      });
    model.setInputAndOutputNodes();
  }
  /*
  @brief Minimal network required to solve the addition problem
  */
  void makeModelSolution(Model<TensorT>& model, bool init_weight_soln = true)
  {
    Node<TensorT> i_rand, i_mask, h, m, mr, o, output,
      h_bias, m_bias, o_bias;
    Link Link_i_rand_to_h, Link_i_mask_to_h,
      Link_h_to_m,
      Link_m_to_o, Link_m_to_mr, Link_mr_to_m,
      Link_h_bias_to_h,
      Link_m_bias_to_m, Link_o_bias_to_o,
      Link_o_to_output;
    Weight<TensorT> Weight_i_rand_to_h, Weight_i_mask_to_h,
      Weight_h_to_m,
      Weight_m_to_o, Weight_m_to_mr, Weight_mr_to_m,
      Weight_h_bias_to_h,
      Weight_m_bias_to_m, Weight_o_bias_to_o,
      Weight_o_to_output;
    // Nodes
    i_rand = Node<TensorT>("Input_000000000000", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    i_mask = Node<TensorT>("Input_000000000001", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    h = Node<TensorT>("h", NodeType::hidden, NodeStatus::initialized, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<ProdOp<TensorT>>(ProdOp<TensorT>()), std::make_shared<ProdErrorOp<TensorT>>(ProdErrorOp<TensorT>()), std::make_shared<ProdWeightGradOp<TensorT>>(ProdWeightGradOp<TensorT>()));
    m = Node<TensorT>("m", NodeType::hidden, NodeStatus::initialized, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    mr = Node<TensorT>("mr", NodeType::hidden, NodeStatus::initialized, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    o = Node<TensorT>("o", NodeType::unmodifiable, NodeStatus::deactivated, std::shared_ptr<ActivationOp<TensorT>>(new ReLUOp<TensorT>()), std::shared_ptr<ActivationOp<TensorT>>(new ReLUGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    output = Node<TensorT>("Output_000000000000", NodeType::output, NodeStatus::deactivated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    h_bias = Node<TensorT>("h_bias", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    m_bias = Node<TensorT>("m_bias", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    o_bias = Node<TensorT>("o_bias", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()), std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()), std::make_shared<SumOp<TensorT>>(SumOp<TensorT>()), std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>()), std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>()));
    output.setLayerName("Output");
    // weights  
    std::shared_ptr<WeightInitOp<TensorT>> weight_init;
    auto solver = std::make_shared<SGDOp<TensorT>>(SGDOp<TensorT>(1e-3, 0.9, 10));
    if (init_weight_soln) {
      weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0)); //solution
    }
    else {
      weight_init = std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>(1.0)); // will not converge
      //weight_init = std::make_shared<RangeWeightInitOp<TensorT>>(RangeWeightInitOp<TensorT>(0.5, 1.5)); // will converge with ADAM learning_rate < 1e-6
    }
    Weight_i_rand_to_h = Weight<TensorT>("Weight_i_rand_to_h", weight_init, solver);
    Weight_i_mask_to_h = Weight<TensorT>("Weight_i_mask_to_h", weight_init, solver);
    Weight_h_to_m = Weight<TensorT>("Weight_h_to_m", weight_init, solver);
    Weight_m_to_mr = Weight<TensorT>("Weight_m_to_mr", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0)), std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()));
    Weight_mr_to_m = Weight<TensorT>("Weight_mr_to_m", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0)), std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()));
    Weight_m_to_o = Weight<TensorT>("Weight_m_to_o", weight_init, solver);
    Weight_o_to_output = Weight<TensorT>("Weight_o_to_output", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0)), std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()));
    Weight_h_bias_to_h = Weight<TensorT>("Weight_h_bias_to_h", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(0.0)), solver);
    Weight_m_bias_to_m = Weight<TensorT>("Weight_m_bias_to_m", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(0.0)), solver);
    Weight_o_bias_to_o = Weight<TensorT>("Weight_o_bias_to_o", std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(0.0)), solver);
    weight_init.reset();
    solver.reset();
    // links
    Link_i_rand_to_h = Link("Link_i_rand_to_h", "Input_000000000000", "h", "Weight_i_rand_to_h");
    Link_i_mask_to_h = Link("Link_i_mask_to_h", "Input_000000000001", "h", "Weight_i_mask_to_h");
    Link_h_to_m = Link("Link_h_to_m", "h", "m", "Weight_h_to_m");
    Link_m_to_o = Link("Link_m_to_o", "m", "o", "Weight_m_to_o");
    Link_o_to_output = Link("Link_o_to_output", "o", "Output_000000000000", "Weight_o_to_output");
    Link_m_to_mr = Link("Link_m_to_mr", "m", "mr", "Weight_m_to_mr");
    Link_mr_to_m = Link("Link_mr_to_m", "mr", "m", "Weight_mr_to_m");
    //Link_m_to_m = Link("Link_m_to_m", "m", "m", "Weight_m_to_m");
    Link_h_bias_to_h = Link("Link_h_bias_to_h", "h_bias", "h", "Weight_h_bias_to_h");
    Link_m_bias_to_m = Link("Link_m_bias_to_m", "m_bias", "m", "Weight_m_bias_to_m");
    Link_o_bias_to_o = Link("Link_o_bias_to_o", "o_bias", "o", "Weight_o_bias_to_o");
    // add nodes, links, and weights to the model
    model.setName("MemoryCell");
    model.addNodes({ i_rand, i_mask, h, m, mr, o, output//,
      //h_bias, m_bias, o_bias 
      });
    model.addWeights({ Weight_i_rand_to_h, Weight_i_mask_to_h,
      Weight_h_to_m,
      Weight_m_to_o, Weight_m_to_mr, Weight_mr_to_m, Weight_o_to_output//,
      //Weight_h_bias_to_h,
      //Weight_m_bias_to_m, 
      //Weight_o_bias_to_o 
      });
    model.addLinks({ Link_i_rand_to_h, Link_i_mask_to_h,
      Link_h_to_m,
      Link_m_to_o, Link_m_to_mr, Link_mr_to_m, Link_o_to_output//,
      //Link_h_bias_to_h,
      //Link_m_bias_to_m, 
      //Link_o_bias_to_o 
      });
    model.setInputAndOutputNodes();
  }
  /*
  @brief LSTM implementation

  References:
    Hochreiter et al. "Long Short-Term Memory". Neural Computation 9, 1735�1780 (1997)
    Chung et al. "Empirical Evaluation of Gated Recurrent Neural Networks on Sequence Modeling". 2014. arXiv:1412.3555v1

  GRU implementation

  References:
    Cho et al. "Learning Phrase Representations using RNN Encoder�Decoder for Statistical Machine Translation". 2014. arXiv:1406.1078v3
    Chung et al. "Empirical Evaluation of Gated Recurrent Neural Networks on Sequence Modeling". 2014. arXiv:1412.3555v1
  */
  void makeModelLSTM(Model<TensorT>& model, const int& n_inputs, int n_blocks = 2, int n_cells = 2, bool add_forget_gate = false, bool specify_layers = false)
  {
    model.setId(0);
    model.setName("LSTM");

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, specify_layers);

    // Define the activation 
    std::shared_ptr<ActivationOp<TensorT>> activation = std::make_shared<TanHOp<TensorT>>(TanHOp<TensorT>());
    std::shared_ptr<ActivationOp<TensorT>> activation_grad = std::make_shared<TanHGradOp<TensorT>>(TanHGradOp<TensorT>());
    std::shared_ptr<ActivationOp<TensorT>> activation_output = std::make_shared<LeakyReLUOp<TensorT>>(LeakyReLUOp<TensorT>());
    std::shared_ptr<ActivationOp<TensorT>> activation_output_grad = std::make_shared<LeakyReLUGradOp<TensorT>>(LeakyReLUGradOp<TensorT>());

    // Define the node integration
    auto integration_op = std::make_shared<SumOp<TensorT>>(SumOp<TensorT>());
    auto integration_error_op = std::make_shared<SumErrorOp<TensorT>>(SumErrorOp<TensorT>());
    auto integration_weight_grad_op = std::make_shared<SumWeightGradOp<TensorT>>(SumWeightGradOp<TensorT>());

    // Define the solver
    auto solver_op = std::make_shared<SGDOp<TensorT>>(SGDOp<TensorT>(1e-3, 0.9, 10));

    // Add the LSTM layer(s)
    std::vector<std::string> node_names = model_builder.addLSTM(model, "LSTM-01", "LSTM-01", node_names_input, n_blocks, n_cells,
      activation, activation_grad, integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>((TensorT)(node_names_input.size() + n_blocks) / 2, 1)),
      solver_op,
      0.0f, 0.0f, true, add_forget_gate, 1, specify_layers);

    // Add a final output layer
    node_names = model_builder.addFullyConnected(model, "FC-Out", "FC-Out", node_names, 1,
      activation_output, activation_output_grad, integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<RandWeightInitOp<TensorT>>(RandWeightInitOp<TensorT>(node_names.size(), 2)),
      solver_op, 0.0f, 0.0f, false, true);
    node_names = model_builder.addSinglyConnected(model, "Output", "Output", node_names, 1,
      std::make_shared<LinearOp<TensorT>>(LinearOp<TensorT>()),
      std::make_shared<LinearGradOp<TensorT>>(LinearGradOp<TensorT>()),
      integration_op, integration_error_op, integration_weight_grad_op,
      std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1)),
      std::make_shared<DummySolverOp<TensorT>>(DummySolverOp<TensorT>()), 0.0f, 0.0f, false, true);

    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }
  void trainingModelLogger(const int& n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const std::vector<std::string>& input_nodes, const TensorT& model_error) override
  { // Left blank intentionally to prevent writing of files during training
  }
  void validationModelLogger(const int& n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const std::vector<std::string>& input_nodes, const TensorT& model_error) override
  { // Left blank intentionally to prevent writing of files during validation
  }
  void trainingModelLogger(const int& n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const std::vector<std::string>& input_nodes, const TensorT& model_error_train, const TensorT& model_error_test,
    const Eigen::Tensor<TensorT, 1>& model_metrics_train, const Eigen::Tensor<TensorT, 1>& model_metrics_test) override {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedEpoch(false);
    model_logger.setLogNodeInputsEpoch(false);
    model_logger.setLogNodeOutputsEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedEpoch(true);
      model_logger.setLogNodeInputsEpoch(true);
      model_logger.setLogNodeOutputsEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 1000 == 0) { // FIXME
      model_logger.setLogExpectedEpoch(true);
      model_logger.setLogNodeInputsEpoch(true);
      model_logger.setLogNodeOutputsEpoch(true);
      model_interpreter.getModelResults(model, true, false, false, true);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_train_headers = { "Train_Error" };
    std::vector<std::string> log_test_headers = { "Test_Error" };
    std::vector<TensorT> log_train_values = { model_error_train };
    std::vector<TensorT> log_test_values = { model_error_test };
    int metric_iter = 0;
    for (const std::string& metric_name : this->getMetricNamesLinearized()) {
      log_train_headers.push_back(metric_name);
      log_test_headers.push_back(metric_name);
      log_train_values.push_back(model_metrics_train(metric_iter));
      log_test_values.push_back(model_metrics_test(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_train_headers, log_test_headers, log_train_values, log_test_values, output_nodes, expected_values, {}, output_nodes, {}, input_nodes, {});
  }
  void evaluationModelLogger(const int& n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const std::vector<std::string>& input_nodes, const Eigen::Tensor<TensorT, 1>& model_metrics) override
  {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedEpoch(false);
    model_logger.setLogNodeInputsEpoch(false);
    model_logger.setLogNodeOutputsEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedEpoch(true);
      model_logger.setLogNodeInputsEpoch(true);
      model_logger.setLogNodeOutputsEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 1 == 0) { // FIXME
      model_logger.setLogExpectedEpoch(true);
      model_logger.setLogNodeInputsEpoch(true);
      model_logger.setLogNodeOutputsEpoch(true);
      model_interpreter.getModelResults(model, true, false, false, true);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_headers;
    std::vector<TensorT> log_values;
    int metric_iter = 0;
    for (const std::string& metric_name : this->getMetricNamesLinearized()) {
      log_headers.push_back(metric_name);
      log_values.push_back(model_metrics(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_headers, {}, log_values, {}, output_nodes, expected_values, {}, output_nodes, {}, input_nodes, {});
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors)override {
    //if (n_epochs % 100 == 0 && n_epochs > 100) {
    //  // anneal the learning rate by half on each plateau
    //  TensorT lr_new = this->reduceLROnPlateau(model_errors, 0.5, 100, 10, 0.1);
    //  if (lr_new < 1.0) {
    //    model_interpreter.updateSolverParams(0, lr_new);
    //    std::cout << "The learning rate has been annealed by a factor of " << lr_new << std::endl;
    //  }
    //}
    if (n_epochs % 1000 == 0 && n_epochs != 0) {
      // save the model every 1000 epochs
      model_interpreter.getModelResults(model, false, true, false, false);
      ModelFile<TensorT> data;
      data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
      ModelInterpreterFileGpu<TensorT> interpreter_data;
      interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    }
  }
};

template<typename TensorT>
class ModelReplicatorExt : public ModelReplicatorExperimental<TensorT>
{};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerExperimentalGpu<TensorT>
{};

template<class ...ParameterTypes>
void main_(const ParameterTypes& ...args) {
  auto parameters = std::make_tuple(args...);

  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  setPopulationTrainerParameters(population_trainer, args...);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the input/output nodes
  std::vector<std::string> input_nodes = { "Input_000000000000", "Input_000000000001" };
  std::vector<std::string> output_nodes = { "Output_000000000000" };

  // define the data simulator
  DataSimulatorExt<float> data_simulator;
  data_simulator.n_mask_ = std::get<EvoNetParameters::Examples::NMask>(parameters).get();
  data_simulator.sequence_length_ = std::get<EvoNetParameters::Examples::SequenceLength>(parameters).get();

  // define the model interpreters
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  setModelInterpreterParameters(model_interpreters, args...);

  // define the model trainer
  ModelTrainerExt<float> model_trainer;
  setModelTrainerParameters(model_trainer, args...);

  std::vector<LossFunctionHelper<float>> loss_function_helpers;
  LossFunctionHelper<float> loss_function_helper2;
  loss_function_helper2.output_nodes_ = output_nodes;
  loss_function_helper2.loss_functions_ = { std::make_shared<MSELossOp<float>>(MSELossOp<float>(1e-24, 1.0)) };
  loss_function_helper2.loss_function_grads_ = { std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>(1e-24, 1.0)) };
  loss_function_helpers.push_back(loss_function_helper2);
  model_trainer.setLossFunctionHelpers(loss_function_helpers);

  std::vector<MetricFunctionHelper<float>> metric_function_helpers;
  MetricFunctionHelper<float> metric_function_helper1;
  metric_function_helper1.output_nodes_ = output_nodes;
  metric_function_helper1.metric_functions_ = { std::make_shared<EuclideanDistOp<float>>(EuclideanDistOp<float>("Mean")), std::make_shared<EuclideanDistOp<float>>(EuclideanDistOp<float>("Var")) };
  metric_function_helper1.metric_names_ = { "EuclideanDist-Mean", "EuclideanDist-Var" };
  metric_function_helpers.push_back(metric_function_helper1);
  model_trainer.setMetricFunctionHelpers(metric_function_helpers);

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false);

  // define the model replicator for growth mode
  ModelReplicatorExt<float> model_replicator;
  setModelReplicatorParameters(model_replicator, args...);

  // define the initial population
  Model<float> model;
  if (std::get<EvoNetParameters::Main::MakeModel>(parameters).get()) {
    std::cout << "Making the model..." << std::endl;
    if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "Minimal") model_trainer.makeModelMinimal(model);
    else if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "Solution") model_trainer.makeModelSolution(model, false);
    else if (std::get<EvoNetParameters::Examples::ModelType>(parameters).get() == "LSTM") model_trainer.makeModelLSTM(model, input_nodes.size(), 1, 1, false);
    model.setId(0);
  }
  else {
    ModelFile<float> model_file;
    ModelInterpreterFileGpu<float> model_interpreter_file;
    loadModelFromParameters(model, model_interpreters.at(0), model_file, model_interpreter_file, args...);
  }
  model.setName(std::get<EvoNetParameters::General::DataDir>(parameters).get() + std::get<EvoNetParameters::Main::ModelName>(parameters).get()); //So that all output will be written to a specific directory

  // Run the training, evaluation, or evolution
  runTrainEvalEvoFromParameters<float>(model, model_interpreters, model_trainer, population_trainer, model_replicator, data_simulator, model_logger, population_logger, input_nodes, args...);
}

// Main
int main(int argc, char** argv)
{
  // Parse the user commands
  int id_int = -1;
  std::string parameters_filename = "";
  parseCommandLineArguments(argc, argv, id_int, parameters_filename);

  // Set the parameter names and defaults
  EvoNetParameters::General::ID id("id", -1);
  EvoNetParameters::General::DataDir data_dir("data_dir", std::string(""));
  EvoNetParameters::Main::DeviceId device_id("device_id", 0);
  EvoNetParameters::Main::ModelName model_name("model_name", "");
  EvoNetParameters::Main::MakeModel make_model("make_model", true);
  EvoNetParameters::Main::LoadModelCsv load_model_csv("load_model_csv", false);
  EvoNetParameters::Main::LoadModelBinary load_model_binary("load_model_binary", false);
  EvoNetParameters::Main::TrainModel train_model("train_model", true);
  EvoNetParameters::Main::EvolveModel evolve_model("evolve_model", false);
  EvoNetParameters::Main::EvaluateModel evaluate_model("evaluate_model", false);
  EvoNetParameters::Main::EvaluateModels evaluate_models("evaluate_models", false);
  EvoNetParameters::Examples::NMask n_mask("n_mask", 2);
  EvoNetParameters::Examples::SequenceLength sequence_length("sequence_length", 25);
  EvoNetParameters::Examples::ModelType model_type("model_type", "Solution");
  EvoNetParameters::Examples::SimulationType simulation_type("simulation_type", "");
  EvoNetParameters::Examples::BiochemicalRxnsFilename biochemical_rxns_filename("biochemical_rxns_filename", "iJO1366.csv");
  EvoNetParameters::PopulationTrainer::PopulationName population_name("population_name", "");
  EvoNetParameters::PopulationTrainer::NGenerations n_generations("n_generations", 1);
  EvoNetParameters::PopulationTrainer::NInterpreters n_interpreters("n_interpreters", 1);
  EvoNetParameters::PopulationTrainer::PruneModelNum prune_model_num("prune_model_num", 10);
  EvoNetParameters::PopulationTrainer::RemoveIsolatedNodes remove_isolated_nodes("remove_isolated_nodes", true);
  EvoNetParameters::PopulationTrainer::CheckCompleteModelInputToOutput check_complete_model_input_to_output("check_complete_model_input_to_output", true);
  EvoNetParameters::PopulationTrainer::PopulationSize population_size("population_size", 128);
  EvoNetParameters::PopulationTrainer::NTop n_top("n_top", 8);
  EvoNetParameters::PopulationTrainer::NRandom n_random("n_random", 8);
  EvoNetParameters::PopulationTrainer::NReplicatesPerModel n_replicates_per_model("n_replicates_per_model", 1);
  EvoNetParameters::PopulationTrainer::ResetModelCopyWeights reset_model_copy_weights("reset_model_copy_weights", true);
  EvoNetParameters::PopulationTrainer::ResetModelTemplateWeights reset_model_template_weights("reset_model_template_weights", true);
  EvoNetParameters::PopulationTrainer::Logging population_logging("population_logging", true);
  EvoNetParameters::PopulationTrainer::SetPopulationSizeFixed set_population_size_fixed("set_population_size_fixed", false);
  EvoNetParameters::PopulationTrainer::SetPopulationSizeDoubling set_population_size_doubling("set_population_size_doubling", true);
  EvoNetParameters::PopulationTrainer::SetTrainingStepsByModelSize set_training_steps_by_model_size("set_training_steps_by_model_size", false);
  EvoNetParameters::ModelTrainer::BatchSize batch_size("batch_size", 32);
  EvoNetParameters::ModelTrainer::MemorySize memory_size("memory_size", 64);
  EvoNetParameters::ModelTrainer::NEpochsTraining n_epochs_training("n_epochs_training", 1000);
  EvoNetParameters::ModelTrainer::NEpochsValidation n_epochs_validation("n_epochs_validation", 25);
  EvoNetParameters::ModelTrainer::NEpochsEvaluation n_epochs_evaluation("n_epochs_evaluation", 10);
  EvoNetParameters::ModelTrainer::NTBTTSteps n_tbtt_steps("n_tbtt_steps", 64);
  EvoNetParameters::ModelTrainer::NTETTSteps n_tett_steps("n_tett_steps", 64);
  EvoNetParameters::ModelTrainer::Verbosity verbosity("verbosity", 1);
  EvoNetParameters::ModelTrainer::LoggingTraining logging_training("logging_training", true);
  EvoNetParameters::ModelTrainer::LoggingValidation logging_validation("logging_validation", false);
  EvoNetParameters::ModelTrainer::LoggingEvaluation logging_evaluation("logging_evaluation", true);
  EvoNetParameters::ModelTrainer::FindCycles find_cycles("find_cycles", true);
  EvoNetParameters::ModelTrainer::FastInterpreter fast_interpreter("fast_interpreter", true);
  EvoNetParameters::ModelTrainer::PreserveOoO preserve_ooo("preserve_ooo", true);
  EvoNetParameters::ModelTrainer::InterpretModel interpret_model("interpret_model", true);
  EvoNetParameters::ModelTrainer::ResetModel reset_model("reset_model", false);
  EvoNetParameters::ModelTrainer::ResetInterpreter reset_interpreter("reset_interpreter", true);
  EvoNetParameters::ModelReplicator::NNodeDownAdditionsLB n_node_down_additions_lb("n_node_down_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeRightAdditionsLB n_node_right_additions_lb("n_node_right_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDownCopiesLB n_node_down_copies_lb("n_node_down_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeRightCopiesLB n_node_right_copies_lb("n_node_right_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkAdditionsLB n_link_additons_lb("n_link_additons_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkCopiesLB n_link_copies_lb("n_link_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDeletionsLB n_node_deletions_lb("n_node_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NLinkDeletionsLB n_link_deletions_lb("n_link_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeActivationChangesLB n_node_activation_changes_lb("n_node_activation_changes_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeIntegrationChangesLB n_node_integration_changes_lb("n_node_integration_changes_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleAdditionsLB n_module_additions_lb("n_module_additions_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleCopiesLB n_module_copies_lb("n_module_copies_lb", 0);
  EvoNetParameters::ModelReplicator::NModuleDeletionsLB n_module_deletions_lb("n_module_deletions_lb", 0);
  EvoNetParameters::ModelReplicator::NNodeDownAdditionsUB n_node_down_additions_ub("n_node_down_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeRightAdditionsUB n_node_right_additions_ub("n_node_right_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeDownCopiesUB n_node_down_copies_ub("n_node_down_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeRightCopiesUB n_node_right_copies_ub("n_node_right_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkAdditionsUB n_link_additons_ub("n_link_additons_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkCopiesUB n_link_copies_ub("n_link_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeDeletionsUB n_node_deletions_ub("n_node_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::NLinkDeletionsUB n_link_deletions_ub("n_link_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeActivationChangesUB n_node_activation_changes_ub("n_node_activation_changes_ub", 0);
  EvoNetParameters::ModelReplicator::NNodeIntegrationChangesUB n_node_integration_changes_ub("n_node_integration_changes_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleAdditionsUB n_module_additions_ub("n_module_additions_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleCopiesUB n_module_copies_ub("n_module_copies_ub", 0);
  EvoNetParameters::ModelReplicator::NModuleDeletionsUB n_module_deletions_ub("n_module_deletions_ub", 0);
  EvoNetParameters::ModelReplicator::SetModificationRateFixed set_modification_rate_fixed("set_modification_rate_fixed", false);
  EvoNetParameters::ModelReplicator::SetModificationRateByPrevError set_modification_rate_by_prev_error("set_modification_rate_by_prev_error", false);
  auto parameters = std::make_tuple(id, data_dir,
    device_id, model_name, make_model, load_model_csv, load_model_binary, train_model, evolve_model, evaluate_model, evaluate_models,
    n_mask, sequence_length, model_type, simulation_type, biochemical_rxns_filename,
    population_name, n_generations, n_interpreters, prune_model_num, remove_isolated_nodes, check_complete_model_input_to_output, population_size, n_top, n_random, n_replicates_per_model, reset_model_copy_weights, reset_model_template_weights, population_logging, set_population_size_fixed, set_population_size_doubling, set_training_steps_by_model_size,
    batch_size, memory_size, n_epochs_training, n_epochs_validation, n_epochs_evaluation, n_tbtt_steps, n_tett_steps, verbosity, logging_training, logging_validation, logging_evaluation, find_cycles, fast_interpreter, preserve_ooo, interpret_model, reset_model, reset_interpreter,
    n_node_down_additions_lb, n_node_right_additions_lb, n_node_down_copies_lb, n_node_right_copies_lb, n_link_additons_lb, n_link_copies_lb, n_node_deletions_lb, n_link_deletions_lb, n_node_activation_changes_lb, n_node_integration_changes_lb, n_module_additions_lb, n_module_copies_lb, n_module_deletions_lb, n_node_down_additions_ub, n_node_right_additions_ub, n_node_down_copies_ub, n_node_right_copies_ub, n_link_additons_ub, n_link_copies_ub, n_node_deletions_ub, n_link_deletions_ub, n_node_activation_changes_ub, n_node_integration_changes_ub, n_module_additions_ub, n_module_copies_ub, n_module_deletions_ub, set_modification_rate_fixed, set_modification_rate_by_prev_error);

  // Read in the parameters
  LoadParametersFromCsv loadParametersFromCsv(id_int, parameters_filename);
  parameters = SmartPeak::apply([&loadParametersFromCsv](auto&& ...args) { return loadParametersFromCsv(args...); }, parameters);

  // Run the application
  SmartPeak::apply([](auto&& ...args) { main_(args ...); }, parameters);
  return 0;
}