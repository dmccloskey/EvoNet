#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicator.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>
#include <SmartPeak/simulator/BiochemicalReaction.h>
#include <unsupported/Eigen/CXX11/Tensor>

using namespace SmartPeak;

// Other extended classes
template<typename TensorT>
class ModelReplicatorExt : public ModelReplicator<TensorT>
{};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerGpu<TensorT>
{};

template<typename TensorT>
class MetDataSimReconstruction : public DataSimulator<TensorT>
{
public:
  void simulateData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_output_nodes = output_data.dimension(2);
    const int n_epochs = input_data.dimension(3);

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
        for (int epochs_iter = 0; epochs_iter < n_epochs; ++epochs_iter) {

          // pick a random sample group name
          //std::string sample_group_name = selectRandomElement(sample_group_names_);
          std::string sample_group_name = this->model_training_.sample_group_names_[0];

          for (int nodes_iter = 0; nodes_iter < n_input_nodes; ++nodes_iter) {
            const TensorT mar = this->model_training_.calculateMAR(
              this->model_training_.metabolomicsData_.at(sample_group_name),
              this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_[nodes_iter]));
            input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = mar;
            output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = mar;
          }
        }
      }
    }
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps) {
    simulateData(input_data, output_data, time_steps);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 4>& input_data, Eigen::Tensor<TensorT, 4>& output_data, Eigen::Tensor<TensorT, 3>& time_steps) {
    simulateData(input_data, output_data, time_steps);
  }
  void simulateDataReconMARs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);
    int n_input_pixels;
    if (train)
      n_input_pixels = this->model_training_.reaction_ids_.size();
    else
      n_input_pixels = this->model_validation_.reaction_ids_.size();

    assert(n_loss_output_nodes == n_input_pixels + 2 * n_encodings_);
    assert(n_metric_output_nodes % n_input_pixels == 0);
    assert(n_input_nodes == n_input_pixels + n_encodings_);

    std::random_device rd{};
    std::mt19937 gen{ rd() };
    std::normal_distribution<> d{ 0.0f, 1.0f };

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        for (int nodes_iter = 0; nodes_iter < n_input_pixels + 2 * n_encodings_; ++nodes_iter) {
          if (nodes_iter < n_input_pixels) {
            TensorT value;
            if (train)
              value = this->model_training_.calculateMAR(
                this->model_training_.metabolomicsData_.at(sample_group_name),
                this->model_training_.biochemicalReactions_.at(this->model_training_.reaction_ids_.at(nodes_iter)));
            else
              value = this->model_validation_.calculateMAR(
                this->model_validation_.metabolomicsData_.at(sample_group_name),
                this->model_validation_.biochemicalReactions_.at(this->model_validation_.reaction_ids_.at(nodes_iter)));
            input_data(batch_iter, memory_iter, nodes_iter) = value;
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0;
            metric_output_data(batch_iter, memory_iter, nodes_iter) = 0;
          }
          else if (nodes_iter >= n_input_pixels && nodes_iter < n_input_pixels + n_encodings_) {
            TensorT random_value;
            if (train)
              random_value = d(gen);
            else
              random_value = 0;
            input_data(batch_iter, memory_iter, nodes_iter) = random_value; // sample from a normal distribution
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // Dummy data for KL divergence mu
          }
          else {
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // Dummy data for KL divergence logvar
          }
        }
      }
    }
  }
  void simulateDataReconSampleConcs(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps, const bool& train)
  {
    // infer data dimensions based on the input tensors
    const int batch_size = input_data.dimension(0);
    const int memory_size = input_data.dimension(1);
    const int n_input_nodes = input_data.dimension(2);
    const int n_loss_output_nodes = loss_output_data.dimension(2);
    const int n_metric_output_nodes = metric_output_data.dimension(2);
    int n_input_pixels;
    if (train)
      n_input_pixels = this->model_training_.component_group_names_.size();
    else
      n_input_pixels = this->model_validation_.component_group_names_.size();

    assert(n_loss_output_nodes == n_input_pixels + 2 * n_encodings_);
    assert(n_metric_output_nodes % n_input_pixels == 0);
    assert(n_input_nodes == n_input_pixels + n_encodings_);

    std::random_device rd{};
    std::mt19937 gen{ rd() };
    std::normal_distribution<> d{ 0.0f, 1.0f };

    for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
      for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {

        // pick a random sample group name
        std::string sample_group_name;
        if (train)
          sample_group_name = selectRandomElement(this->model_training_.sample_group_names_);
        else
          sample_group_name = selectRandomElement(this->model_validation_.sample_group_names_);

        for (int nodes_iter = 0; nodes_iter < n_input_pixels + 2 * n_encodings_; ++nodes_iter) {
          if (nodes_iter < n_input_pixels) {
            TensorT value;
            if (train)
              value = this->model_training_.getRandomConcentration(
                this->model_training_.metabolomicsData_.at(sample_group_name),
                this->model_training_.component_group_names_.at(nodes_iter));
            else
              value = this->model_validation_.getRandomConcentration(
                this->model_validation_.metabolomicsData_.at(sample_group_name),
                this->model_validation_.component_group_names_.at(nodes_iter));
            input_data(batch_iter, memory_iter, nodes_iter) = value;
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0;
            metric_output_data(batch_iter, memory_iter, nodes_iter) = 0;
          }
          else if (nodes_iter >= n_input_pixels && nodes_iter < n_input_pixels + n_encodings_) {
            TensorT random_value;
            if (train)
              random_value = d(gen);
            else
              random_value = 0;
            input_data(batch_iter, memory_iter, nodes_iter) = random_value; // sample from a normal distribution
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // Dummy data for KL divergence mu
          }
          else {
            loss_output_data(batch_iter, memory_iter, nodes_iter) = 0; // Dummy data for KL divergence logvar
          }
        }
      }
    }
  }
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataReconMARs(input_data, loss_output_data, metric_output_data, time_steps, true);
    else simulateDataReconSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, true);
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) {
    if (simulate_MARs_) simulateDataReconMARs(input_data, loss_output_data, metric_output_data, time_steps, false);
    else simulateDataReconSampleConcs(input_data, loss_output_data, metric_output_data, time_steps, false);
  }

  BiochemicalReactionModel<TensorT> model_training_;
  BiochemicalReactionModel<TensorT> model_validation_;
  int n_encodings_;
  bool sample_concs_ = false;
  bool simulate_MARs_ = true;
};

template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  /*
  @brief Fully connected variational reconstruction model
  */
  void makeModelFCVAE(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const int& n_encodings, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, bool add_norm = true) {
    model.setId(0);
    model.setName("VAE");

    const int n_en_hidden_0 = 64;
    const int n_en_hidden_1 = 64;
    const int n_en_hidden_2 = 0;
    const int n_de_hidden_0 = 64;
    const int n_de_hidden_1 = 64;
    const int n_de_hidden_2 = 0;


    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the encoding layers
    std::vector<std::string> node_names = node_names_input;
    if (n_en_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN0", "EN0", node_names, n_en_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN0-Norm", "EN0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN0-Norm-gain", "EN0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN1", "EN1", node_names, n_en_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN1-Norm", "EN1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN1-Norm-gain", "EN1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN2", "EN2", node_names, n_en_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN2-Norm", "EN2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN2-Norm-gain", "EN2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the mu and log var layers
    std::vector<std::string> node_names_mu = model_builder.addFullyConnected(model, "Mu", "Mu", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    std::vector<std::string> node_names_logvar = model_builder.addFullyConnected(model, "LogVar", "LogVar", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names_mu)
      model.nodes_.at(node_name)->setType(NodeType::output);
    for (const std::string& node_name : node_names_logvar)
      model.nodes_.at(node_name)->setType(NodeType::output);

    // Add the Variational Encoding layer
    node_names = model_builder.addGaussianEncoding(model, "Encoding", "Encoding", node_names_mu, node_names_logvar, true);

    // Add the decoding layers
    if (n_de_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE0", "DE0", node_names, n_de_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE0-Norm", "DE0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE0-Norm-gain", "DE0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE1", "DE1", node_names, n_de_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE1-Norm", "DE1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE1-Norm-gain", "DE1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE2", "DE2", node_names, n_de_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE2-Norm", "DE2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE2-Norm-gain", "DE2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the final output layer
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    // Subtract out the pre-processed input data to test against all 0's
    model_builder.addSinglyConnected(model, "Output", node_names_input, node_names,
      std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(-1)),
      std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0f, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief Add data preprocessing steps
  */
  void addDataPreproccessingSteps(Model<TensorT>& model, std::vector<std::string>& node_names, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input) {
    ModelBuilder<TensorT> model_builder;
    // Data pre-processing steps
    if (log_transform_input) {
      node_names = model_builder.addSinglyConnected(model, "LogScaleInput", "LogScaleInput", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LogOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LogGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, true, true);
    }
    if (linear_scale_input) {
      node_names = model_builder.addLinearScale(model, "LinearScaleInput", "LinearScaleInput", node_names, 0, 1, true);
    }
    if (standardize_input) {
      node_names = model_builder.addNormalization(model, "StandardizeInput", "StandardizeInput", node_names, true);
    }
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors) {
    // Check point the model every 1000 epochs
    if (n_epochs % 1000 == 0 && n_epochs != 0) {
      model_interpreter.getModelResults(model, false, true, false);
      ModelFile<TensorT> data;
      data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
      ModelInterpreterFileGpu<TensorT> interpreter_data;
      interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    }
  }
  void trainingModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const TensorT & model_error_train, const TensorT & model_error_test,
    const Eigen::Tensor<TensorT, 1> & model_metrics_train, const Eigen::Tensor<TensorT, 1> & model_metrics_test)
  {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedPredictedEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 1000 == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_interpreter.getModelResults(model, true, false, false);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_train_headers = { "Train_Error" };
    std::vector<std::string> log_test_headers = { "Test_Error" };
    std::vector<TensorT> log_train_values = { model_error_train };
    std::vector<TensorT> log_test_values = { model_error_test };
    int metric_iter = 0;
    for (const std::string& metric_name : this->metric_names_) {
      log_train_headers.push_back(metric_name);
      log_test_headers.push_back(metric_name);
      log_train_values.push_back(model_metrics_train(metric_iter));
      log_test_values.push_back(model_metrics_test(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_train_headers, log_test_headers, log_train_values, log_test_values, output_nodes, expected_values);
  }
};

/// Script to run the reconstruction network
void main_reconstruction(const std::string& biochem_rxns_filename,
  const std::string& metabo_data_filename_train, const std::string& meta_data_filename_train,
  const std::string& metabo_data_filename_test, const std::string& meta_data_filename_test,
  bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimReconstruction<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data

  // Training data
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_train);
  reaction_model.readMetaData(meta_data_filename_train);
  reaction_model.findComponentGroupNames();
  if (simulate_MARs) {
    reaction_model.findMARs();
    reaction_model.findMARs(true, false);
    reaction_model.findMARs(false, true);
    reaction_model.removeRedundantMARs();
  }
  reaction_model.findLabels("subject");
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename_test);
  reaction_model.readMetaData(meta_data_filename_test);
  reaction_model.findComponentGroupNames();
  if (simulate_MARs) {
    reaction_model.findMARs();
    reaction_model.findMARs(true, false);
    reaction_model.findMARs(false, true);
    reaction_model.removeRedundantMARs();
  }
  reaction_model.findLabels("subject");
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // Define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes = n_input_nodes;
  const int encoding_size = 8;
  metabolomics_data.n_encodings_ = encoding_size;
  std::vector<std::string> input_nodes;
  std::vector<std::string> output_nodes;

  // Make the input nodes
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the encoding nodes and add them to the input
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Encoding_%012d-Sampler", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the reconstruction nodes
  for (int i = 0; i < n_output_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Output_%012d", i);
    std::string name(name_char);
    output_nodes.push_back(name);
  }

  // Make the mu nodes
  std::vector<std::string> encoding_nodes_mu;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Mu_%012d", i);
    std::string name(name_char);
    encoding_nodes_mu.push_back(name);
  }

  // Make the encoding nodes
  std::vector<std::string> encoding_nodes_logvar;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "LogVar_%012d", i);
    std::string name(name_char);
    encoding_nodes_logvar.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(64);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(100000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionOp<float>>(new BCEWithLogitsOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceMuOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceLogVarOp<float>(1e-6, 1.0)) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new MSEGradOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionGradOp<float>>(new BCEWithLogitsGradOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceMuGradOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceLogVarGradOp<float>(1e-6, 1.0)) });
  model_trainer.setLossOutputNodes({ output_nodes, encoding_nodes_mu, encoding_nodes_logvar });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new MAEOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes });
  model_trainer.setMetricNames({ "MAE" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    model_trainer.makeModelFCVAE(model, n_input_nodes, n_output_nodes, encoding_size, true, false, false, false); // normalization type 1
    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

// Main
int main(int argc, char** argv)
{
  // Set the data directories
  //const std::string data_dir = "C:/Users/dmccloskey/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  const std::string data_dir = "C:/Users/domccl/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  //const std::string data_dir = "/home/user/Data/";

  // Make the filenames
  const std::string biochem_rxns_filename = data_dir + "iJO1366.csv";

  // ALEsKOs01
  const std::string metabo_data_filename_train = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  const std::string meta_data_filename_train = data_dir + "ALEsKOs01_MetaData_train.csv";
  const std::string metabo_data_filename_test = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  const std::string meta_data_filename_test = data_dir + "ALEsKOs01_MetaData_test.csv";

  //// IndustrialStrains0103
  //const std::string metabo_data_filename_train = data_dir + "IndustrialStrains0103_Metabolomics_train.csv";
  //const std::string meta_data_filename_train = data_dir + "IndustrialStrains0103_MetaData_train.csv";
  //const std::string metabo_data_filename_test = data_dir + "IndustrialStrains0103_Metabolomics_test.csv";
  //const std::string meta_data_filename_test = data_dir + "IndustrialStrains0103_MetaData_test.csv";

  main_reconstruction(biochem_rxns_filename, metabo_data_filename_train, meta_data_filename_train,
    metabo_data_filename_test, meta_data_filename_test, true, false, true);
  return 0;
}