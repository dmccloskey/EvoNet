#include "hip/hip_runtime.h"
/**TODO:  Add copyright*/

#include <SmartPeak/ml/PopulationTrainerGpu.h>
#include <SmartPeak/ml/ModelTrainerGpu.h>
#include <SmartPeak/ml/ModelReplicator.h>
#include <SmartPeak/ml/ModelBuilder.h>
#include <SmartPeak/io/PopulationTrainerFile.h>
#include <SmartPeak/io/ModelInterpreterFileGpu.h>

#include "Metabolomics_example.h"

using namespace SmartPeak;

// Other extended classes
template<typename TensorT>
class ModelReplicatorExt : public ModelReplicator<TensorT>
{};

template<typename TensorT>
class PopulationTrainerExt : public PopulationTrainerGpu<TensorT>
{};

template<typename TensorT>
class ModelTrainerExt : public ModelTrainerGpu<TensorT>
{
public:
  Model<TensorT> makeModel() { return Model<TensorT>(); }
  /*
  @brief Fully connected classifier
  */
  void makeModelFCClass(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, bool add_norm = true) {
    model.setId(0);
    model.setName("Classifier");

    const int n_hidden_0 = 64;
    const int n_hidden_1 = 64;
    const int n_hidden_2 = 0;

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names, linear_scale_input, log_transform_input, standardize_input);

    // Add the hidden layers
    if (n_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC0", "FC0", node_names, n_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC0-Norm", "FC0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC0-Norm-gain", "FC0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC1-Norm", "FC1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC1-Norm-gain", "FC1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "FC2", "FC2", node_names, n_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "FC2-Norm", "FC2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "FC2-Norm-gain", "FC2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, true, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief CovNet classifier
  */
  void makeModelCovNetClass(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input,
    int n_hidden_0 = 64, int n_depth_1 = 32, int n_depth_2 = 2, int n_fc = 16, bool add_norm = false, bool specify_layers = false) {
    model.setId(0);
    model.setName("CovNet");

    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, specify_layers);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the hidden layers
    if (n_hidden_0 > 0) {
      node_names_input = model_builder.addFullyConnected(model, "FC0", "FC0", node_names_input, n_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_input.size() + n_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names_input = model_builder.addNormalization(model, "FC0-Norm", "FC0-Norm", node_names_input, true);
        node_names_input = model_builder.addSinglyConnected(model, "FC0-Norm-gain", "FC0-Norm-gain", node_names_input, node_names_input.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
          0.0, 0.0, true, true);
      }
    }

    // Add the first convolution -> max pool -> LeakyReLU layers
    std::vector<std::vector<std::string>> node_names_l0;
    for (size_t d = 0; d < n_depth_1; ++d) {
      std::vector<std::string> node_names;
      std::string conv_name = "Conv0-" + std::to_string(d);
      node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_input,
        sqrt(node_names_input.size()), sqrt(node_names_input.size()), 0, 0,
        2, 2, 1, 0, 0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(5, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
      if (add_norm) {
        std::string norm_name = "Norm0-" + std::to_string(d);
        node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names, true);
        std::string gain_name = "Gain0-" + std::to_string(d);
        node_names = model_builder.addSinglyConnected(model, gain_name, gain_name, node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
          0.0, 0.0, true, true);
      }
      //std::string pool_name = "Pool0-" + std::to_string(d);
      //node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
      //  sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
      //  2, 2, 2, 0, 0,
      //  std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      //  std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      //  std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
      //  std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
      //  std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
      //  std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
      //  std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
      node_names_l0.push_back(node_names);
    }

    // Add the second convolution -> max pool -> LeakyReLU layers
    std::vector<std::vector<std::string>> node_names_l1;
    int l_cnt = 0;
    for (const std::vector<std::string> &node_names_l : node_names_l0) {
      for (size_t d = 0; d < n_depth_2; ++d) {
        std::vector<std::string> node_names;
        std::string conv_name = "Conv1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        node_names = model_builder.addConvolution(model, conv_name, conv_name, node_names_l,
          sqrt(node_names_l.size()), sqrt(node_names_l.size()), 0, 0,
          2, 2, 1, 0, 0,
          std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
          std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(5, 1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
        if (add_norm) {
          std::string norm_name = "Norm1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
          node_names = model_builder.addNormalization(model, norm_name, norm_name, node_names, true);
          std::string gain_name = "Gain1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
          node_names = model_builder.addSinglyConnected(model, gain_name, gain_name, node_names, node_names.size(),
            std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
            std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
            std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
            std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
            std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
            std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
            std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
            0.0, 0.0, true, true);
        }
        //std::string pool_name = "Pool1-" + std::to_string(l_cnt) + "-" + std::to_string(d);
        //node_names = model_builder.addConvolution(model, pool_name, pool_name, node_names,
        //  sqrt(node_names.size()), sqrt(node_names.size()), 1, 1,
        //  2, 2, 2, 0, 0,
        //  std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //  std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        //  std::shared_ptr<IntegrationOp<TensorT>>(new MaxOp<float>()),
        //  std::shared_ptr<IntegrationErrorOp<TensorT>>(new MaxErrorOp<TensorT>()),
        //  std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new MaxWeightGradOp<TensorT>()),
        //  std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1.0)),
        //  std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, false, specify_layers);
        node_names_l1.push_back(node_names);
      }
      ++l_cnt;
    }

    // Linearize the node names
    std::vector<std::string> node_names;
    if (node_names_l1.size()) {
      for (const std::vector<std::string> &node_names_l : node_names_l1) {
        for (const std::string &node_name : node_names_l) {
          node_names.push_back(node_name);
        }
      }
    }
    else {
      for (const std::vector<std::string> &node_names_l : node_names_l0) {
        for (const std::string &node_name : node_names_l) {
          node_names.push_back(node_name);
        }
      }
    }

    // Add the FC layers
    node_names = model_builder.addFullyConnected(model, "FC1", "FC1", node_names, n_fc,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size() + n_fc, 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, specify_layers);
    if (add_norm) {
      node_names = model_builder.addNormalization(model, "FC1-Norm", "FC1-Norm", node_names, true);
      node_names = model_builder.addSinglyConnected(model, "FC1-Norm-gain", "FC1-Norm-gain", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)),
        0.0, 0.0, true, true);
    }
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>(node_names.size(), 2)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(5e-4, 0.9, 0.999, 1e-8, 10)), 0.0f, 0.0f, false, true);

    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief Fully connected variational reconstruction model
  */
  void makeModelFCVAE(Model<TensorT>& model, const int& n_inputs, const int& n_outputs, const int& n_encodings, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, bool add_norm = true) {
    model.setId(0);
    model.setName("VAE");

    const int n_en_hidden_0 = 64;
    const int n_en_hidden_1 = 64;
    const int n_en_hidden_2 = 0;
    const int n_de_hidden_0 = 64;
    const int n_de_hidden_1 = 64;
    const int n_de_hidden_2 = 0;


    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the encoding layers
    std::vector<std::string> node_names = node_names_input;
    if (n_en_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN0", "EN0", node_names, n_en_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN0-Norm", "EN0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN0-Norm-gain", "EN0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN1", "EN1", node_names, n_en_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN1-Norm", "EN1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN1-Norm-gain", "EN1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN2", "EN2", node_names, n_en_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN2-Norm", "EN2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN2-Norm-gain", "EN2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the mu and log var layers
    std::vector<std::string> node_names_mu = model_builder.addFullyConnected(model, "Mu", "Mu", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    std::vector<std::string> node_names_logvar = model_builder.addFullyConnected(model, "LogVar", "LogVar", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names_mu)
      model.nodes_.at(node_name)->setType(NodeType::output);
    for (const std::string& node_name : node_names_logvar)
      model.nodes_.at(node_name)->setType(NodeType::output);

    // Add the Variational Encoding layer
    node_names = model_builder.addGaussianEncoding(model, "Encoding", "Encoding", node_names_mu, node_names_logvar, true);

    // Add the decoding layers
    if (n_de_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE0", "DE0", node_names, n_de_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE0-Norm", "DE0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE0-Norm-gain", "DE0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE1", "DE1", node_names, n_de_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE1-Norm", "DE1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE1-Norm-gain", "DE1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE2", "DE2", node_names, n_de_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE2-Norm", "DE2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE2-Norm-gain", "DE2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the final output layer
    node_names = model_builder.addFullyConnected(model, "Output", "Output", node_names, n_outputs,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    // Subtract out the pre-processed input data to test against all 0's
    model_builder.addSinglyConnected(model, "Output", node_names_input, node_names,
      std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(-1)),
      std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0f, true);

    // Specify the output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);
    model.setInputAndOutputNodes();
  }

  /*
  @brief Fully connected multitask model for variational reconstruction and classification
  */
  void makeModelFCMultiTask(Model<TensorT>& model, const int& n_inputs, const int& n_outputs_recon, const int& n_outputs_class, const int& n_encodings, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input, bool add_norm = true) {
    model.setId(0);
    model.setName("MultiTask");
    const int n_en_hidden_0 = 64;
    const int n_en_hidden_1 = 64;
    const int n_en_hidden_2 = 0;
    const int n_de_hidden_0 = 64;
    const int n_de_hidden_1 = 64;
    const int n_de_hidden_2 = 0;
    ModelBuilder<TensorT> model_builder;

    // Add the inputs
    std::vector<std::string> node_names_input = model_builder.addInputNodes(model, "Input", "Input", n_inputs, true);

    // Data pre-processing steps
    this->addDataPreproccessingSteps(model, node_names_input, linear_scale_input, log_transform_input, standardize_input);

    // Add the encoding layers
    std::vector<std::string> node_names = node_names_input;
    if (n_en_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN0", "EN0", node_names, n_en_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN0-Norm", "EN0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN0-Norm-gain", "EN0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN1", "EN1", node_names, n_en_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN1-Norm", "EN1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN1-Norm-gain", "EN1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_en_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "EN2", "EN2", node_names, n_en_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_en_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "EN2-Norm", "EN2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "EN2-Norm-gain", "EN2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the mu and log var layers
    std::vector<std::string> node_names_mu = model_builder.addFullyConnected(model, "Mu", "Mu", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    std::vector<std::string> node_names_logvar = model_builder.addFullyConnected(model, "LogVar", "LogVar", node_names, n_encodings,
      std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_encodings) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the Encoding Mu and LogVar output nodes
    for (const std::string& node_name : node_names_mu)
      model.nodes_.at(node_name)->setType(NodeType::output);
    for (const std::string& node_name : node_names_logvar)
      model.nodes_.at(node_name)->setType(NodeType::output);

    // Add the Variational Encoding layer
    std::vector<std::string> node_names_encoding = model_builder.addGaussianEncoding(model, "Encoding", "Encoding", node_names_mu, node_names_logvar, true);

    // Add the decoding layers
    if (n_de_hidden_0 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE0", "DE0", node_names_encoding, n_de_hidden_0,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_encoding.size() + n_de_hidden_0) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE0-Norm", "DE0-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE0-Norm-gain", "DE0-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_1 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE1", "DE1", node_names, n_de_hidden_1,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_1) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE1-Norm", "DE1-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE1-Norm-gain", "DE1-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }
    if (n_de_hidden_2 > 0) {
      node_names = model_builder.addFullyConnected(model, "DE2", "DE2", node_names, n_de_hidden_2,
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearOp<TensorT>()),
        //std::shared_ptr<ActivationOp<TensorT>>(new LinearGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_de_hidden_2) / 2, 1)),
        std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
      if (add_norm) {
        node_names = model_builder.addNormalization(model, "DE2-Norm", "DE2-Norm", node_names, true);
        node_names = model_builder.addSinglyConnected(model, "DE2-Norm-gain", "DE2-Norm-gain", node_names, node_names.size(),
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()), // Nonlinearity occures after the normalization
          std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
          std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
          std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
          std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
          std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
          std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)),
          0.0, 0.0, true, true);
      }
    }

    // Add the final reconstruction output layer
    node_names = model_builder.addFullyConnected(model, "Output-Recon", "Output-Recon", node_names, n_outputs_recon,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names.size() + n_outputs_recon) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);
    // Subtract out the pre-processed input data to test against all 0's
    model_builder.addSinglyConnected(model, "Output-Recon", node_names_input, node_names,
      std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(-1)),
      std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0f, true);

    // Specify the reconstruction output node types
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);

    // Add the classification output layer
    node_names = model_builder.addFullyConnected(model, "Output-Class", "Output-Class", node_names_mu, n_outputs_class,
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUOp<TensorT>()),
      std::shared_ptr<ActivationOp<TensorT>>(new LeakyReLUGradOp<TensorT>()),
      std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
      std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
      std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
      std::shared_ptr<WeightInitOp<TensorT>>(new RandWeightInitOp<TensorT>((int)(node_names_mu.size() + n_outputs_class) / 2, 1)),
      std::shared_ptr<SolverOp<TensorT>>(new AdamOp<TensorT>(1e-4, 0.9, 0.999, 1e-8)), 0.0f, 0.0f, false, true);

    // Specify the classificaiton output node types manually
    for (const std::string& node_name : node_names)
      model.getNodesMap().at(node_name)->setType(NodeType::output);

    // Set the input and output nodes
    model.setInputAndOutputNodes();

    //// Check that the model is set-up correctly
    //if (!model.checkCompleteInputToOutput())
    //  std::cout << "There is a problem with the model!" << std::endl;
  }

  /*
  @brief Add data preprocessing steps
  */
  void addDataPreproccessingSteps(Model<TensorT>& model, std::vector<std::string>& node_names, const bool& linear_scale_input, const bool& log_transform_input, const bool& standardize_input) {
    ModelBuilder<TensorT> model_builder;
    // Data pre-processing steps
    if (log_transform_input) {
      node_names = model_builder.addSinglyConnected(model, "LogScaleInput", "LogScaleInput", node_names, node_names.size(),
        std::shared_ptr<ActivationOp<TensorT>>(new LogOp<TensorT>()),
        std::shared_ptr<ActivationOp<TensorT>>(new LogGradOp<TensorT>()),
        std::shared_ptr<IntegrationOp<TensorT>>(new SumOp<TensorT>()),
        std::shared_ptr<IntegrationErrorOp<TensorT>>(new SumErrorOp<TensorT>()),
        std::shared_ptr<IntegrationWeightGradOp<TensorT>>(new SumWeightGradOp<TensorT>()),
        std::shared_ptr<WeightInitOp<TensorT>>(new ConstWeightInitOp<TensorT>(1)),
        std::shared_ptr<SolverOp<TensorT>>(new DummySolverOp<TensorT>()), 0.0, 0.0, true, true);
    }
    if (linear_scale_input) {
      node_names = model_builder.addLinearScale(model, "LinearScaleInput", "LinearScaleInput", node_names, 0, 1, true);
    }
    if (standardize_input) {
      node_names = model_builder.addNormalization(model, "StandardizeInput", "StandardizeInput", node_names, true);
    }
  }
  void adaptiveTrainerScheduler(
    const int& n_generations,
    const int& n_epochs,
    Model<TensorT>& model,
    ModelInterpreterGpu<TensorT>& model_interpreter,
    const std::vector<float>& model_errors) {
    // Check point the model every 1000 epochs
    //if (n_epochs % 1000 == 0 && n_epochs != 0) {
    //  model_interpreter.getModelResults(model, false, true, false);
    //  ModelFile<TensorT> data;
    //  data.storeModelBinary(model.getName() + "_" + std::to_string(n_epochs) + "_model.binary", model);
    //  ModelInterpreterFileGpu<TensorT> interpreter_data;
    //  interpreter_data.storeModelInterpreterBinary(model.getName() + "_" + std::to_string(n_epochs) + "_interpreter.binary", model_interpreter);
    //}
  }
  void trainingModelLogger(const int & n_epochs, Model<TensorT>& model, ModelInterpreterGpu<TensorT>& model_interpreter, ModelLogger<TensorT>& model_logger,
    const Eigen::Tensor<TensorT, 3>& expected_values, const std::vector<std::string>& output_nodes, const TensorT & model_error_train, const TensorT & model_error_test,
    const Eigen::Tensor<TensorT, 1> & model_metrics_train, const Eigen::Tensor<TensorT, 1> & model_metrics_test)
  {
    // Set the defaults
    model_logger.setLogTimeEpoch(true);
    model_logger.setLogTrainValMetricEpoch(true);
    model_logger.setLogExpectedPredictedEpoch(false);

    // initialize all logs
    if (n_epochs == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_logger.initLogs(model);
    }

    // Per n epoch logging
    if (n_epochs % 10 == 0) {
      model_logger.setLogExpectedPredictedEpoch(true);
      model_interpreter.getModelResults(model, true, false, false);
    }

    // Create the metric headers and data arrays
    std::vector<std::string> log_train_headers = { "Train_Error" };
    std::vector<std::string> log_test_headers = { "Test_Error" };
    std::vector<TensorT> log_train_values = { model_error_train };
    std::vector<TensorT> log_test_values = { model_error_test };
    int metric_iter = 0;
    for (const std::string& metric_name : this->metric_names_) {
      log_train_headers.push_back(metric_name);
      log_test_headers.push_back(metric_name);
      log_train_values.push_back(model_metrics_train(metric_iter));
      log_test_values.push_back(model_metrics_test(metric_iter));
      ++metric_iter;
    }
    model_logger.writeLogs(model, n_epochs, log_train_headers, log_test_headers, log_train_values, log_test_values, output_nodes, expected_values);
  }
};

/*
@brief Example using intracellular E. coli metabolomics data
  taken from re-grown glycerol stock solutions on Glucose M9 at mid-exponential phase
  from adaptive laboratory evolution (ALE) experiments following gene knockout (KO)
*/

/// Script to run the time-course Summary
void main_statistics_timecourseSummary(const std::string& data_dir,
  bool run_timeCourse_Ref = false, bool run_timeCourse_Gnd = false, bool run_timeCourse_SdhCB = false, bool run_timeCourse_Pgi = false, bool run_timeCourse_PtsHIcrr = false,
  bool run_timeCourse_TpiA = false)
{
  // define the data simulator
  BiochemicalReactionModel<float> metabolomics_data;

  std::string
    timeCourse_Ref_filename, timeCourse_Gnd_filename, timeCourse_SdhCB_filename, timeCourse_Pgi_filename, timeCourse_PtsHIcrr_filename,
    timeCourse_TpiA_filename,
    timeCourseSampleSummary_Ref_filename, timeCourseSampleSummary_Gnd_filename, timeCourseSampleSummary_SdhCB_filename, timeCourseSampleSummary_Pgi_filename, timeCourseSampleSummary_PtsHIcrr_filename,
    timeCourseSampleSummary_TpiA_filename,
    timeCourseFeatureSummary_Ref_filename, timeCourseFeatureSummary_Gnd_filename, timeCourseFeatureSummary_SdhCB_filename, timeCourseFeatureSummary_Pgi_filename, timeCourseFeatureSummary_PtsHIcrr_filename,
    timeCourseFeatureSummary_TpiA_filename;

  // filenames
  timeCourse_Ref_filename = data_dir + "EColi_timeCourse_Ref.csv";
  timeCourse_Gnd_filename = data_dir + "EColi_timeCourse_Gnd.csv";
  timeCourse_SdhCB_filename = data_dir + "EColi_timeCourse_SdhCB.csv";
  timeCourse_Pgi_filename = data_dir + "EColi_timeCourse_Pgi.csv";
  timeCourse_PtsHIcrr_filename = data_dir + "EColi_timeCourse_PtsHIcrr.csv";
  timeCourse_TpiA_filename = data_dir + "EColi_timeCourse_TpiA.csv";
  timeCourseSampleSummary_Ref_filename = data_dir + "EColi_timeCourseSampleSummary_Ref.csv";
  timeCourseSampleSummary_Gnd_filename = data_dir + "EColi_timeCourseSampleSummary_Gnd.csv";
  timeCourseSampleSummary_SdhCB_filename = data_dir + "EColi_timeCourseSampleSummary_SdhCB.csv";
  timeCourseSampleSummary_Pgi_filename = data_dir + "EColi_timeCourseSampleSummary_Pgi.csv";
  timeCourseSampleSummary_PtsHIcrr_filename = data_dir + "EColi_timeCourseSampleSummary_PtsHIcrr.csv";
  timeCourseSampleSummary_TpiA_filename = data_dir + "EColi_timeCourseSampleSummary_TpiA.csv";
  timeCourseFeatureSummary_Ref_filename = data_dir + "EColi_timeCourseFeatureSummary_Ref.csv";
  timeCourseFeatureSummary_Gnd_filename = data_dir + "EColi_timeCourseFeatureSummary_Gnd.csv";
  timeCourseFeatureSummary_SdhCB_filename = data_dir + "EColi_timeCourseFeatureSummary_SdhCB.csv";
  timeCourseFeatureSummary_Pgi_filename = data_dir + "EColi_timeCourseFeatureSummary_Pgi.csv";
  timeCourseFeatureSummary_PtsHIcrr_filename = data_dir + "EColi_timeCourseFeatureSummary_PtsHIcrr.csv";
  timeCourseFeatureSummary_TpiA_filename = data_dir + "EColi_timeCourseFeatureSummary_TpiA.csv";

  if (run_timeCourse_Ref) {
    // Read in the data
    PWData timeCourseRef;
    ReadPWData(timeCourse_Ref_filename, timeCourseRef);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCourseRef, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_Ref_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_Ref_filename, pw_feature_summaries);
  }

  if (run_timeCourse_Gnd) {
    // Read in the data
    PWData timeCourseGnd;
    ReadPWData(timeCourse_Gnd_filename, timeCourseGnd);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCourseGnd, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_Gnd_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_Gnd_filename, pw_feature_summaries);
  }

  if (run_timeCourse_SdhCB) {
    // Read in the data
    PWData timeCourseSdhCB;
    ReadPWData(timeCourse_SdhCB_filename, timeCourseSdhCB);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCourseSdhCB, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_SdhCB_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_SdhCB_filename, pw_feature_summaries);
  }

  if (run_timeCourse_Pgi) {
    // Read in the data
    PWData timeCoursePgi;
    ReadPWData(timeCourse_Pgi_filename, timeCoursePgi);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCoursePgi, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_Pgi_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_Pgi_filename, pw_feature_summaries);
  }

  if (run_timeCourse_PtsHIcrr) {
    // Read in the data
    PWData timeCoursePtsHIcrr;
    ReadPWData(timeCourse_PtsHIcrr_filename, timeCoursePtsHIcrr);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCoursePtsHIcrr, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_PtsHIcrr_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_PtsHIcrr_filename, pw_feature_summaries);
  }

  if (run_timeCourse_TpiA) {
    // Read in the data
    PWData timeCourseTpiA;
    ReadPWData(timeCourse_TpiA_filename, timeCourseTpiA);

    // Summarize the data
    PWSampleSummaries pw_sample_summaries;
    PWFeatureSummaries pw_feature_summaries;
    PWTotalSummary pw_total_summary;
    PWSummary(timeCourseTpiA, pw_sample_summaries, pw_feature_summaries, pw_total_summary);

    // Export to file
    WritePWSampleSummaries(timeCourseSampleSummary_TpiA_filename, pw_sample_summaries);
    WritePWFeatureSummaries(timeCourseFeatureSummary_TpiA_filename, pw_feature_summaries);
  }
}

/// Script to run the time-course MARs analysis
void main_statistics_timecourse(const std::string& data_dir,
  bool run_timeCourse_Ref = false, bool run_timeCourse_Gnd = false, bool run_timeCourse_SdhCB = false, bool run_timeCourse_Pgi = false, bool run_timeCourse_PtsHIcrr = false,
  bool run_timeCourse_TpiA = false)
{
  // define the data simulator
  BiochemicalReactionModel<float> metabolomics_data;

  std::string biochem_rxns_filename, metabo_data_filename, meta_data_filename,
    timeCourse_Ref_filename, timeCourse_Gnd_filename, timeCourse_SdhCB_filename, timeCourse_Pgi_filename, timeCourse_PtsHIcrr_filename,
    timeCourse_TpiA_filename;
  std::vector<std::string> pre_samples,
    timeCourse_Ref_samples, timeCourse_Gnd_samples, timeCourse_SdhCB_samples, timeCourse_Pgi_samples, timeCourse_PtsHIcrr_samples,
    timeCourse_TpiA_samples;
  // filenames
  biochem_rxns_filename = data_dir + "iJO1366.csv";
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData.csv";
  timeCourse_Ref_filename = data_dir + "EColi_timeCourse_Ref.csv";
  timeCourse_Gnd_filename = data_dir + "EColi_timeCourse_Gnd.csv";
  timeCourse_SdhCB_filename = data_dir + "EColi_timeCourse_SdhCB.csv";
  timeCourse_Pgi_filename = data_dir + "EColi_timeCourse_Pgi.csv";
  timeCourse_PtsHIcrr_filename = data_dir + "EColi_timeCourse_PtsHIcrr.csv";
  timeCourse_TpiA_filename = data_dir + "EColi_timeCourse_TpiA.csv";
  timeCourse_Ref_samples = { "Evo04", "Evo04Evo01EP", "Evo04Evo02EP" };
  timeCourse_Gnd_samples = { "Evo04", "Evo04gnd", "Evo04gndEvo01EP", "Evo04gndEvo02EP", "Evo04gndEvo03EP" };
  timeCourse_SdhCB_samples = { "Evo04", "Evo04sdhCB", "Evo04sdhCBEvo01EP", "Evo04sdhCBEvo02EP", "Evo04sdhCBEvo03EP", "Evo04sdhCBEvo03EP-2", "Evo04sdhCBEvo03EP-3", "Evo04sdhCBEvo03EP-4", "Evo04sdhCBEvo03EP-5", "Evo04sdhCBEvo03EP-6" };
  timeCourse_Pgi_samples = { "Evo04", "Evo04pgi", "Evo04pgiEvo01EP", "Evo04pgiEvo01J01", "Evo04pgiEvo01J02", "Evo04pgiEvo02EP", "Evo04pgiEvo02J01", "Evo04pgiEvo02J02", "Evo04pgiEvo02J03", "Evo04pgiEvo03EP", "Evo04pgiEvo03J01", "Evo04pgiEvo03J02", "Evo04pgiEvo03J03", "Evo04pgiEvo04EP", "Evo04pgiEvo04J01", "Evo04pgiEvo04J02", "Evo04pgiEvo04J03", "Evo04pgiEvo05EP", "Evo04pgiEvo05J01", "Evo04pgiEvo05J02", "Evo04pgiEvo05J03", "Evo04pgiEvo06EP", "Evo04pgiEvo06J01", "Evo04pgiEvo06J02", "Evo04pgiEvo06J03", "Evo04pgiEvo07EP", "Evo04pgiEvo07J01", "Evo04pgiEvo07J02", "Evo04pgiEvo07J03", "Evo04pgiEvo08EP", "Evo04pgiEvo08J01", "Evo04pgiEvo08J02", "Evo04pgiEvo08J03" };
  timeCourse_PtsHIcrr_samples = { "Evo04", "Evo04ptsHIcrr", "Evo04ptsHIcrrEvo01EP", "Evo04ptsHIcrrEvo01J01", "Evo04ptsHIcrrEvo01J03", "Evo04ptsHIcrrEvo02EP", "Evo04ptsHIcrrEvo02J01", "Evo04ptsHIcrrEvo02J03", "Evo04ptsHIcrrEvo03EP", "Evo04ptsHIcrrEvo03J01", "Evo04ptsHIcrrEvo03J03", "Evo04ptsHIcrrEvo03J04", "Evo04ptsHIcrrEvo04EP", "Evo04ptsHIcrrEvo04J01", "Evo04ptsHIcrrEvo04J03", "Evo04ptsHIcrrEvo04J04" };
  timeCourse_TpiA_samples = { "Evo04", "Evo04tpiA", "Evo04tpiAEvo01EP", "Evo04tpiAEvo01J01", "Evo04tpiAEvo01J03", "Evo04tpiAEvo02EP", "Evo04tpiAEvo02J01", "Evo04tpiAEvo02J03", "Evo04tpiAEvo03EP", "Evo04tpiAEvo03J01", "Evo04tpiAEvo03J03", "Evo04tpiAEvo04EP", "Evo04tpiAEvo04J01", "Evo04tpiAEvo04J03" };

  // read in the data
  metabolomics_data.readBiochemicalReactions(biochem_rxns_filename);
  metabolomics_data.readMetabolomicsData(metabo_data_filename);
  metabolomics_data.readMetaData(meta_data_filename);
  metabolomics_data.findComponentGroupNames();
  metabolomics_data.findMARs();
  metabolomics_data.findMARs(true, false);
  metabolomics_data.findMARs(false, true);
  metabolomics_data.findLabels();

  if (run_timeCourse_Ref) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCourseRef = PWComparison(metabolomics_data, timeCourse_Ref_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_Ref_filename, timeCourseRef);
  }

  if (run_timeCourse_Gnd) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCourseGnd = PWComparison(metabolomics_data, timeCourse_Gnd_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_Gnd_filename, timeCourseGnd);
  }

  if (run_timeCourse_SdhCB) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCourseSdhCB = PWComparison(metabolomics_data, timeCourse_SdhCB_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_SdhCB_filename, timeCourseSdhCB);
  }

  if (run_timeCourse_Pgi) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCoursePgi = PWComparison(metabolomics_data, timeCourse_Pgi_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_Pgi_filename, timeCoursePgi);
  }

  if (run_timeCourse_PtsHIcrr) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCoursePtsHIcrr = PWComparison(metabolomics_data, timeCourse_PtsHIcrr_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_PtsHIcrr_filename, timeCoursePtsHIcrr);
  }

  if (run_timeCourse_TpiA) {
    // Find significant pair-wise MARS between each sample (one vs one)
    PWData timeCourseTpiA = PWComparison(metabolomics_data, timeCourse_TpiA_samples, 10000, 0.05, 1.0);

    // Export to file
    WritePWData(timeCourse_TpiA_filename, timeCourseTpiA);
  }
}

/// Script to run the classification network
void main_classification(const std::string& data_dir, bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimClassification<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data
  std::string biochem_rxns_filename, metabo_data_filename, meta_data_filename;
  biochem_rxns_filename = data_dir + "iJO1366.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_train.csv";

  // Training data
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_test.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes = reaction_model.labels_.size();
  std::vector<std::string> input_nodes;
  std::vector<std::string> output_nodes;
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }
  for (int i = 0; i < n_output_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Output_%012d", i);
    std::string name(name_char);
    output_nodes.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(64);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(10000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new CrossEntropyWithLogitsOp<float>()),
    std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>()) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new CrossEntropyWithLogitsGradOp<float>()),
    std::shared_ptr<LossFunctionGradOp<float>>(new MSEGradOp<float>()) });
  model_trainer.setLossOutputNodes({
    output_nodes,
    output_nodes });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new AccuracyMCMicroOp<float>()), std::shared_ptr<MetricFunctionOp<float>>(new PrecisionMCMicroOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes, output_nodes });
  model_trainer.setMetricNames({ "AccuracyMCMicro", "PrecisionMCMicro" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, false, false, false, false); // normalization type 0
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, false, false, false); // normalization type 1
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, false, true, false); // normalization type 2
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, true, false, false); // normalization type 3
    //model_trainer.makeModelFCClass(model, n_input_nodes, n_output_nodes, true, true, true, false); // normalization type 4

    //model_trainer.makeModelCovNetClass(model, n_input_nodes, n_output_nodes, true, true, false, 64, 16, 0, 32, false, true); // normalization type 3

    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

/// Script to run the reconstruction network
void main_reconstruction(const std::string& data_dir, bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimReconstruction<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data
  std::string biochem_rxns_filename, metabo_data_filename, meta_data_filename;
  biochem_rxns_filename = data_dir + "iJO1366.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_train.csv";

  // Training data
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_test.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // Define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes = n_input_nodes;
  const int encoding_size = 2;
  metabolomics_data.n_encodings_ = encoding_size;
  std::vector<std::string> input_nodes;
  std::vector<std::string> output_nodes;

  // Make the input nodes
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the encoding nodes and add them to the input
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Encoding_%012d-Sampler", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the reconstruction nodes
  for (int i = 0; i < n_output_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Output_%012d", i);
    std::string name(name_char);
    output_nodes.push_back(name);
  }

  // Make the mu nodes
  std::vector<std::string> encoding_nodes_mu;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Mu_%012d", i);
    std::string name(name_char);
    encoding_nodes_mu.push_back(name);
  }

  // Make the encoding nodes
  std::vector<std::string> encoding_nodes_logvar;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "LogVar_%012d", i);
    std::string name(name_char);
    encoding_nodes_logvar.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(16);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(100000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionOp<float>>(new BCEWithLogitsOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceMuOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceLogVarOp<float>(1e-6, 0.1)) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new MSEGradOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionGradOp<float>>(new BCEWithLogitsGradOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceMuGradOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceLogVarGradOp<float>(1e-6, 0.1)) });
  model_trainer.setLossOutputNodes({ output_nodes, encoding_nodes_mu, encoding_nodes_logvar });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new MAEOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes });
  model_trainer.setMetricNames({ "MAE" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    model_trainer.makeModelFCVAE(model, n_input_nodes, n_output_nodes, encoding_size, true, true, false, false); // normalization type 3
    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

/// Script to run the reconstruction network
void main_multiTask(const std::string& data_dir, bool make_model = true, bool simulate_MARs = true, bool sample_concs = true)
{
  // define the population trainer parameters
  PopulationTrainerExt<float> population_trainer;
  population_trainer.setNGenerations(1);
  population_trainer.setNTop(3);
  population_trainer.setNRandom(3);
  population_trainer.setNReplicatesPerModel(3);
  population_trainer.setLogging(true);

  // define the population logger
  PopulationLogger<float> population_logger(true, true);

  // define the multithreading parameters
  const int n_hard_threads = std::thread::hardware_concurrency();
  //const int n_threads = n_hard_threads / 2; // the number of threads
  //char threads_cout[512];
  //sprintf(threads_cout, "Threads for population training: %d, Threads for model training/validation: %d\n",
  //	n_hard_threads, 2);
  //std::cout << threads_cout;
  const int n_threads = 1;

  // define the data simulator
  BiochemicalReactionModel<float> reaction_model;
  MetDataSimMultiTask<float> metabolomics_data;
  std::string model_name = "0_Metabolomics";

  // Read in the training and validation data
  std::string biochem_rxns_filename, metabo_data_filename, meta_data_filename;
  biochem_rxns_filename = data_dir + "iJO1366.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_train.csv";

  // Training data
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_train.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_training_ = reaction_model;

  // Validation data
  reaction_model.clear();
  metabo_data_filename = data_dir + "ALEsKOs01_Metabolomics_test.csv";
  meta_data_filename = data_dir + "ALEsKOs01_MetaData_test.csv";
  reaction_model.readBiochemicalReactions(biochem_rxns_filename, true);
  reaction_model.readMetabolomicsData(metabo_data_filename);
  reaction_model.readMetaData(meta_data_filename);
  reaction_model.findComponentGroupNames();
  reaction_model.findMARs();
  reaction_model.findMARs(true, false);
  reaction_model.findMARs(false, true);
  reaction_model.removeRedundantMARs();
  reaction_model.findLabels();
  metabolomics_data.model_validation_ = reaction_model;
  metabolomics_data.simulate_MARs_ = simulate_MARs;
  metabolomics_data.sample_concs_ = sample_concs;

  // Checks for the training and validation data
  assert(metabolomics_data.model_validation_.reaction_ids_.size() == metabolomics_data.model_training_.reaction_ids_.size());
  assert(metabolomics_data.model_validation_.labels_.size() == metabolomics_data.model_training_.labels_.size());
  assert(metabolomics_data.model_validation_.component_group_names_.size() == metabolomics_data.model_training_.component_group_names_.size());

  // Define the model input/output nodes
  int n_input_nodes;
  if (simulate_MARs) n_input_nodes = reaction_model.reaction_ids_.size();
  else n_input_nodes = reaction_model.component_group_names_.size();
  const int n_output_nodes_recon = n_input_nodes;
  const int n_output_nodes_class = reaction_model.labels_.size();
  const int encoding_size = 3;
  metabolomics_data.n_encodings_ = encoding_size;

  // Make the input nodes
  std::vector<std::string> input_nodes;
  for (int i = 0; i < n_input_nodes; ++i) {
    char name_char[512];
    sprintf(name_char, "Input_%012d", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the encoding nodes and add them to the input
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Encoding_%012d-Sampler", i);
    std::string name(name_char);
    input_nodes.push_back(name);
  }

  // Make the reconstruction nodes
  std::vector<std::string> output_nodes_recon;
  for (int i = 0; i < n_output_nodes_recon; ++i) {
    char name_char[512];
    sprintf(name_char, "Output-Recon_%012d", i);
    std::string name(name_char);
    output_nodes_recon.push_back(name);
  }

  // Make the mu nodes
  std::vector<std::string> encoding_nodes_mu;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "Mu_%012d", i);
    std::string name(name_char);
    encoding_nodes_mu.push_back(name);
  }

  // Make the encoding nodes
  std::vector<std::string> encoding_nodes_logvar;
  for (int i = 0; i < encoding_size; ++i) {
    char name_char[512];
    sprintf(name_char, "LogVar_%012d", i);
    std::string name(name_char);
    encoding_nodes_logvar.push_back(name);
  }

  // Make the classification nodes
  std::vector<std::string> output_nodes_class;
  for (int i = 0; i < n_output_nodes_class; ++i) {
    char name_char[512];
    sprintf(name_char, "Output-Class_%012d", i);
    std::string name(name_char);
    output_nodes_class.push_back(name);
  }

  // define the model trainers and resources for the trainers
  std::vector<ModelInterpreterGpu<float>> model_interpreters;
  for (size_t i = 0; i < n_threads; ++i) {
    ModelResources model_resources = { ModelDevice(0, 1) };
    ModelInterpreterGpu<float> model_interpreter(model_resources);
    model_interpreters.push_back(model_interpreter);
  }
  ModelTrainerExt<float> model_trainer;
  model_trainer.setBatchSize(16);
  model_trainer.setMemorySize(1);
  model_trainer.setNEpochsTraining(100000);
  model_trainer.setNEpochsValidation(0);
  model_trainer.setVerbosityLevel(1);
  model_trainer.setLogging(true, false, false);
  model_trainer.setFindCycles(false);
  model_trainer.setFastInterpreter(true);
  model_trainer.setPreserveOoO(true);
  model_trainer.setLossFunctions({
    std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionOp<float>>(new BCEWithLogitsOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceMuOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionOp<float>>(new KLDivergenceLogVarOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionOp<float>>(new CrossEntropyWithLogitsOp<float>()),
    std::shared_ptr<LossFunctionOp<float>>(new MSEOp<float>()) });
  model_trainer.setLossFunctionGrads({
    std::shared_ptr<LossFunctionGradOp<float>>(new MSEGradOp<float>(1e-6, 1.0)),
    //std::shared_ptr<LossFunctionGradOp<float>>(new BCEWithLogitsGradOp<float>(1e-6, 1.0)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceMuGradOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionGradOp<float>>(new KLDivergenceLogVarGradOp<float>(1e-6, 0.1)),
    std::shared_ptr<LossFunctionGradOp<float>>(new CrossEntropyWithLogitsGradOp<float>()),
    std::shared_ptr<LossFunctionGradOp<float>>(new MSEGradOp<float>()) });
  model_trainer.setLossOutputNodes({ output_nodes_recon, encoding_nodes_mu, encoding_nodes_logvar,
    output_nodes_class, output_nodes_class });
  model_trainer.setMetricFunctions({ std::shared_ptr<MetricFunctionOp<float>>(new MAEOp<float>()), 
    std::shared_ptr<MetricFunctionOp<float>>(new AccuracyMCMicroOp<float>()), 
    std::shared_ptr<MetricFunctionOp<float>>(new PrecisionMCMicroOp<float>()) });
  model_trainer.setMetricOutputNodes({ output_nodes_recon, output_nodes_class, output_nodes_class });
  model_trainer.setMetricNames({ "MAE", "AccuracyMCMicro", "PrecisionMCMicro" });

  // define the model logger
  ModelLogger<float> model_logger(true, true, false, false, false, false, false, false);

  // initialize the model replicator
  ModelReplicatorExt<float> model_replicator;

  // define the initial population
  std::cout << "Initializing the population..." << std::endl;
  //std::vector<Model<float>> population;
  Model<float> model;
  if (make_model) {
    model_trainer.makeModelFCMultiTask(model, n_input_nodes, n_output_nodes_recon, n_output_nodes_class, encoding_size, true, false, false, false); // normalization type 1
    //population = { model };
  }
  else {
    // TODO
  }

  // Train the model
  std::pair<std::vector<float>, std::vector<float>> model_errors = model_trainer.trainModel(model, metabolomics_data,
    input_nodes, model_logger, model_interpreters.front());

  //// Evolve the population
  //std::vector<std::vector<std::tuple<int, std::string, float>>> models_validation_errors_per_generation = population_trainer.evolveModels(
  //	population, model_trainer, model_interpreters, model_replicator, metabolomics_data, model_logger, population_logger, input_nodes);

  //PopulationTrainerFile<float> population_trainer_file;
  //population_trainer_file.storeModels(population, "Metabolomics");
  //population_trainer_file.storeModelValidations("MetabolomicsValidationErrors.csv", models_validation_errors_per_generation);
}

// Main
int main(int argc, char** argv)
{
  // Set the data directories
  //std::string data_dir = "C:/Users/dmccloskey/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  std::string data_dir = "C:/Users/domccl/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  //std::string data_dir = "C:/Users/domccl/Dropbox (UCSD SBRG)/Metabolomics_KALE/";
  //std::string data_dir = "/home/user/Data/";

  //main_statistics_timecourse(data_dir, 
  //	true, true, true, true, true,
  //	true);
  //main_statistics_timecourseSummary(data_dir, 
  //	true, true, true, true, true,
  //	true);
  main_classification(data_dir, true, true, true);
  //main_reconstruction(data_dir, true, false, true);
  //main_multiTask(data_dir, true, false, true);
  return 0;
}