/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp> // Sort
#endif

#define EIGEN_USE_THREADS
#include <unsupported/Eigen/CXX11/Tensor>
#include <chrono>
#include <functional> // Hash
#include <unordered_map> // Hash
#include <algorithm>  // Sort

using namespace std;

#if COMPILE_WITH_CUDA
std::size_t getStrHash(std::unordered_map<std::size_t, std::string>& hash_table, const std::string& str) {
  std::size_t hash = std::hash<std::string>{}(str);
  auto found = hash_table.emplace(hash, str);
  return hash;
}
template<int N>
void convertHashTensorToStrTensor(std::size_t* hash_data, std::string* string_data, const std::unordered_map<std::size_t, std::string>& hash_table, const int& dim_sizes...) {
  Eigen::TensorMap<Eigen::Tensor<std::size_t, N>> hash_tensor(hash_data, dim_sizes);
  Eigen::TensorMap<Eigen::Tensor<std::string, N>> string_tensor(string_data, dim_sizes);
  string_tensor = hash_tensor.unaryExpr([&hash_table](const std::size_t& elem) 
  { return hash_table.at(elem); });
}
template<int N, typename DeviceT>
void convertStrTensorToHashTensor(std::size_t* hash_data, std::string* string_data, const DeviceT& device, const int& dim_sizes...) {
  Eigen::TensorMap<Eigen::Tensor<std::size_t, N>> hash_tensor(hash_data, dim_sizes);
  Eigen::TensorMap<Eigen::Tensor<std::string, N>> string_tensor(string_data, dim_sizes);
  hash_tensor.device(device) = string_tensor.unaryExpr([](const std::string& elem)
  { return std::hash<std::string>{}(elem); });
}

template<typename DeviceT>
void stringCompareGpuEx(const std::size_t& dim1, DeviceT& cpuDevice) {
  // compare a string to a 1D tensor of strings
	assert(hipSetDevice(0) == hipSuccess); // is this needed?

	hipStream_t stream;

	std::size_t bytes = dim1 * sizeof(std::size_t);
  std::size_t bytes_str = dim1 * sizeof(std::string);
  std::string* h_str_in1;
	std::size_t* h_in1;
  std::size_t* d_in1;
  std::size_t* h_out1;
  std::string* h_str_out1;
  std::size_t* d_out1;

	// initialize the streams
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);

  auto startTime = std::chrono::high_resolution_clock::now();

	// allocate memory
	assert(hipHostAlloc((void**)(&h_in1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_str_in1), bytes_str, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_out1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_str_out1), bytes_str, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_in1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_out1), bytes) == hipSuccess);

	// initialize the GpuDevice
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device_(&stream_device);

  Eigen::TensorMap<Eigen::Tensor<std::string, 1>> in1_str(h_str_in1, dim1);
  in1_str.setConstant("Hello");
  in1_str(0) = "H";
  in1_str(1) = "He";
  in1_str(2) = "hello";
  convertStrTensorToHashTensor<1>(h_in1, h_str_in1, cpuDevice, (int)dim1);
  Eigen::TensorMap<Eigen::Tensor<std::size_t, 1>> in1(h_in1, dim1);

	device_.memcpyHostToDevice(d_in1, h_in1, bytes);

	Eigen::TensorMap<Eigen::Tensor<std::size_t, 1>> gpu_in1(d_in1, dim1);
  Eigen::TensorMap<Eigen::Tensor<std::size_t, 1>> gpu_out1(d_out1, dim1);  
	gpu_out1.device(device_) = (gpu_in1 == gpu_in1.constant(std::hash<std::string>{}("hello"))).select(gpu_in1, gpu_in1.constant(std::hash<std::string>{}("NA")));

	device_.memcpyDeviceToHost(h_out1, d_out1, bytes);

  assert(hipStreamSynchronize(stream) == hipSuccess);
  assert(hipStreamDestroy(stream) == hipSuccess);

  Eigen::TensorMap<Eigen::Tensor<std::size_t, 1>> out1(h_out1, dim1);
  assert(out1(0) == std::hash<std::string>{}("NA"));
  assert(out1(1) == std::hash<std::string>{}("NA"));
  assert(out1(2) == std::hash<std::string>{}("hello"));
  assert(out1(3) == std::hash<std::string>{}("NA"));

  // NOTE: ~ 5294 ms GPU and default device ~4161 ms GPU and 8 threads
  Eigen::TensorMap<Eigen::Tensor<std::string, 1>> out1_str(h_str_out1, dim1);
  out1_str.device(cpuDevice) = (out1 == in1).select(in1_str, in1_str.constant("NA"));
  assert(out1_str(0) == "NA");
  assert(out1_str(1) == "NA");
  assert(out1_str(2) == "hello");
  assert(out1_str(3) == "NA");

	auto endTime = std::chrono::high_resolution_clock::now();
	int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
	std::cout << "GPU took: " << time_to_run << " ms." << std::endl;

	// free all resources

	assert(hipFree(d_in1) == hipSuccess);
	assert(hipFree(d_out1) == hipSuccess);

	assert(hipHostFree(h_in1) == hipSuccess);
  assert(hipHostFree(h_str_in1) == hipSuccess);
	assert(hipHostFree(h_out1) == hipSuccess);
  assert(hipHostFree(h_str_out1) == hipSuccess);
};
void stringCompareGpuCharEx(const std::size_t& dim1, const std::size_t& n_char) {
  // compare a string to a 1D tensor of strings
  assert(hipSetDevice(0) == hipSuccess); // is this needed?

  hipStream_t stream;
  std::size_t bytes = dim1 * n_char * sizeof(char);
  char* h_in1;
  char* d_in1;
  char* d_scratch1;
  char* h_out1;
  char* d_out1;

  // initialize the streams
  assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);

  auto startTime = std::chrono::high_resolution_clock::now();
  
  // allocate memory
  assert(hipHostAlloc((void**)(&h_in1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_out1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipMalloc((void**)(&d_in1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_scratch1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_out1), bytes) == hipSuccess);

  // initialize the GpuDevice
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device_(&stream_device);

  Eigen::ThreadPool pool(8);
  Eigen::ThreadPoolDevice cpuDevice(&pool, 8);

  Eigen::TensorMap<Eigen::Tensor<char, 2>> in1(h_in1, n_char, dim1);
  in1.setZero();
  //in1.chip(2, 1).slice(0, 5);

  device_.memcpyHostToDevice(d_in1, h_in1, bytes);

  Eigen::TensorMap<Eigen::Tensor<char, 2>> gpu_in1(d_in1, n_char, dim1);
  Eigen::TensorMap<Eigen::Tensor<char, 2>> gpu_scratch1(d_scratch1, n_char, 1);
  Eigen::TensorMap<Eigen::Tensor<char, 2>> gpu_out1(d_out1, n_char, dim1);
  //gpu_scratch1.chip(0, 1).device(device_) = "hello";
  //gpu_scratch1.slice(Eigen::array<int, 2>({ 0,0 }), Eigen::array<int, 2>({ (int)n_char, 0 })).device(device_) = "hello";
  gpu_out1.device(device_) = (gpu_in1 == gpu_scratch1.broadcast(Eigen::array<int, 2>({ 1, (int)dim1 }))).select(gpu_in1, gpu_in1.constant('\0'));

  device_.memcpyDeviceToHost(h_out1, d_out1, bytes);

  assert(hipStreamSynchronize(stream) == hipSuccess);
  assert(hipStreamDestroy(stream) == hipSuccess);

  Eigen::TensorMap<Eigen::Tensor<char, 2>> out1(h_out1, n_char, dim1);
  //std::cout << "NA: " << out1.chip(0, 1) << "hello: " << out1.chip(2, 1);
  //assert(out1.chip(0, 1) == "NA");
  //assert(out1.chip(1, 1) == "NA");
  //assert(out1.chip(2, 1) == "hello");
  //assert(out1.chip(3, 1) == "NA");

  // NOTE: ~ 752 ms with char of 16
  //       ~ 5,639 ms with char of 128
  //       ~ 11,111 ms with char of 256

  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "GPU took: " << time_to_run << " ms." << std::endl;

  // free all resources

  assert(hipFree(d_in1) == hipSuccess);
  assert(hipFree(d_out1) == hipSuccess);
  assert(hipFree(d_scratch1) == hipSuccess);

  assert(hipHostFree(h_in1) == hipSuccess);
  assert(hipHostFree(h_out1) == hipSuccess);
};
template<typename DeviceT>
void stringCompareCpuEx(const std::size_t& dim1, DeviceT& cpuDevice) {

  std::string* h_str_in1 = new std::string[dim1];
  std::string* h_str_out1 = new std::string[dim1];;
  auto startTime = std::chrono::high_resolution_clock::now();

  Eigen::TensorMap<Eigen::Tensor<std::string, 1>> in1_str(h_str_in1, dim1);
  in1_str.setConstant("Hello");
  in1_str(0) = "H";
  in1_str(1) = "He";
  in1_str(2) = "hello";

  // NOTE: ~ 6291 ms
  Eigen::TensorMap<Eigen::Tensor<std::string, 1>> out1_str(h_str_out1, dim1);
  out1_str.device(cpuDevice) = (in1_str == in1_str.constant("hello")).select(in1_str, in1_str.constant("NA"));
  assert(out1_str(0) == "NA");
  assert(out1_str(1) == "NA");
  assert(out1_str(2) == "hello");
  assert(out1_str(3) == "NA");

  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "Cpu took: " << time_to_run << " ms." << std::endl;
};

template<typename NumericT>
void numericSortGpuEx(const std::size_t& dim1) {
  assert(hipSetDevice(0) == hipSuccess); // is this needed?

  hipStream_t stream;

  std::size_t bytes = dim1 * sizeof(NumericT);
  std::size_t bytes_index = dim1 * sizeof(int);
  NumericT* h_in1;
  int* h_index_in1;
  NumericT* d_in1;
  int* d_index_in1;
  NumericT* h_out1;
  int* h_index_out1;
  NumericT* d_out1;
  int* d_index_out1;

  // initialize the streams
  assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);

  auto startTime = std::chrono::high_resolution_clock::now();

  // allocate memory
  assert(hipHostAlloc((void**)(&h_in1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_index_in1), bytes_index, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_out1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_index_out1), bytes_index, hipHostMallocDefault) == hipSuccess);
  assert(hipMalloc((void**)(&d_in1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_index_in1), bytes_index) == hipSuccess);
  assert(hipMalloc((void**)(&d_out1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_index_out1), bytes_index) == hipSuccess);

  // initialize the GpuDevice
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device_(&stream_device);

  Eigen::TensorMap<Eigen::Tensor<int, 1>> index_in1(h_index_in1, dim1);
  index_in1.setZero();
  for (int i = 1; i < dim1; ++i) {
    index_in1(i) = index_in1(i - 1) + 1;
  }
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> in1(h_in1, dim1);
  in1.setConstant(dim1);
  for (int i = 1; i < dim1; ++i) {
    in1(i) = in1(i - 1) - 1;
  }

  device_.memcpyHostToDevice(d_in1, h_in1, bytes);
  device_.memcpyHostToDevice(d_index_in1, h_index_in1, bytes_index);

  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> gpu_in1(d_in1, dim1);
  Eigen::TensorMap<Eigen::Tensor<int, 1>> gpu_index_in1(d_index_in1, dim1);
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> gpu_out1(d_out1, dim1);
  Eigen::TensorMap<Eigen::Tensor<int, 1>> gpu_index_out1(d_index_out1, dim1);

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
    d_in1, d_out1, d_index_in1, d_index_out1, dim1, 0, sizeof(NumericT) * 8, stream);

  // Allocate temporary storage
  assert(hipMalloc((void**)(&d_temp_storage), temp_storage_bytes) == hipSuccess);

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
    d_in1, d_out1, d_index_in1, d_index_out1, dim1, 0, sizeof(NumericT) * 8, stream);

  device_.memcpyDeviceToHost(h_out1, d_out1, bytes);
  device_.memcpyDeviceToHost(h_index_out1, d_index_out1, bytes);

  assert(hipStreamSynchronize(stream) == hipSuccess);
  assert(hipStreamDestroy(stream) == hipSuccess);

  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> out1(h_out1, dim1);
  Eigen::TensorMap<Eigen::Tensor<int, 1>> index_out1(h_index_out1, dim1);
  assert(out1(0) == 1);
  assert(out1(dim1-1) == dim1);
  assert(index_out1(0) == dim1 - 1);
  assert(index_out1(dim1 - 1) == 0);

  // NOTE: ~ 97 ms with dim1=1e6

  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "GPU took: " << time_to_run << " ms." << std::endl;

  // free all resources

  assert(hipFree(d_in1) == hipSuccess);
  assert(hipFree(d_index_in1) == hipSuccess);
  assert(hipFree(d_out1) == hipSuccess);
  assert(hipFree(d_index_out1) == hipSuccess);

  assert(hipHostFree(h_in1) == hipSuccess);
  assert(hipHostFree(h_index_in1) == hipSuccess);
  assert(hipHostFree(h_out1) == hipSuccess);
  assert(hipHostFree(h_index_out1) == hipSuccess);
}
template<typename NumericT>
void numericSortCpuEx(const std::size_t& dim1) {

  std::size_t bytes = dim1 * sizeof(NumericT);
  std::size_t bytes_index = dim1 * sizeof(int);
  NumericT* h_in1 =  new NumericT[dim1];
  int* h_index_in1 = new int[dim1];
  NumericT* h_out1 = new NumericT[dim1];
  int* h_index_out1 = new int[dim1];
  
  auto startTime = std::chrono::high_resolution_clock::now();

  Eigen::TensorMap<Eigen::Tensor<int, 1>> index_in1(h_index_in1, dim1);
  index_in1.setZero();
  for (int i = 1; i < dim1; ++i) {
    index_in1(i) = index_in1(i - 1) + 1;
  }
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> in1(h_in1, dim1);
  in1.setConstant(dim1);
  for (int i = 1; i < dim1; ++i) {
    in1(i) = in1(i - 1) - 1;
  }

  // Create a copy
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> out1(h_out1, dim1);
  out1 = in1;
  Eigen::TensorMap<Eigen::Tensor<int, 1>> index_out1(h_index_out1, dim1);
  index_out1 = index_in1;

  // Run sorting operation
  std::sort(index_out1.data(), index_out1.data() + index_out1.size(), [&out1](const int&a, const int&b) {
    return out1(a) < out1(b);
  });
  std::sort(out1.data(), out1.data() + out1.size(), [](const NumericT&a, const NumericT&b) {
    return a < b;
  });

  assert(out1(0) == 1);
  assert(out1(dim1 - 1) == dim1);
  assert(index_out1(0) == dim1 - 1);
  assert(index_out1(dim1 - 1) == 0);

  // NOTE: ~ 1666 ms with dim1=1e6
  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "CPU took: " << time_to_run << " ms." << std::endl;
}

template<typename NumericT>
void numericCompareGpuEx(const std::size_t& dim1) {
  // compare a string to a 1D tensor of strings
  assert(hipSetDevice(0) == hipSuccess); // is this needed?

  hipStream_t stream;

  std::size_t bytes = dim1 * sizeof(NumericT);
  NumericT* h_in1;
  NumericT* d_in1;
  NumericT* h_out1;
  NumericT* d_out1;

  // initialize the streams
  assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);

  auto startTime = std::chrono::high_resolution_clock::now();

  // allocate memory
  assert(hipHostAlloc((void**)(&h_in1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipHostAlloc((void**)(&h_out1), bytes, hipHostMallocDefault) == hipSuccess);
  assert(hipMalloc((void**)(&d_in1), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_out1), bytes) == hipSuccess);

  // initialize the GpuDevice
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device_(&stream_device);

  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> in1(h_in1, dim1);
  in1.setZero();
  in1(0) = 1;
  in1(1) = 2;
  in1(2) = 3;

  device_.memcpyHostToDevice(d_in1, h_in1, bytes);

  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> gpu_in1(d_in1, dim1);
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> gpu_out1(d_out1, dim1);
  gpu_out1.device(device_) = (gpu_in1 > gpu_in1.constant(0)).select(gpu_in1, gpu_in1.constant(10));

  device_.memcpyDeviceToHost(h_out1, d_out1, bytes);

  assert(hipStreamSynchronize(stream) == hipSuccess);
  assert(hipStreamDestroy(stream) == hipSuccess);

  // NOTE: ~ 5294 ms GPU and default device ~4161 ms GPU and 8 threads
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> out1(h_out1, dim1);
  assert(out1(0) == 1);
  assert(out1(1) == 2);
  assert(out1(2) == 3);
  assert(out1(3) == 10);

  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "GPU took: " << time_to_run << " ms." << std::endl;

  // free all resources
  assert(hipFree(d_in1) == hipSuccess);
  assert(hipFree(d_out1) == hipSuccess);

  assert(hipHostFree(h_in1) == hipSuccess);
  assert(hipHostFree(h_out1) == hipSuccess);
};
template<typename NumericT, typename DeviceT>
void numericCompareCpuEx(const std::size_t& dim1, DeviceT& cpuDevice) {

  NumericT* h_in1 = new NumericT[dim1];
  NumericT* h_out1 = new NumericT[dim1];;
  auto startTime = std::chrono::high_resolution_clock::now();

  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> in1(h_in1, dim1);
  in1.setZero();
  in1(0) = 1;
  in1(1) = 2;
  in1(2) = 3;

  // NOTE: ~ 6291 ms
  Eigen::TensorMap<Eigen::Tensor<NumericT, 1>> out1(h_out1, dim1);
  out1.device(cpuDevice) = (in1 > in1.constant(0)).select(in1, in1.constant(10));
  assert(out1(0) == 1);
  assert(out1(1) == 2);
  assert(out1(2) == 3);
  assert(out1(3) == 10);

  auto endTime = std::chrono::high_resolution_clock::now();
  int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
  std::cout << "Cpu took: " << time_to_run << " ms." << std::endl;
};

int main(int argc, char** argv)
{
	hipError_t err = hipDeviceReset();

  // get the device memory
  size_t free_byte, total_byte;
  hipMemGetInfo(&free_byte, &total_byte);
  std::cout << "Free memory: " << free_byte << "; Total memory: " << total_byte << std::endl;

  // get the number of gpus
  int n_gpus = 0;
  hipGetDeviceCount(&n_gpus);
  if (n_gpus > 0)
    std::cout << n_gpus << " were found." << std::endl;

  // Benchmarks to run
  bool string_comparison = false;
  bool sort_comparison = false;
  bool numeric_comparison = true;
  Eigen::ThreadPool pool(8);
  Eigen::ThreadPoolDevice cpuDevice(&pool, 8);
  Eigen::DefaultDevice defaultDevice;

	// String comparison tests
  // GPU could be improved by https://github.com/NVIDIA/nvstrings or https://nvlabs.github.io/nvbio/structnvbio_1_1cuda_1_1_compression_sort.html
  if (string_comparison) {

    stringCompareGpuEx(1e6, cpuDevice);
    stringCompareGpuCharEx(1e6, 128);
    stringCompareCpuEx(1e6, defaultDevice);
    stringCompareCpuEx(1e6, cpuDevice);

    stringCompareGpuEx(1e3, cpuDevice);
    stringCompareGpuCharEx(1e3, 128);
    stringCompareCpuEx(1e3, defaultDevice);
    stringCompareCpuEx(1e3, cpuDevice);
  }

  // Sorting comparison tests
  // TODO: add test that applies sorted index to sort all other Tensors
  //       in a single Gpu call by linearizing across the Tensor columns (col storage)
  if (sort_comparison) {
    numericSortGpuEx<float>(1e6);
    numericSortCpuEx<float>(1e6);

    numericSortGpuEx<float>(1e3);
    numericSortCpuEx<float>(1e3);
  }

  // Numeric comparison tests
  if (numeric_comparison) {
    numericCompareGpuEx<float>(1e6);
    numericCompareCpuEx<float>(1e6, defaultDevice);
    numericCompareCpuEx<float>(1e6, cpuDevice);

    numericCompareGpuEx<float>(1e3);
    numericCompareCpuEx<float>(1e3, defaultDevice);
    numericCompareCpuEx<float>(1e3, cpuDevice);
  }

  // Group by and count comparison
  // see hipcub::DeviceRunLengthEncode::Encode in https://nvlabs.github.io/cub/structcub_1_1_device_run_length_encode.html#ab25e5e8289fe198b8fea68ac5f010118

  // Binning and histogram comparison
  // see hipcub::DeviceHistogram::HistogramEven, hipcub::DeviceHistogram::HistogramRange
  // see hipcub::DeviceHistogram::MultiHistogramEven, hipcub::DeviceHistogram::MultiHistogramRange

  // Selection comparison in https://nvlabs.github.io/cub/structcub_1_1_device_select.html#details
  // flagged selection using hipcub::DeviceSelect::Flagged
  // if selection using hipcub::DeviceSelect::If (composed of Tensor select and flagged selection)
  // unique selection using hipcub::DeviceSelect::Unique

  return 0;
}
#endif