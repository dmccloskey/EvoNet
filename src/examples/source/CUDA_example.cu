/**TODO:  Add copyright*/

#ifndef EVONET_CUDA
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#include <unsupported/Eigen/CXX11/Tensor>
#include <SmartPeak/core/GPUDevice.h>
#include <SmartPeak/core/CPUDevice.h>

using namespace SmartPeak;
using namespace std;

// adapted from "eigen / unsupported / test / cxx11_tensor_cuda.cu"
void asyncExample() {
	assert(hipSetDevice(0) == hipSuccess); // is this needed?

	const int max_streams = 32;
	hipStream_t streams[max_streams];

	std::size_t in1_bytes = 40 * 50 * 70 * sizeof(float);
	std::size_t in2_bytes = 40 * 50 * 70 * sizeof(float);
	std::size_t out_bytes = 40 * 50 * 70 * sizeof(float);

	float* h_in1[max_streams];
	float* h_in2[max_streams];
	float* h_out[max_streams];

	float* d_in1[max_streams];
	float* d_in2[max_streams];
	float* d_out[max_streams];
	for (int i = 0; i < max_streams; ++i) {
		// initialize the streams
		assert(hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking) == hipSuccess);

		// allocate memory
		assert(hipHostAlloc((void**)(&h_in1[i]), in1_bytes, hipHostMallocDefault) == hipSuccess);
		assert(hipHostAlloc((void**)(&h_in2[i]), in2_bytes, hipHostMallocDefault) == hipSuccess);
		assert(hipHostAlloc((void**)(&h_out[i]), out_bytes, hipHostMallocDefault) == hipSuccess);
		assert(hipMalloc((void**)(&d_in1[i]), in1_bytes) == hipSuccess);
		assert(hipMalloc((void**)(&d_in2[i]), in2_bytes) == hipSuccess);
		assert(hipMalloc((void**)(&d_out[i]), out_bytes) == hipSuccess);
	}

	auto startTime = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < max_streams; ++i) {

		// initialize the GpuDevice
		Eigen::GpuStreamDevice stream_device(&streams[i], 0);
		Eigen::GpuDevice device_(&stream_device);

		Eigen::TensorMap<Eigen::Tensor<float, 3> > in1(h_in1[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > in2(h_in2[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > out(h_out[i], 40, 50, 70);
		in1 = in1.random() + in1.constant(10.0f);
		in2 = in2.random() + in2.constant(10.0f);

		assert(hipMemcpyAsync(d_in1[i], in1.data(), in1_bytes, hipMemcpyHostToDevice, streams[i]) == hipSuccess);
		assert(hipMemcpyAsync(d_in2[i], in2.data(), in2_bytes, hipMemcpyHostToDevice, streams[i]) == hipSuccess);
		//device_.memcpyHostToDevice(d_in1[i], in1.data(), in1_bytes);
		//device_.memcpyHostToDevice(d_in2[i], in2.data(), in2_bytes);

		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out[i], 40, 50, 70);

		gpu_out.device(device_) = gpu_in1 + gpu_in2;

		assert(hipMemcpyAsync(out.data(), d_out[i], out_bytes, hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
		//device_.memcpyDeviceToHost(h_out[i], d_out[i], out_bytes);
	}
	auto endTime = std::chrono::high_resolution_clock::now();
	int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
	std::cout << "took: " << time_to_run << " ms." << std::endl;

	// free all resources
	for (int i = 0; i < max_streams; ++i) {

		assert(hipFree(d_in1[i]) == hipSuccess);
		assert(hipFree(d_in2[i]) == hipSuccess);
		assert(hipFree(d_out[i]) == hipSuccess);

		assert(hipHostFree(h_in1[i]) == hipSuccess);
		assert(hipHostFree(h_in2[i]) == hipSuccess);
		assert(hipHostFree(h_out[i]) == hipSuccess);

		assert(hipStreamSynchronize(streams[i]) == hipSuccess);
		assert(hipStreamDestroy(streams[i]) == hipSuccess);
	}
};
void syncExample() {
	assert(hipSetDevice(0) == hipSuccess); // is this needed?

	const int max_streams = 32;

	std::size_t in1_bytes = 40 * 50 * 70 * sizeof(float);
	std::size_t in2_bytes = 40 * 50 * 70 * sizeof(float);
	std::size_t out_bytes = 40 * 50 * 70 * sizeof(float);

	float* d_in1[max_streams];
	float* d_in2[max_streams];
	float* d_out[max_streams];

	// initialize the streams
	for (int i = 0; i < max_streams; ++i) {
		// allocate memory
		assert(hipMalloc((void**)(&d_in1[i]), in1_bytes) == hipSuccess);
		assert(hipMalloc((void**)(&d_in2[i]), in2_bytes) == hipSuccess);
		assert(hipMalloc((void**)(&d_out[i]), out_bytes) == hipSuccess);
	}

	auto startTime = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < max_streams; ++i) {
		// initialize the GpuDevice
		Eigen::GpuStreamDevice stream_device(0);
		Eigen::GpuDevice device_(&stream_device);

		Eigen::Tensor<float, 3> in1(40, 50, 70);
		Eigen::Tensor<float, 3> in2(40, 50, 70);
		Eigen::Tensor<float, 3> out(40, 50, 70);
		in1 = in1.random() + in1.constant(10.0f);
		in2 = in2.random() + in2.constant(10.0f);

		device_.memcpyHostToDevice(d_in1[i], in1.data(), in1_bytes);
		device_.memcpyHostToDevice(d_in2[i], in2.data(), in2_bytes);

		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2[i], 40, 50, 70);
		Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out[i], 40, 50, 70);

		gpu_out.device(device_) = gpu_in1 + gpu_in2;

		device_.memcpyDeviceToHost(out.data(), d_out[i], out_bytes);
		assert(hipStreamSynchronize(device_.stream()) == hipSuccess);
	}
	auto endTime = std::chrono::high_resolution_clock::now();
	int time_to_run = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
	std::cout << "took: " << time_to_run << " ms." << std::endl;
	// free all resources
	for (int i = 0; i < max_streams; ++i) {

		assert(hipFree(d_in1[i]) == hipSuccess);
		assert(hipFree(d_in2[i]) == hipSuccess);
		assert(hipFree(d_out[i]) == hipSuccess);
	}
};

int main(int argc, char** argv)
{
	asyncExample();
	syncExample();
	
	// get the number of async engines
	hipDeviceProp_t prop;
	int whichDevice;
	int deviceOverlap;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	std::cout << prop.asyncEngineCount << std::endl;
	std::cout << prop.multiProcessorCount << std::endl;

	// get the number of gpus
	int n_gpus = 0;
	hipGetDeviceCount(&n_gpus);
  if (n_gpus > 0)
  {
	std::cout << n_gpus <<" were found." << std::endl;
  }

  return 0;
}