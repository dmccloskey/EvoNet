
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <cuda.h>
#include <cuda_runtime.h>
#include <unsupported/Eigen/CXX11/Tensor>
#include <SmartPeak/core/Preprocessing.h>
#include <SmartPeak/ml/ActivationFunctionTensor.h>

using namespace SmartPeak;
using namespace std;

void test_operationfunctionReluTensorOp() 
{
  ReLUTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{0,0}, {0,0}},
		{{0,0}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionReluGradTensorOp() 
{
  ReLUGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{0,0}, {0,0}},
		{{0,0}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionEluTensorOp() 
{
  ELUTensorOp<double, Eigen::GpuDevice> operation(1.0);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-0.63212055882855767,-0.63212055882855767}, {0,0}},
		{{-0.99995460007023751,-0.99995460007023751}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionEluGradTensorOp() 
{
  ELUGradTensorOp<double, Eigen::GpuDevice> operation(1.0);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{0.36787944117144233,0.36787944117144233}, {0,0}},
		{{4.5399929762490743e-05,4.5399929762490743e-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionSigmoidTensorOp() 
{
  SigmoidTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0.5,0.5}, {0,0}},
		{{0.7310585786300049,0.7310585786300049}, {0,0}},
		{{0.99995460213129761,0.99995460213129761}, {0,0}},
		{{0.2689414213699951,0.2689414213699951}, {0,0}},
		{{4.5397868702434395e-05,4.5397868702434395e-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionSigmoidGradTensorOp() 
{
  SigmoidGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0.25,0.25}, {0,0}},
		{{0.19661193324148185,0.19661193324148185}, {0,0}},
		{{4.5395807735907655e-05,4.5395807735907655e-05}, {0,0}},
		{{0.19661193324148185,0.19661193324148185}, {0,0}},
		{{4.53958091e-05,4.53958091e-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionTanHTensorOp() 
{
  TanHTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0.0,0.0}, {0,0}},
		{{0.76159415595576485,0.76159415595576485}, {0,0}},
		{{0.99999999587769262,0.99999999587769262}, {0,0}},
		{{-0.76159415595576485,-0.76159415595576485}, {0,0}},
		{{-0.99999999587769262,-0.99999999587769262}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionTanHGradTensorOp() 
{
  TanHGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{0.41997434161402614,0.41997434161402614}, {0,0}},
		{{8.2446145466263943e-09,8.2446145466263943e-09}, {0,0}},
		{{0.41997434161402614,0.41997434161402614}, {0,0}},
		{{8.2446145466263943e-09,8.2446145466263943e-09}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

// [TODO: need to re-implement]
void test_operationfunctionReTanHTensorOp() 
{
  ReTanHTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{0.76159415595576485,0.76159415595576485}, {0,0}},
		{{0.99999999587769262,0.99999999587769262}, {0,0}},
		{{0,0}, {0,0}},
		{{0,0}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

// TODO: need to re-implement
void test_operationfunctionReTanHGradTensorOp() 
{
  ReTanHGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{0.41997434161402614,0.41997434161402614}, {0,0}},
		{{8.2446147686709992e-09,8.2446147686709992e-09}, {0,0}},
		{{0,0}, {0,0}},
		{{0,0}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionLinearTensorOp()
{
	LinearTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionLinearGradTensorOp()
{
	LinearGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionInverseTensorOp()
{
	InverseTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{0.1,0.1}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-0.1,-0.1}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i,j,k), test(i,j,k)));
			}
		}
	}
}

void test_operationfunctionInverseGradTensorOp()
{
	InverseGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-0.01,-0.01}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-0.01,-0.01}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionExponentialTensorOp()
{
	ExponentialTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionExponentialGradTensorOp()
{
	ExponentialGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionLogTensorOp()
{
	LogTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} 
		});
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{-13.815510557964274,-13.815510557964274}, {0,0}},
		{{0,0}, {0,0}},
		{{2.3025850929940459,2.3025850929940459}, {0,0}},
		{{-13.815510557964274,-13.815510557964274}, {0,0}},
		{{-13.815510557964274,-13.815510557964274}, {0,0}}
		});

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionLogGradTensorOp()
{
	LogGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1000000000,1000000000}, {0,0}},
		{{1,1}, {0,0}},
		{{0.1,0.1}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-0.1,-0.1}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionPowTensorOp()
{
	PowTensorOp<double, Eigen::GpuDevice> operation(0.5);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{3.1622776601683795,3.1622776601683795}, {0,0}},
		{{-1.0e9,-1.0e9}, {0,0}},  // TODO: Clip does not fix -nan(ind)
		{{-1.0e9,-1.0e9}, {0,0}}});

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionPowGradTensorOp()
{
	PowGradTensorOp<double, Eigen::GpuDevice> operation(0.5);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1.0e9,1.0e9}, {0,0}},
		{{0.5,0.5}, {0,0}},
		{{0.15811388300841897,0.15811388300841897}, {0,0}},
		{{-1.0e9,-1.0e9}, {0,0}},  // TODO: Clip does not fix -nan(ind)
		{{-1.0e9,-1.0e9}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionLeakyReLUTensorOp()
{
	LeakyReLUTensorOp<double, Eigen::GpuDevice> operation(0.1);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-0.1,-0.1}, {0,0}},
		{{-1,-1}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionLeakyReLUGradTensorOp()
{
	LeakyReLUGradTensorOp<double, Eigen::GpuDevice> operation(0.1);
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{1,1}, {0,0}},
		{{0.1,0.1}, {0,0}},
		{{0.1,0.1}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				assert(assert_close(output(i, j, k), test(i, j, k)));
			}
		}
	}
}

void test_operationfunctionSinTensorOp()
{
	SinTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				//assert(assert_close(output(i, j, k), test(i, j, k))); //TODO: fixme
			}
		}
	}
}

void test_operationfunctionSinGradTensorOp()
{
	SinGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				//assert(assert_close(output(i, j, k), test(i, j, k))); //TODO: fixme
			}
		}
	}
}

void test_operationfunctionCosTensorOp()
{
	CosTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				//assert(assert_close(output(i, j, k), test(i, j, k))); //TODO: fixme
			}
		}
	}
}

void test_operationfunctionCosGradTensorOp()
{
	CosGradTensorOp<double, Eigen::GpuDevice> operation;
	const int batch_size = 5;
	const int memory_size = 2;
	const int layer_size = 2;
	cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);
	Eigen::Tensor<double, 3> input(batch_size, memory_size, layer_size);
	input.setValues({
		{{0,0}, {0,0}},
		{{1,1}, {0,0}},
		{{10,10}, {0,0}},
		{{-1,-1}, {0,0}},
		{{-10,-10}, {0,0}} });
	Eigen::Tensor<double, 3> output(batch_size, memory_size, layer_size);
	output.setZero();
	Eigen::Tensor<double, 3> test(batch_size, memory_size, layer_size);
	test.setValues({
		{{1,1}, {0,0}},
		{{2.718281828,2.718281828}, {0,0}},
		{{22026.46579,22026.46579}, {0,0}},
		{{0.367879441,0.367879441}, {0,0}},
		{{4.53999E-05,4.53999E-05}, {0,0}} });

	operation(input.data(), output.data(), batch_size, memory_size, layer_size, 0, device);

	// Test
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			for (int k = 0; k < layer_size; ++k) {
				//assert(assert_close(output(i, j, k), test(i, j, k))); //TODO: fixme
			}
		}
	}
}

int main(int argc, char** argv)
{
  test_operationfunctionReluTensorOp();
  test_operationfunctionReluGradTensorOp();
  test_operationfunctionEluTensorOp();
  test_operationfunctionEluGradTensorOp();
  test_operationfunctionSigmoidTensorOp();
  test_operationfunctionSigmoidGradTensorOp();
  test_operationfunctionTanHTensorOp();
  test_operationfunctionTanHGradTensorOp();
  test_operationfunctionReTanHTensorOp();
  test_operationfunctionReTanHGradTensorOp();
  test_operationfunctionLinearTensorOp();
  test_operationfunctionLinearGradTensorOp();
  test_operationfunctionInverseTensorOp();
  test_operationfunctionInverseGradTensorOp();
  test_operationfunctionExponentialTensorOp();
  test_operationfunctionExponentialGradTensorOp();
  test_operationfunctionLogTensorOp();
  test_operationfunctionLogGradTensorOp();
  test_operationfunctionPowTensorOp();
  test_operationfunctionPowGradTensorOp();
  test_operationfunctionLeakyReLUTensorOp();
  test_operationfunctionLeakyReLUGradTensorOp();
  test_operationfunctionSinTensorOp();
  test_operationfunctionSinGradTensorOp();
  test_operationfunctionCosTensorOp();
  test_operationfunctionCosGradTensorOp();
  return 0;
}
#endif