/**TODO:  Add copyright*/

#ifndef EVONET_CUDA

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <SmartPeak/core/KernalManager.h>

using namespace SmartPeak;
using namespace std;

void test_nodeActivationGpuDevice()
{
	GpuKernal<float> kernal;
	const int device_id = 0;

	ActivationOpWrapper<float, Eigen::GpuDevice>* activation_function = new ReLUOpWrapper<float, Eigen::GpuDevice>();
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int source_time_step = 0;
	const int node_time_step = 0;

	float* h_node_input;
	float* d_node_input;
	float* h_node_output;
	float* d_node_output;
	float* h_node_dt;
	float* d_node_dt;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_node_input), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_input), bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_node_output), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_output), bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_node_dt), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_dt), bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_input(h_node_input, batch_size, memory_size, layer_size);
	node_input.setValues({ {{-1, 1}, {0, 0}},
		{{-2, 2}, {0, 0}},
		{{-3, 3}, {0, 0}},
		{{-4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_output(h_node_output, batch_size, memory_size, layer_size);
	node_output.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_dt(h_node_dt, batch_size, memory_size, layer_size);
	node_dt.setConstant(1);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeNodeActivation(
		h_node_input,
		d_node_input,
		h_node_output,
		d_node_output,
		h_node_dt,
		d_node_dt,
		activation_function,
		batch_size,
		memory_size,
		layer_size,
		node_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 3> expected_output(batch_size, memory_size, layer_size);
	expected_output.setValues({ {{0, 1}, {0, 0}},
		{{0, 2}, {0, 0}},
		{{0, 3}, {0, 0}},
		{{0, 4}, {0, 0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				std::cout << "[Output] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_output(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_output(batch_iter, memory_iter, node_iter) == expected_output(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(hipHostFree(h_node_input) == hipSuccess);
	assert(hipFree(d_node_input) == hipSuccess);
	assert(hipHostFree(h_node_output) == hipSuccess);
	assert(hipFree(d_node_output) == hipSuccess);
	assert(hipHostFree(h_node_dt) == hipSuccess);
	assert(hipFree(d_node_dt) == hipSuccess);
}

void test_nodeDerivativeGpuDevice()
{
	GpuKernal<float> kernal;
	const int device_id = 0;

	ActivationOpWrapper<float, Eigen::GpuDevice>* activation_grad_function = new ReLUGradOpWrapper<float, Eigen::GpuDevice>();
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int source_time_step = 0;
	const int node_time_step = 0;

	float* h_node_output;
	float* d_node_output;
	float* h_node_derivative;
	float* d_node_derivative;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_node_output), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_output), bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_node_derivative), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_derivative), bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_output(h_node_output, batch_size, memory_size, layer_size);
	node_output.setValues({ {{-1, 1}, {0, 0}},
		{{-2, 2}, {0, 0}},
		{{-3, 3}, {0, 0}},
		{{-4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_derivative(h_node_derivative, batch_size, memory_size, layer_size);
	node_derivative.setConstant(0);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeNodeDerivative(
		h_node_output,
		d_node_output,
		h_node_derivative,
		d_node_derivative,
		activation_grad_function,
		batch_size,
		memory_size,
		layer_size,
		node_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 3> expected_derivative(batch_size, memory_size, layer_size);
	expected_derivative.setValues({ {{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				std::cout << "[Derivative] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_derivative(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_derivative(batch_iter, memory_iter, node_iter) == expected_derivative(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(hipHostFree(h_node_output) == hipSuccess);
	assert(hipFree(d_node_output) == hipSuccess);
	assert(hipHostFree(h_node_derivative) == hipSuccess);
	assert(hipFree(d_node_derivative) == hipSuccess);
}

void test_forwardPropogationGpuDevice()
{
	GpuKernal<float> kernal;
	const int device_id = 0;

	IntegrationOp<float, Eigen::GpuDevice>* integration_function = new FullyConnectedSumOp<float, Eigen::GpuDevice>();
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;
	const int source_time_steps = 0;
	const int sink_time_step = 0;

	float* h_source_outputs;
	float* d_source_outputs;
	float* h_weights;
	float* d_weights;
	float* h_sink_input;
	float* d_sink_input;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_source_outputs), source_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_source_outputs), source_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weights), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weights), weight_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_sink_input), sink_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_sink_input), sink_bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_output(h_source_outputs, batch_size, memory_size, source_layer_size);
	source_output.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weights, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_input(h_sink_input, batch_size, memory_size, sink_layer_size);
	sink_input.setConstant(0);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeForwardPropogation(
		h_source_outputs,
		d_source_outputs,
		h_weights,
		d_weights,
		h_sink_input,
		d_sink_input,
		integration_function,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		source_time_steps,
		sink_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 3> expected_input(batch_size, memory_size, sink_layer_size);
	expected_input.setValues({ {{2}, {0}},
		{{4}, {0}},
		{{6}, {0}},
		{{8}, {0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < sink_layer_size; ++node_iter) {
				std::cout << "[Input] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << sink_input(batch_iter, memory_iter, node_iter) << std::endl;
				assert(sink_input(batch_iter, memory_iter, node_iter) == expected_input(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(hipHostFree(h_source_outputs) == hipSuccess);
	assert(hipFree(d_source_outputs) == hipSuccess);
	assert(hipHostFree(h_weights) == hipSuccess);
	assert(hipFree(d_weights) == hipSuccess);
	assert(hipHostFree(h_sink_input) == hipSuccess);
	assert(hipFree(d_sink_input) == hipSuccess);
}

void test_backwardPropogationGpuDevice()
{
	const int device_id = 0;
	GpuKernal<float> kernal;

	IntegrationErrorOp<float, Eigen::GpuDevice>* integration_function = new FullyConnectedSumErrorOp<float, Eigen::GpuDevice>();
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;
	const int source_time_step = 0;
	const int sink_time_step = 0;

	float* h_source_errors;
	float* d_source_errors;
	float* h_source_inputs;
	float* d_source_inputs;
	float* h_weights;
	float* d_weights;
	float* h_sink_error;
	float* d_sink_error;
	float* h_sink_output;
	float* d_sink_output;
	float* h_sink_derivative;
	float* d_sink_derivative;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_source_errors), source_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_source_errors), source_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_source_inputs), source_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_source_inputs), source_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weights), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weights), weight_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_sink_error), sink_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_sink_error), sink_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_sink_derivative), sink_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_sink_derivative), sink_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_sink_output), sink_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_sink_output), sink_bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_error(h_source_errors, batch_size, memory_size, source_layer_size);
	source_error.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_input(h_source_inputs, batch_size, memory_size, source_layer_size);
	source_input.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weights, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_derivative(h_sink_derivative, batch_size, memory_size, sink_layer_size);
	sink_derivative.setConstant(2);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_error(h_sink_error, batch_size, memory_size, sink_layer_size);
	sink_error.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_output(h_sink_output, batch_size, memory_size, sink_layer_size);
	sink_output.setConstant(1);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeBackwardPropogation(
		h_source_errors,
		d_source_errors,
		h_source_inputs,
		d_source_inputs,
		h_sink_output,
		d_sink_output,
		h_weights,
		d_weights,
		h_sink_error,
		d_sink_error,
		h_sink_derivative,
		d_sink_derivative,
		source_layer_size,
		integration_function,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		source_time_step,
		sink_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 3> expected_error(batch_size, memory_size, sink_layer_size);
	expected_error.setValues({ {{4}, {0}},
		{{8}, {0}},
		{{12}, {0}},
		{{16}, {0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < sink_layer_size; ++node_iter) {
				std::cout << "[Sink Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << sink_error(batch_iter, memory_iter, node_iter) << std::endl;
				assert(sink_error(batch_iter, memory_iter, node_iter) == expected_error(batch_iter, memory_iter, node_iter));
			}
		}
	}

	assert(hipHostFree(h_source_errors) == hipSuccess);
	assert(hipFree(d_source_errors) == hipSuccess);
	assert(hipHostFree(h_source_inputs) == hipSuccess);
	assert(hipFree(d_source_inputs) == hipSuccess);
	assert(hipHostFree(h_weights) == hipSuccess);
	assert(hipFree(d_weights) == hipSuccess);
	assert(hipHostFree(h_sink_error) == hipSuccess);
	assert(hipFree(d_sink_error) == hipSuccess);
	assert(hipHostFree(h_sink_derivative) == hipSuccess);
	assert(hipFree(d_sink_derivative) == hipSuccess);
	assert(hipHostFree(h_sink_output) == hipSuccess);
	assert(hipFree(d_sink_output) == hipSuccess);
}

void test_modelErrorGpuDevice()
{
	const int device_id = 0;
	GpuKernal<float> kernal;

	MSEOp<float, Eigen::GpuDevice>* loss_function = new MSEOp<float, Eigen::GpuDevice>;
	MSEGradOp<float, Eigen::GpuDevice>* loss_grad_function = new MSEGradOp<float, Eigen::GpuDevice>;
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int time_step = 0;

	float* h_predicted;
	float* d_predicted;
	float* h_node_errors;
	float* d_node_errors;
	float* h_model_error;
	float* d_model_error;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	std::size_t model_bytes = batch_size * memory_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_predicted), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_predicted), bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_node_errors), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_node_errors), bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_model_error), model_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_model_error), model_bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> predicted(h_predicted, batch_size, memory_size, layer_size);
	predicted.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> model_error(h_model_error, batch_size, memory_size);
	model_error.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_error(h_node_errors, batch_size, memory_size, layer_size);
	node_error.setConstant(0);
	Eigen::Tensor<float, 2> expected(batch_size, layer_size);
	expected.setConstant(1);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeModelErrors(
		expected,
		h_predicted,
		d_predicted,
		h_model_error,
		d_model_error,
		h_node_errors,
		d_node_errors,
		loss_function,
		loss_grad_function,
		batch_size,
		memory_size,
		layer_size,
		time_step,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 2> expected_model_error(batch_size, memory_size);
	expected_model_error.setValues({ {0, 0}, {0.5, 0}, {2.0, 0}, {4.5, 0} });
	Eigen::Tensor<float, 3> expected_node_error(batch_size, memory_size, layer_size);
	expected_node_error.setValues({
		{ {0, 0 }, { 0, 0 } },
		{ {-0.5, -0.5 }, { 0, 0 } },
		{ {-1, -1 }, { 0, 0 } },
		{ {-1.5, -1.5 }, { 0, 0 } } });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			std::cout << "[Model Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << " = " << model_error(batch_iter, memory_iter) << std::endl;
			assert(model_error(batch_iter, memory_iter) == expected_model_error(batch_iter, memory_iter));
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				std::cout << "[Node Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_error(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_error(batch_iter, memory_iter, node_iter) == expected_node_error(batch_iter, memory_iter, node_iter));
			}
		}
	}

	assert(hipHostFree(h_predicted) == hipSuccess);
	assert(hipFree(d_predicted) == hipSuccess);
	assert(hipHostFree(h_node_errors) == hipSuccess);
	assert(hipFree(d_node_errors) == hipSuccess);
	assert(hipHostFree(h_model_error) == hipSuccess);
	assert(hipFree(d_model_error) == hipSuccess);
}

void test_weightErrorGpuDevice()
{
	const int device_id = 0;
	GpuKernal<float> kernal;

	IntegrationWeightGradOp<float, Eigen::GpuDevice>* integration_function = new FullyConnectedSumWeightGradOp<float, Eigen::GpuDevice>();
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;

	float* h_sink_errors;
	float* d_sink_errors;
	float* h_source_outputs;
	float* d_source_outputs;
	float* h_source_inputs;
	float* d_source_inputs;
	float* h_weight;
	float* d_weight;
	float* h_weight_error;
	float* d_weight_error;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_sink_errors), sink_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_sink_errors), sink_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_source_outputs), source_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_source_outputs), source_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_source_inputs), source_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_source_inputs), source_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weight), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weight), weight_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weight_error), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weight_error), weight_bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_error(h_sink_errors, batch_size, memory_size, sink_layer_size);
	sink_error.setValues({ {{1}, {1}},
		{{2}, {1}},
		{{3}, {0}},
		{{4}, {0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_output(h_source_outputs, batch_size, memory_size, source_layer_size);
	source_output.setValues({ {{1, 1}, {1, 1}},
		{{2, 2}, {2, 2}},
		{{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_input(h_source_inputs, batch_size, memory_size, source_layer_size);
	source_input.setValues({ {{2, 2}, {0, 0}},
		{{4, 4}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{4, 4}, {0, 0}} });

	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weight, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight_error(h_weight_error, source_layer_size, sink_layer_size);
	weight_error.setConstant(0);

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeWeightErrors(
		h_sink_errors,
		d_sink_errors,
		h_source_outputs,
		d_source_outputs,
		h_source_inputs,
		d_source_inputs,
		source_layer_size,
		integration_function,
		h_weight,
		d_weight,
		h_weight_error,
		d_weight_error,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 2> expected_weight_error(source_layer_size, sink_layer_size);
	expected_weight_error.setValues({ {-4.75}, {-4.75} });

	for (int source_iter = 0; source_iter < source_layer_size; ++source_iter) {
		for (int sink_iter = 0; sink_iter < sink_layer_size; ++sink_iter) {
			std::cout << "[Weight Error] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << " = " << weight_error(source_iter, sink_iter) << std::endl;
			assert(weight_error(source_iter, sink_iter) == expected_weight_error(source_iter, sink_iter));
		}
	}

	assert(hipHostFree(h_sink_errors) == hipSuccess);
	assert(hipFree(d_sink_errors) == hipSuccess);
	assert(hipHostFree(h_source_outputs) == hipSuccess);
	assert(hipFree(d_source_outputs) == hipSuccess);
	assert(hipHostFree(h_source_inputs) == hipSuccess);
	assert(hipFree(d_source_inputs) == hipSuccess);
	assert(hipHostFree(h_weight) == hipSuccess);
	assert(hipFree(d_weight) == hipSuccess);
	assert(hipHostFree(h_weight_error) == hipSuccess);
	assert(hipFree(d_weight_error) == hipSuccess);
}

void test_weightUpdateGpuDevice(){
	const int device_id = 0;
	GpuKernal<float> kernal;

	SolverOp<float, Eigen::GpuDevice>* solver_function = new SGDOp<float, Eigen::GpuDevice>();
	const int source_layer_size = 2;
	const int sink_layer_size = 1;

	float* h_solver_params = new float[source_layer_size * sink_layer_size * 3];
	float* d_solver_params = new float[source_layer_size * sink_layer_size * 3];
	float* h_weight;
	float* d_weight;
	float* h_weight_error;
	float* d_weight_error;

	assert(hipSetDevice(device_id) == hipSuccess); // is this needed?

	// allocate memory
	std::size_t solver_bytes = source_layer_size * sink_layer_size * 3 * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(hipHostAlloc((void**)(&h_solver_params), solver_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_solver_params), solver_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weight), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weight), weight_bytes) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_weight_error), weight_bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_weight_error), weight_bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> solver_params(h_solver_params, source_layer_size, sink_layer_size, 3);
	solver_params.setValues({ {{0.01, 0.99, 0.0}},
		{{0.01, 0.99, 0.0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weight, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight_error(h_weight_error, source_layer_size, sink_layer_size);
	weight_error.setValues({ {-0.2},	{-20} });

	// Set up the device
	hipStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) == hipSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeWeightUpdate(
		h_weight,
		d_weight,
		h_solver_params,
		d_solver_params,
		h_weight_error,
		d_weight_error,
		solver_function,
		source_layer_size,
		sink_layer_size,
		device,
		true,
		true);

	// Synchronize the stream
	hipError_t err = hipStreamQuery(stream);
	assert(hipStreamSynchronize(stream) == hipSuccess);
	assert(hipStreamDestroy(stream) == hipSuccess);

	Eigen::Tensor<float, 2> expected_weights(source_layer_size, sink_layer_size);
	expected_weights.setValues({ {1.00398}, {1.398} });

	Eigen::Tensor<float, 3> expected_params(source_layer_size, sink_layer_size, 3);
	expected_params.setValues({ {{0.01, 0.99, 0.002}},
		{{0.01, 0.99, 0.2}} });

	for (int source_iter = 0; source_iter < source_layer_size; ++source_iter) {
		for (int sink_iter = 0; sink_iter < sink_layer_size; ++sink_iter) {
			std::cout << "[Weight] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << " = " << weight(source_iter, sink_iter) << std::endl;
			assert(weight(source_iter, sink_iter) == expected_weights(source_iter, sink_iter));
			for (int param_iter = 0; param_iter < 3; ++param_iter) {
				std::cout << "[Params] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << ", Param Iter: " << param_iter << " = " << solver_params(source_iter, sink_iter, param_iter) << std::endl;
				assert(solver_params(source_iter, sink_iter, param_iter) == expected_params(source_iter, sink_iter, param_iter));
			}
		}
	}

	assert(hipHostFree(h_solver_params) == hipSuccess);
	assert(hipFree(d_solver_params) == hipSuccess);
	assert(hipHostFree(h_weight) == hipSuccess);
	assert(hipFree(d_weight) == hipSuccess);
	assert(hipHostFree(h_weight_error) == hipSuccess);
	assert(hipFree(d_weight_error) == hipSuccess);

}

int main(int argc, char** argv)
{
	test_nodeActivationGpuDevice();
	test_nodeDerivativeGpuDevice();
	test_forwardPropogationGpuDevice();
	test_backwardPropogationGpuDevice();
	test_modelErrorGpuDevice();
	test_weightErrorGpuDevice();
	test_weightUpdateGpuDevice();
	return 0;
}
#endif