/**TODO:  Add copyright*/

#ifndef EVONET_CUDA

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <SmartPeak/core/KernalManager.h>

using namespace SmartPeak;
using namespace std;

void test_exampleGpu1() {
	GpuKernal kernal(0, 1);

	std::size_t bytes = 2 * sizeof(float);

	float* h_in1;
	float* h_in2;
	float* h_out;

	float* d_in1;
	float* d_in2;
	float* d_out;

	assert(hipSetDevice(kernal.getDeviceID()) == hipSuccess); // is this needed?

	// allocate memory
	assert(hipHostAlloc((void**)(&h_in1), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_in2), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipHostAlloc((void**)(&h_out), bytes, hipHostMallocDefault) == hipSuccess);
	assert(hipMalloc((void**)(&d_in1), bytes) == hipSuccess);
	assert(hipMalloc((void**)(&d_in2), bytes) == hipSuccess);
	assert(hipMalloc((void**)(&d_out), bytes) == hipSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 1>> in1(h_in1, 2);
	Eigen::TensorMap<Eigen::Tensor<float, 1>> in2(h_in2, 2);
	in1.setConstant(10.0f);
	in2.setConstant(10.0f);

	kernal.initKernal();
	hipStream_t stream = kernal.getStream();
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	device.memcpyHostToDevice(d_in1, in1.data(), bytes);
	device.memcpyHostToDevice(d_in2, in2.data(), bytes);

	auto test = [&](float* d_in1, float* d_in2, float* d_out, Eigen::GpuDevice& device) {
		Eigen::TensorMap<Eigen::Tensor<float, 1>> gpu_in1(d_in1, 2);
		Eigen::TensorMap<Eigen::Tensor<float, 1>> gpu_in2(d_in2, 2);
		Eigen::TensorMap<Eigen::Tensor<float, 1>> gpu_out(d_out, 2);

		gpu_out.device(device) = gpu_in1 + gpu_in2;
	};

	test(d_in1, d_in2, d_out, device);

	device.memcpyDeviceToHost(h_out, d_out, bytes);

	kernal.syncKernal();
	Eigen::TensorMap<Eigen::Tensor<float, 1>> out(h_out, 2);
	assert(out(0) == 20.0f);
	assert(out(1) == 20.0f);

	assert(hipFree(d_in1) == hipSuccess);
	assert(hipFree(d_in2) == hipSuccess);
	assert(hipFree(d_out) == hipSuccess);

	assert(hipHostFree(h_in1) == hipSuccess);
	assert(hipHostFree(h_in2) == hipSuccess);
	assert(hipHostFree(h_out) == hipSuccess);

	kernal.destroyKernal();
}

int main(int argc, char** argv)
{
	test_exampleGpu1();
	return 0;
}
#endif