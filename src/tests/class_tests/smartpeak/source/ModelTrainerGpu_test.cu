
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/
#if COMPILE_WITH_CUDA

#include <SmartPeak/ml/ModelTrainerGpu.h>

using namespace SmartPeak;
using namespace std;
template<typename TensorT>

class DataSimulatorDAGToy : public DataSimulator<TensorT> {
public:
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override {
    // Make the input data
    input_data.setValues({ {{1, 5, 1, 1}}, {{2, 6, 1, 1}}, {{3, 7, 1, 1}}, {{4, 8, 1, 1}} });

    // Make the output data
    loss_output_data.setValues({ {{0, 1}}, {{0, 1}}, {{0, 1}}, {{0, 1}} });
    metric_output_data.setValues({ {{0, 1}}, {{0, 1}}, {{0, 1}}, {{0, 1}} });

    // Make the simulation time_steps
    time_steps.setConstant(1);
  };
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override {
    // Make the input data
    input_data.setValues({ {{1, 1, 5, 1}}, {{1, 1, 2, 6}}, {{1, 1, 3, 7}}, {{1, 1, 4, 8 }} });

    // Make the output data
    loss_output_data.setValues({ {{1, 0}}, {{1, 0}}, {{1, 0}}, {{1, 0}} });
    metric_output_data.setValues({ {{0, 1}}, {{0, 1}}, {{0, 1}}, {{0, 1}} });

    // Make the simulation time_steps
    time_steps.setConstant(1);
  };
};

template<typename TensorT>
class DataSimulatorDCGToy : public DataSimulator<TensorT> {
public:
  void simulateTrainingData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override {
    // Make the input data
    input_data.setValues(
      { {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
      {{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
      {{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
      {{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
      {{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
    );

    // Make the output data
    loss_output_data.setValues(
      { { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
      { { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
      { { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
      { { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
      { { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });
    metric_output_data.setValues(
      { { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
      { { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
      { { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
      { { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
      { { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });

    // Make the simulation time_steps
    time_steps.setValues({
      {1, 1, 1, 1, 1, 1, 1, 1},
      {1, 1, 1, 1, 1, 1, 1, 1},
      {1, 1, 1, 1, 1, 1, 1, 1},
      {1, 1, 1, 1, 1, 1, 1, 1},
      {1, 1, 1, 1, 1, 1, 1, 1} }
    );
  }
  void simulateValidationData(Eigen::Tensor<TensorT, 3>& input_data, Eigen::Tensor<TensorT, 3>& loss_output_data, Eigen::Tensor<TensorT, 3>& metric_output_data, Eigen::Tensor<TensorT, 2>& time_steps) override {
    simulateTrainingData(input_data, loss_output_data, metric_output_data, time_steps);
  }
};

template<typename TensorT>
class DAGToyModelTrainer : public ModelTrainerGpu<TensorT> {
public:
	Model<TensorT> makeModel() override
	{
		// CopyNPasted from Model_DAG_Test
		Node<TensorT> i1, i2, h1, h2, o1, o2, b1, b2;
		Link l1, l2, l3, l4, lb1, lb2, l5, l6, l7, l8, lb3, lb4;
		Weight<TensorT> w1, w2, w3, w4, wb1, wb2, w5, w6, w7, w8, wb3, wb4;
		Model<TensorT> model1;

		// Toy network: 1 hidden layer, fully connected, DAG
		i1 = Node<TensorT>("0", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		i2 = Node<TensorT>("1", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		h1 = Node<TensorT>("2", NodeType::hidden, NodeStatus::initialized, std::make_shared<ReLUOp<float>>(ReLUOp<float>()), std::make_shared<ReLUGradOp<float>>(ReLUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		h2 = Node<TensorT>("3", NodeType::hidden, NodeStatus::initialized, std::make_shared<ReLUOp<float>>(ReLUOp<float>()), std::make_shared<ReLUGradOp<float>>(ReLUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		o1 = Node<TensorT>("4", NodeType::output, NodeStatus::initialized, std::make_shared<ReLUOp<float>>(ReLUOp<float>()), std::make_shared<ReLUGradOp<float>>(ReLUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		o2 = Node<TensorT>("5", NodeType::output, NodeStatus::initialized, std::make_shared<ReLUOp<float>>(ReLUOp<float>()), std::make_shared<ReLUGradOp<float>>(ReLUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		b1 = Node<TensorT>("6", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		b2 = Node<TensorT>("7", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));

		// weights  
		std::shared_ptr<WeightInitOp<TensorT>> weight_init;
		std::shared_ptr<SolverOp<TensorT>> solver;
		// weight_init.reset(new RandWeightInitOp(1.0)); // No random init for testing
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w1 = Weight<TensorT>("0", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w2 = Weight<TensorT>("1", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w3 = Weight<TensorT>("2", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w4 = Weight<TensorT>("3", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		wb1 = Weight<TensorT>("4", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		wb2 = Weight<TensorT>("5", weight_init, solver);
		// input layer + bias
		l1 = Link("0", "0", "2", "0");
		l2 = Link("1", "0", "3", "1");
		l3 = Link("2", "1", "2", "2");
		l4 = Link("3", "1", "3", "3");
		lb1 = Link("4", "6", "2", "4");
		lb2 = Link("5", "6", "3", "5");
		// weights
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w5 = Weight<TensorT>("6", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w6 = Weight<TensorT>("7", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w7 = Weight<TensorT>("8", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		w8 = Weight<TensorT>("9", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		wb3 = Weight<TensorT>("10", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
		wb4 = Weight<TensorT>("11", weight_init, solver);
		// hidden layer + bias
		l5 = Link("6", "2", "4", "6");
		l6 = Link("7", "2", "5", "7");
		l7 = Link("8", "3", "4", "8");
		l8 = Link("9", "3", "5", "9");
		lb3 = Link("10", "7", "4", "10");
		lb4 = Link("11", "7", "5", "11");
		model1.setId(1);
		model1.addNodes({ i1, i2, h1, h2, o1, o2, b1, b2 });
		model1.addWeights({ w1, w2, w3, w4, wb1, wb2, w5, w6, w7, w8, wb3, wb4 });
		model1.addLinks({ l1, l2, l3, l4, lb1, lb2, l5, l6, l7, l8, lb3, lb4 });
    model1.setInputAndOutputNodes();
		return model1;
	}
	void adaptiveTrainerScheduler (
		const int& n_generations,
		const int& n_epochs,
		Model<TensorT>& model,
		ModelInterpreterGpu<TensorT>& model_interpreter,
		const std::vector<TensorT>& model_errors) override {}
};

void test_DAGToy() 
{
	std::cout << "Model DAG training test..." << std::endl;
  // Define the makeModel and trainModel scripts
  DAGToyModelTrainer<float> trainer;

	// Define the model resources
	ModelResources model_resources = { ModelDevice(0, 1) };

  // Test parameters
  trainer.setBatchSize(4);
  trainer.setMemorySize(1);
  trainer.setNEpochsTraining(20);
	trainer.setNEpochsValidation(20);
	trainer.setLogging(false, false);
  const std::vector<std::string> input_nodes = {"0", "1", "6", "7"}; // true inputs + biases
  const std::vector<std::string> output_nodes = {"4", "5"};
	trainer.setLossFunctions({ std::make_shared<MSELossOp<float>>(MSELossOp<float>()) });
	trainer.setLossFunctionGrads({ std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>()) });
	trainer.setLossOutputNodes({ output_nodes });

  // Make the input data
  Eigen::Tensor<float, 4> input_data(trainer.getBatchSize(), trainer.getMemorySize(), (int)input_nodes.size(), trainer.getNEpochsTraining());
  Eigen::Tensor<float, 3> input_tmp(trainer.getBatchSize(), trainer.getMemorySize(), (int)input_nodes.size()); 
  input_tmp.setValues({{{1, 5, 1, 1}}, {{2, 6, 1, 1}}, {{3, 7, 1, 1}}, {{4, 8, 1, 1}}});
  for (int batch_iter=0; batch_iter<trainer.getBatchSize(); ++batch_iter)
    for (int memory_iter=0; memory_iter<trainer.getMemorySize(); ++memory_iter)
      for (int nodes_iter=0; nodes_iter<(int)input_nodes.size(); ++nodes_iter)
        for (int epochs_iter=0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter)
          input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = input_tmp(batch_iter, memory_iter, nodes_iter);
  
  // Make the output data
  Eigen::Tensor<float, 4> output_data(trainer.getBatchSize(), trainer.getMemorySize(), (int)output_nodes.size(), trainer.getNEpochsTraining());
  Eigen::Tensor<float, 2> output_tmp(trainer.getBatchSize(), (int)output_nodes.size());
  output_tmp.setValues({{0, 1}, {0, 1}, {0, 1}, {0, 1}});
  for (int batch_iter=0; batch_iter<trainer.getBatchSize(); ++batch_iter){
		for (int memory_iter = 0; memory_iter<trainer.getMemorySize(); ++memory_iter){
			for (int nodes_iter=0; nodes_iter<(int)output_nodes.size(); ++nodes_iter){
				for (int epochs_iter=0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter){
					if (memory_iter == 0)
						output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = output_tmp(batch_iter, nodes_iter);
					else
						output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = 0.0;
				}
			}
		}
	}

  // Make the simulation time_steps
	Eigen::Tensor<float, 3> time_steps(trainer.getBatchSize(), trainer.getMemorySize(), trainer.getNEpochsTraining());
	Eigen::Tensor<float, 2> time_steps_tmp(trainer.getBatchSize(), trainer.getMemorySize());
	time_steps_tmp.setValues({
		{ 1 },
		{ 1 },
		{ 1 },
		{ 1 }}
	);
	for (int batch_iter = 0; batch_iter<trainer.getBatchSize(); ++batch_iter)
		for (int memory_iter = 0; memory_iter<trainer.getMemorySize(); ++memory_iter)
			for (int epochs_iter = 0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter)
				time_steps(batch_iter, memory_iter, epochs_iter) = time_steps_tmp(batch_iter, memory_iter);

  Model<float> model1 = trainer.makeModel();
  trainer.trainModel(model1, input_data, output_data, time_steps,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error = model1.getError().sum();
  assert(total_error(0) <= 757.0);

  std::vector<float> validation_errors = trainer.validateModel(model1, input_data, output_data, time_steps,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error2 = model1.getError().sum();
  assert(total_error2(0) <= 757.0);
  assert(validation_errors[0] <= 757.0);

	// TODO evaluateModel
}

void test_DAGToy2()
{
  // Define the makeModel and trainModel scripts
  DAGToyModelTrainer<float> trainer;

  // Define the model resources
  ModelResources model_resources = { ModelDevice(0, 1) };

  // Test parameters
  trainer.setBatchSize(4);
  trainer.setMemorySize(1);
  trainer.setNEpochsTraining(20);
  trainer.setNEpochsValidation(20);
  trainer.setLogging(false, false);
  const std::vector<std::string> input_nodes = { "0", "1", "6", "7" }; // true inputs + biases
  const std::vector<std::string> output_nodes = { "4", "5" };
  trainer.setLossFunctions({ std::make_shared<MSELossOp<float>>(MSELossOp<float>()) });
  trainer.setLossFunctionGrads({ std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>()) });
  trainer.setLossOutputNodes({ output_nodes });
  trainer.setMetricFunctions({ std::make_shared<MAEOp<float>>(MAEOp<float>()) });
  trainer.setMetricOutputNodes({ output_nodes });
  trainer.setMetricNames({ "MAE" });

  DataSimulatorDAGToy<float> data_simulator;

  Model<float> model1 = trainer.makeModel();
  std::pair<std::vector<float>, std::vector<float>> errors = trainer.trainModel(model1, data_simulator,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error = model1.getError().sum();
  assert(total_error(0) <= 757.0);
  assert(errors.first.back() <= 757.0);
  assert(errors.second.back() <= 486.0);

  std::pair<std::vector<float>, std::vector<float>> validation_errors = trainer.validateModel(model1, data_simulator,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error_validation = model1.getError().sum();
  assert(total_error_validation(0) <= 749.843);
  assert(validation_errors.first.back() <= 749.843);
  assert(validation_errors.second.back() <= 455.844);

  // TODO evaluateModel
}

template<typename TensorT>
class DCGToyModelTrainer : public ModelTrainerGpu<TensorT>
{
public:
	Model<TensorT> makeModel() override
	{
		// CopyNPasted from Model_DCG_Test
		Node<TensorT> i1, h1, o1, b1, b2;
		Link l1, l2, l3, lb1, lb2;
		Weight<TensorT> w1, w2, w3, wb1, wb2;
		Model<TensorT> model2;
		// Toy network: 1 hidden layer, fully connected, DCG
		i1 = Node<TensorT>("0", NodeType::input, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		h1 = Node<TensorT>("1", NodeType::hidden, NodeStatus::initialized, std::make_shared<ELUOp<float>>(ELUOp<float>()), std::make_shared<ELUGradOp<float>>(ELUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		o1 = Node<TensorT>("2", NodeType::output, NodeStatus::initialized, std::make_shared<ELUOp<float>>(ELUOp<float>()), std::make_shared<ELUGradOp<float>>(ELUGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		b1 = Node<TensorT>("3", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		b2 = Node<TensorT>("4", NodeType::bias, NodeStatus::activated, std::make_shared<LinearOp<float>>(LinearOp<float>()), std::make_shared<LinearGradOp<float>>(LinearGradOp<float>()), std::make_shared<SumOp<float>>(SumOp<float>()), std::make_shared<SumErrorOp<float>>(SumErrorOp<float>()), std::make_shared<SumWeightGradOp<float>>(SumWeightGradOp<float>()));
		// weights  
		std::shared_ptr<WeightInitOp<TensorT>> weight_init;
		std::shared_ptr<SolverOp<TensorT>> solver;
		// weight_init.reset(new RandWeightInitOp(1.0)); // No random init for testing
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<TensorT>(0.01, 0.9, 0.999, 1e-8));
		w1 = Weight<TensorT>("0", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<TensorT>(0.01, 0.9, 0.999, 1e-8));
		w2 = Weight<TensorT>("1", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<TensorT>(0.01, 0.9, 0.999, 1e-8));
		w3 = Weight<TensorT>("2", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<TensorT>(0.01, 0.9, 0.999, 1e-8));
		wb1 = Weight<TensorT>("3", weight_init, solver);
		weight_init = std::make_shared<ConstWeightInitOp<TensorT>>(ConstWeightInitOp<TensorT>(1.0));
		solver.reset(new AdamOp<TensorT>(0.01, 0.9, 0.999, 1e-8));
		wb2 = Weight<TensorT>("4", weight_init, solver);
		weight_init.reset();
		solver.reset();
		// links
		l1 = Link("0", "0", "1", "0");
		l2 = Link("1", "1", "2", "1");
		l3 = Link("2", "1", "1", "2");
		lb1 = Link("3", "3", "1", "3");
		lb2 = Link("4", "4", "2", "4");
		model2.setId(2);
		model2.addNodes({ i1, h1, o1, b1, b2 });
		model2.addWeights({ w1, w2, w3, wb1, wb2 });
		model2.addLinks({ l1, l2, l3, lb1, lb2 });
    model2.setInputAndOutputNodes();
		return model2;
	}
	void adaptiveTrainerScheduler (
		const int& n_generations,
		const int& n_epochs,
		Model<TensorT>& model,
		ModelInterpreterGpu<TensorT>& model_interpreter,
		const std::vector<TensorT>& model_errors) override {}
};

void test_DCGToy() 
{
	std::cout << "Model DCG training test..." << std::endl;
  // Define the makeModel and trainModel scripts
  DCGToyModelTrainer<float> trainer;

	// Define the model resources
	ModelResources model_resources = { ModelDevice(0, 1) };

  // Test parameters
  trainer.setBatchSize(5);
  trainer.setMemorySize(8);
  trainer.setNEpochsTraining(50);
	trainer.setNEpochsValidation(50);
  const std::vector<std::string> input_nodes = {"0", "3", "4"}; // true inputs + biases
  const std::vector<std::string> output_nodes = {"2"};
	trainer.setLossFunctions({ std::make_shared<MSELossOp<float>>(MSELossOp<float>()) });
	trainer.setLossFunctionGrads({ std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>()) });
	trainer.setLossOutputNodes({ output_nodes });

  // Make the input data
  Eigen::Tensor<float, 4> input_data(trainer.getBatchSize(), trainer.getMemorySize(), (int)input_nodes.size(), trainer.getNEpochsTraining());
  Eigen::Tensor<float, 3> input_tmp(trainer.getBatchSize(), trainer.getMemorySize(), (int)input_nodes.size()); 
  input_tmp.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
  );
  for (int batch_iter=0; batch_iter<trainer.getBatchSize(); ++batch_iter)
    for (int memory_iter=0; memory_iter<trainer.getMemorySize(); ++memory_iter)
      for (int nodes_iter=0; nodes_iter<(int)input_nodes.size(); ++nodes_iter)
        for (int epochs_iter=0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter)
          input_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = input_tmp(batch_iter, memory_iter, nodes_iter);
  
  // Make the output data
  Eigen::Tensor<float, 4> output_data(trainer.getBatchSize(), trainer.getMemorySize(), (int)output_nodes.size(), trainer.getNEpochsTraining());
  Eigen::Tensor<float, 3> output_tmp(trainer.getBatchSize(), trainer.getMemorySize(), (int)output_nodes.size()); 
  output_tmp.setValues(
		{ { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
		{ { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
		{ { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
		{ { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
		{ { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });
  for (int batch_iter=0; batch_iter<trainer.getBatchSize(); ++batch_iter)
		for (int memory_iter = 0; memory_iter<trainer.getMemorySize(); ++memory_iter)
			for (int nodes_iter=0; nodes_iter<(int)output_nodes.size(); ++nodes_iter)
				for (int epochs_iter=0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter)
					output_data(batch_iter, memory_iter, nodes_iter, epochs_iter) = output_tmp(batch_iter, memory_iter, nodes_iter);

  // Make the simulation time_steps
  Eigen::Tensor<float, 3> time_steps(trainer.getBatchSize(), trainer.getMemorySize(), trainer.getNEpochsTraining());
  Eigen::Tensor<float, 2> time_steps_tmp(trainer.getBatchSize(), trainer.getMemorySize()); 
  time_steps_tmp.setValues({
    {1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1},
    {1, 1, 1, 1, 1, 1, 1, 1}}
  );
  for (int batch_iter=0; batch_iter<trainer.getBatchSize(); ++batch_iter)
    for (int memory_iter=0; memory_iter<trainer.getMemorySize(); ++memory_iter)
      for (int epochs_iter=0; epochs_iter<trainer.getNEpochsTraining(); ++epochs_iter)
        time_steps(batch_iter, memory_iter, epochs_iter) = time_steps_tmp(batch_iter, memory_iter);

  Model<float> model1 = trainer.makeModel();

  trainer.trainModel(model1, input_data, output_data, time_steps,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error = model1.getError().sum();
  assert(total_error(0) <= 1492.6);

  std::vector<float> validation_errors = trainer.validateModel(model1, input_data, output_data, time_steps,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error2 = model1.getError().sum();
  assert(total_error2(0) <= 1492.6);
  assert(validation_errors[0] <= 1492.6);
  // TODO evaluateModel
}

void test_DCGToy2()
{
  // Define the makeModel and trainModel scripts
  DCGToyModelTrainer<float> trainer;

  // Define the model resources
  ModelResources model_resources = { ModelDevice(0, 1) };

  // Test parameters
  trainer.setBatchSize(5);
  trainer.setMemorySize(8);
  trainer.setNEpochsTraining(50);
  trainer.setNEpochsValidation(50);
  const std::vector<std::string> input_nodes = { "0", "3", "4" }; // true inputs + biases
  const std::vector<std::string> output_nodes = { "2" };
  trainer.setLossFunctions({ std::make_shared<MSELossOp<float>>(MSELossOp<float>()) });
  trainer.setLossFunctionGrads({ std::make_shared<MSELossGradOp<float>>(MSELossGradOp<float>()) });
  trainer.setLossOutputNodes({ output_nodes });
  trainer.setMetricFunctions({ std::make_shared<MAEOp<float>>(MAEOp<float>()) });
  trainer.setMetricOutputNodes({ output_nodes });
  trainer.setMetricNames({ "MAE" });

  // Make data simulator
  DataSimulatorDCGToy<float> data_simulator;

  Model<float> model1 = trainer.makeModel();

  std::pair<std::vector<float>, std::vector<float>> errors = trainer.trainModel(model1, data_simulator,
    input_nodes, ModelLogger<float>(), ModelInterpreterGpu<float>(model_resources));

  const Eigen::Tensor<float, 0> total_error2 = model1.getError().sum();
  assert(total_error2(0) <= 1492.6);
  assert(errors.first.back() <= 1492.6);
  assert(errors.second.back() <= 1492.6);
  // TODO evaluateModel
}

int main(int argc, char** argv)
{
	//test_DAGToy();
  test_DAGToy2();
	//test_DCGToy();
  test_DCGToy2();
	return 0;
}

#endif