
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <cuda.h>
#include <cuda_runtime.h>

#include <EvoNet/ml/ModelKernalGpu.h>

using namespace EvoNet;
using namespace std;

void test_constructorGpuDevice()
{
	ModelKernalGpu<float>* ptr = nullptr;
	ModelKernalGpu<float>* nullPointer = nullptr;
	ptr = new ModelKernalGpu<float>();
	assert(ptr != nullPointer);
}

void test_destructorGpuDevice()
{
	ModelKernalGpu<float>* ptr = nullptr;
	ptr = new ModelKernalGpu<float>();
	delete ptr;
}

void test_nodeActivationGpuDevice()
{
	ModelKernalGpu<float> kernal;
	const int device_id = 0;

	std::shared_ptr<ActivationTensorOp<float, Eigen::GpuDevice>> activation_function = std::make_shared<ReLUTensorOp<float, Eigen::GpuDevice>>(ReLUTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int source_time_step = 0;
	const int node_time_step = 0;

	float* h_node_input;
	float* d_node_input;
	float* h_node_output;
	float* d_node_output;
	float* h_node_dt;
	float* d_node_dt;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_node_input), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_input), bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_node_output), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_output), bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_node_dt), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_dt), bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_input(h_node_input, batch_size, memory_size, layer_size);
	node_input.setValues({ {{-1, 1}, {0, 0}},
		{{-2, 2}, {0, 0}},
		{{-3, 3}, {0, 0}},
		{{-4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_output(h_node_output, batch_size, memory_size, layer_size);
	node_output.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_dt(h_node_dt, batch_size, memory_size, layer_size);
	node_dt.setConstant(1);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeNodeActivation(
		h_node_input,
		d_node_input,
		h_node_output,
		d_node_output,
		h_node_dt,
		d_node_dt,
		activation_function,
		batch_size,
		memory_size,
		layer_size,
		node_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 3> expected_output(batch_size, memory_size, layer_size);
	expected_output.setValues({ {{0, 1}, {0, 0}},
		{{0, 2}, {0, 0}},
		{{0, 3}, {0, 0}},
		{{0, 4}, {0, 0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				//std::cout << "[Output] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_output(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_output(batch_iter, memory_iter, node_iter) == expected_output(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(cudaFreeHost(h_node_input) == cudaSuccess);
	assert(cudaFree(d_node_input) == cudaSuccess);
	assert(cudaFreeHost(h_node_output) == cudaSuccess);
	assert(cudaFree(d_node_output) == cudaSuccess);
	assert(cudaFreeHost(h_node_dt) == cudaSuccess);
	assert(cudaFree(d_node_dt) == cudaSuccess);
}

void test_nodeDerivativeGpuDevice()
{
	ModelKernalGpu<float> kernal;
	const int device_id = 0;

	std::shared_ptr<ActivationTensorOp<float, Eigen::GpuDevice>> activation_grad_function = std::make_shared<ReLUGradTensorOp<float, Eigen::GpuDevice>>(ReLUGradTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int source_time_step = 0;
	const int node_time_step = 0;

	float* h_node_output;
	float* d_node_output;
	float* h_node_derivative;
	float* d_node_derivative;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_node_output), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_output), bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_node_derivative), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_derivative), bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_output(h_node_output, batch_size, memory_size, layer_size);
	node_output.setValues({ {{-1, 1}, {0, 0}},
		{{-2, 2}, {0, 0}},
		{{-3, 3}, {0, 0}},
		{{-4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_derivative(h_node_derivative, batch_size, memory_size, layer_size);
	node_derivative.setConstant(0);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeNodeDerivative(
		h_node_output,
		d_node_output,
		h_node_derivative,
		d_node_derivative,
		activation_grad_function,
		batch_size,
		memory_size,
		layer_size,
		node_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 3> expected_derivative(batch_size, memory_size, layer_size);
	expected_derivative.setValues({ {{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}},
		{{0, 1}, {0, 0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				//std::cout << "[Derivative] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_derivative(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_derivative(batch_iter, memory_iter, node_iter) == expected_derivative(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(cudaFreeHost(h_node_output) == cudaSuccess);
	assert(cudaFree(d_node_output) == cudaSuccess);
	assert(cudaFreeHost(h_node_derivative) == cudaSuccess);
	assert(cudaFree(d_node_derivative) == cudaSuccess);
}

void test_forwardPropogationGpuDevice()
{
	ModelKernalGpu<float> kernal;
	const int device_id = 0;

	std::shared_ptr<IntegrationTensorOp<float, Eigen::GpuDevice>> integration_function = std::make_shared<SumTensorOp<float, Eigen::GpuDevice>>(SumTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;
	const int source_time_steps = 0;
	const int sink_time_step = 0;

	float* h_source_outputs;
	float* d_source_outputs;
	float* h_weights;
	float* d_weights;
	float* h_sink_input;
	float* d_sink_input;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_source_outputs), source_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_source_outputs), source_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weights), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weights), weight_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_sink_input), sink_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_sink_input), sink_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_output(h_source_outputs, batch_size, memory_size, source_layer_size);
	source_output.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weights, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_input(h_sink_input, batch_size, memory_size, sink_layer_size);
	sink_input.setConstant(0);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeForwardPropogation(
		h_source_outputs,
		d_source_outputs,
		h_weights,
		d_weights,
		h_sink_input,
		d_sink_input,
		integration_function,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		source_time_steps,
		sink_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 3> expected_input(batch_size, memory_size, sink_layer_size);
	expected_input.setValues({ {{2}, {0}},
		{{4}, {0}},
		{{6}, {0}},
		{{8}, {0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < sink_layer_size; ++node_iter) {
				//std::cout << "[Input] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << sink_input(batch_iter, memory_iter, node_iter) << std::endl;
				assert(sink_input(batch_iter, memory_iter, node_iter) == expected_input(batch_iter, memory_iter, node_iter));
			}
		}
	}

	// release resources
	assert(cudaFreeHost(h_source_outputs) == cudaSuccess);
	assert(cudaFree(d_source_outputs) == cudaSuccess);
	assert(cudaFreeHost(h_weights) == cudaSuccess);
	assert(cudaFree(d_weights) == cudaSuccess);
	assert(cudaFreeHost(h_sink_input) == cudaSuccess);
	assert(cudaFree(d_sink_input) == cudaSuccess);
}

void test_backwardPropogationGpuDevice()
{
	const int device_id = 0;
	ModelKernalGpu<float> kernal;

	std::shared_ptr<IntegrationErrorTensorOp<float, Eigen::GpuDevice>> integration_function = std::make_shared<SumErrorTensorOp<float, Eigen::GpuDevice>>(SumErrorTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;
	const int source_time_step = 0;
	const int sink_time_step = 0;

	float* h_source_errors;
	float* d_source_errors;
	float* h_source_inputs;
	float* d_source_inputs;
	float* h_weights;
	float* d_weights;
	float* h_sink_error;
	float* d_sink_error;
	float* h_sink_output;
	float* d_sink_output;
	float* h_sink_derivative;
	float* d_sink_derivative;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_source_errors), source_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_source_errors), source_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_source_inputs), source_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_source_inputs), source_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weights), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weights), weight_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_sink_error), sink_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_sink_error), sink_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_sink_derivative), sink_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_sink_derivative), sink_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_sink_output), sink_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_sink_output), sink_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_error(h_source_errors, batch_size, memory_size, source_layer_size);
	source_error.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_input(h_source_inputs, batch_size, memory_size, source_layer_size);
	source_input.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weights, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_derivative(h_sink_derivative, batch_size, memory_size, sink_layer_size);
	sink_derivative.setConstant(2);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_error(h_sink_error, batch_size, memory_size, sink_layer_size);
	sink_error.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_output(h_sink_output, batch_size, memory_size, sink_layer_size);
	sink_output.setConstant(1);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeBackwardPropogation(
		h_source_errors,
		d_source_errors,
		h_source_inputs,
		d_source_inputs,
		h_sink_output,
		d_sink_output,
		h_weights,
		d_weights,
		h_sink_error,
		d_sink_error,
		h_sink_derivative,
		d_sink_derivative,
		source_layer_size,
		integration_function,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		source_time_step,
		sink_time_step,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 3> expected_error(batch_size, memory_size, sink_layer_size);
	expected_error.setValues({ {{4}, {0}},
		{{8}, {0}},
		{{12}, {0}},
		{{16}, {0}} });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			for (int node_iter = 0; node_iter < sink_layer_size; ++node_iter) {
				//std::cout << "[Sink Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << sink_error(batch_iter, memory_iter, node_iter) << std::endl;
				assert(sink_error(batch_iter, memory_iter, node_iter) == expected_error(batch_iter, memory_iter, node_iter));
			}
		}
	}

	assert(cudaFreeHost(h_source_errors) == cudaSuccess);
	assert(cudaFree(d_source_errors) == cudaSuccess);
	assert(cudaFreeHost(h_source_inputs) == cudaSuccess);
	assert(cudaFree(d_source_inputs) == cudaSuccess);
	assert(cudaFreeHost(h_weights) == cudaSuccess);
	assert(cudaFree(d_weights) == cudaSuccess);
	assert(cudaFreeHost(h_sink_error) == cudaSuccess);
	assert(cudaFree(d_sink_error) == cudaSuccess);
	assert(cudaFreeHost(h_sink_derivative) == cudaSuccess);
	assert(cudaFree(d_sink_derivative) == cudaSuccess);
	assert(cudaFreeHost(h_sink_output) == cudaSuccess);
	assert(cudaFree(d_sink_output) == cudaSuccess);
}

void test_modelErrorGpuDevice()
{
	const int device_id = 0;
	ModelKernalGpu<float> kernal;

	std::shared_ptr<LossFunctionTensorOp<float, Eigen::GpuDevice>> loss_function = std::make_shared<MSELossTensorOp<float, Eigen::GpuDevice>>(MSELossTensorOp<float, Eigen::GpuDevice>());
	std::shared_ptr<LossFunctionGradTensorOp<float, Eigen::GpuDevice>> loss_grad_function = std::make_shared<MSELossGradTensorOp<float, Eigen::GpuDevice>>(MSELossGradTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int layer_size = 2;
	const int time_step = 0;

	float* h_predicted;
	float* d_predicted;
	float* h_node_errors;
	float* d_node_errors;
	float* h_model_error;
	float* d_model_error;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
	std::size_t model_bytes = batch_size * memory_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_predicted), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_predicted), bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_node_errors), bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_node_errors), bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_model_error), model_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_model_error), model_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> predicted(h_predicted, batch_size, memory_size, layer_size);
	predicted.setValues({ {{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{3, 3}, {0, 0}},
		{{4, 4}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> model_error(h_model_error, batch_size, memory_size);
	model_error.setConstant(0);
	Eigen::TensorMap<Eigen::Tensor<float, 3>> node_error(h_node_errors, batch_size, memory_size, layer_size);
	node_error.setConstant(0);
	Eigen::Tensor<float, 2> expected(batch_size, layer_size);
	expected.setConstant(1);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeModelErrors(
		expected,
		h_predicted,
		d_predicted,
		h_model_error,
		d_model_error,
		h_node_errors,
		d_node_errors,
		loss_function,
		loss_grad_function,
		batch_size,
		memory_size,
		layer_size,
		time_step,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 2> expected_model_error(batch_size, memory_size);
	expected_model_error.setValues({ {0, 0}, {0.5, 0}, {2.0, 0}, {4.5, 0} });
	Eigen::Tensor<float, 3> expected_node_error(batch_size, memory_size, layer_size);
	expected_node_error.setValues({
		{ {0, 0 }, { 0, 0 } },
		{ {-0.5, -0.5 }, { 0, 0 } },
		{ {-1, -1 }, { 0, 0 } },
		{ {-1.5, -1.5 }, { 0, 0 } } });

	for (int batch_iter = 0; batch_iter < batch_size; ++batch_iter) {
		for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
			//std::cout << "[Model Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << " = " << model_error(batch_iter, memory_iter) << std::endl;
			assert(model_error(batch_iter, memory_iter) == expected_model_error(batch_iter, memory_iter));
			for (int node_iter = 0; node_iter < layer_size; ++node_iter) {
				//std::cout << "[Node Error] Batch iter: " << batch_iter << ", Memory Iter: " << memory_iter << ", Node Iter: " << node_iter << " = " << node_error(batch_iter, memory_iter, node_iter) << std::endl;
				assert(node_error(batch_iter, memory_iter, node_iter) == expected_node_error(batch_iter, memory_iter, node_iter));
			}
		}
	}

	assert(cudaFreeHost(h_predicted) == cudaSuccess);
	assert(cudaFree(d_predicted) == cudaSuccess);
	assert(cudaFreeHost(h_node_errors) == cudaSuccess);
	assert(cudaFree(d_node_errors) == cudaSuccess);
	assert(cudaFreeHost(h_model_error) == cudaSuccess);
	assert(cudaFree(d_model_error) == cudaSuccess);
}

void test_modelMetricGpuDevice()
{
  const int device_id = 0;
  ModelKernalGpu<float> kernal;

  std::shared_ptr<MetricFunctionTensorOp<float, Eigen::GpuDevice>> metric_function = std::make_shared<MAETensorOp<float, Eigen::GpuDevice>>(MAETensorOp<float, Eigen::GpuDevice>());
  const int batch_size = 4;
  const int memory_size = 2;
  const int layer_size = 2;
  const int n_metrics = 1;
  const int time_step = 0;
  const int metric_index = 0;

  float* h_predicted;
  float* d_predicted;
  float* h_model_metric;
  float* d_model_metric;

  assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

  // allocate memory
  std::size_t bytes = batch_size * memory_size * layer_size * sizeof(float);
  std::size_t model_bytes = n_metrics * memory_size * sizeof(float);
  assert(cudaHostAlloc((void**)(&h_predicted), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_predicted), bytes) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_model_metric), model_bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_model_metric), model_bytes) == cudaSuccess);

  Eigen::TensorMap<Eigen::Tensor<float, 3>> predicted(h_predicted, batch_size, memory_size, layer_size);
  predicted.setValues({ {{1, 1}, {0, 0}},
    {{2, 2}, {0, 0}},
    {{3, 3}, {0, 0}},
    {{4, 4}, {0, 0}} });
  Eigen::TensorMap<Eigen::Tensor<float, 2>> model_metric(h_model_metric, n_metrics, memory_size);
  model_metric.setConstant(0);
  Eigen::Tensor<float, 2> expected(batch_size, layer_size);
  expected.setConstant(1);

  // Set up the device
  cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  bool success = kernal.executeModelMetric(
    expected,
    h_predicted,
    d_predicted,
    h_model_metric,
    d_model_metric,
    metric_function,
    batch_size,
    memory_size,
    layer_size,
    n_metrics,
    time_step,
    metric_index,
    device,
    true,
    true);

  // Synchronize the stream
  cudaError_t err = cudaStreamQuery(stream);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);

  Eigen::Tensor<float, 2> expected_model_metric(batch_size, memory_size);
  expected_model_metric.setValues({ {1.5, 0} });

  for (int metric_iter = 0; metric_iter < n_metrics; ++metric_iter) {
    for (int memory_iter = 0; memory_iter < memory_size; ++memory_iter) {
      //std::cout << "[Model Metric] Metric iter: " << metric_iter << ", Memory Iter: " << memory_iter << " = " << model_metric(metric_iter, memory_iter) << std::endl;
      assert(model_metric(metric_iter, memory_iter) == expected_model_metric(metric_iter, memory_iter));
    }
  }

  assert(cudaFreeHost(h_predicted) == cudaSuccess);
  assert(cudaFree(d_predicted) == cudaSuccess);
  assert(cudaFreeHost(h_model_metric) == cudaSuccess);
  assert(cudaFree(d_model_metric) == cudaSuccess);
}

void test_weightErrorGpuDevice()
{
	const int device_id = 0;
	ModelKernalGpu<float> kernal;

	std::shared_ptr<IntegrationWeightGradTensorOp<float, Eigen::GpuDevice>> integration_function = std::make_shared<SumWeightGradTensorOp<float, Eigen::GpuDevice>>(SumWeightGradTensorOp<float, Eigen::GpuDevice>());
	const int batch_size = 4;
	const int memory_size = 2;
	const int source_layer_size = 2;
	const int sink_layer_size = 1;

	float* h_sink_errors;
	float* d_sink_errors;
	float* h_source_outputs;
	float* d_source_outputs;
	float* h_source_inputs;
	float* d_source_inputs;
	float* h_weight;
	float* d_weight;
	float* h_weight_error;
	float* d_weight_error;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t source_bytes = batch_size * memory_size * source_layer_size * sizeof(float);
	std::size_t sink_bytes = batch_size * memory_size * sink_layer_size * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_sink_errors), sink_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_sink_errors), sink_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_source_outputs), source_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_source_outputs), source_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_source_inputs), source_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_source_inputs), source_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weight), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weight), weight_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weight_error), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weight_error), weight_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> sink_error(h_sink_errors, batch_size, memory_size, sink_layer_size);
	sink_error.setValues({ {{1}, {1}},
		{{2}, {1}},
		{{3}, {0}},
		{{4}, {0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_output(h_source_outputs, batch_size, memory_size, source_layer_size);
	source_output.setValues({ {{1, 1}, {1, 1}},
		{{2, 2}, {2, 2}},
		{{1, 1}, {0, 0}},
		{{2, 2}, {0, 0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 3>> source_input(h_source_inputs, batch_size, memory_size, source_layer_size);
	source_input.setValues({ {{2, 2}, {0, 0}},
		{{4, 4}, {0, 0}},
		{{2, 2}, {0, 0}},
		{{4, 4}, {0, 0}} });

	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weight, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight_error(h_weight_error, source_layer_size, sink_layer_size);
	weight_error.setConstant(0);

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeWeightErrors(
		h_sink_errors,
		d_sink_errors,
		h_source_outputs,
		d_source_outputs,
		h_source_inputs,
		d_source_inputs,
		source_layer_size,
		integration_function,
		h_weight,
		d_weight,
		h_weight_error,
		d_weight_error,
		batch_size,
		memory_size,
		source_layer_size,
		sink_layer_size,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 2> expected_weight_error(source_layer_size, sink_layer_size);
	expected_weight_error.setValues({ {-4.75}, {-4.75} });

	for (int source_iter = 0; source_iter < source_layer_size; ++source_iter) {
		for (int sink_iter = 0; sink_iter < sink_layer_size; ++sink_iter) {
			//std::cout << "[Weight Error] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << " = " << weight_error(source_iter, sink_iter) << std::endl;
			assert(weight_error(source_iter, sink_iter) == expected_weight_error(source_iter, sink_iter));
		}
	}

	assert(cudaFreeHost(h_sink_errors) == cudaSuccess);
	assert(cudaFree(d_sink_errors) == cudaSuccess);
	assert(cudaFreeHost(h_source_outputs) == cudaSuccess);
	assert(cudaFree(d_source_outputs) == cudaSuccess);
	assert(cudaFreeHost(h_source_inputs) == cudaSuccess);
	assert(cudaFree(d_source_inputs) == cudaSuccess);
	assert(cudaFreeHost(h_weight) == cudaSuccess);
	assert(cudaFree(d_weight) == cudaSuccess);
	assert(cudaFreeHost(h_weight_error) == cudaSuccess);
	assert(cudaFree(d_weight_error) == cudaSuccess);
}

void test_sharedWeightErrorGpuDevice()
{
	const int device_id = 0;
	ModelKernalGpu<float> kernal;

	const int source_layer_size = 2;
	const int sink_layer_size = 2;
	const int n_shared_weights = 1;

	float* h_shared_weights;
	float* d_shared_weights;
	float* h_weight_error;
	float* d_weight_error;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t shared_weights_bytes = source_layer_size * sink_layer_size * n_shared_weights * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_shared_weights), shared_weights_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_shared_weights), shared_weights_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weight_error), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weight_error), weight_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> shared_weights(h_shared_weights, source_layer_size, sink_layer_size, n_shared_weights);
	shared_weights.setValues({ 
		{{1}, {1}},
		{{0}, {0}}
		});
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight_error(h_weight_error, source_layer_size, sink_layer_size);
	weight_error.setValues({ {1, 2}, {3, 4} });

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeSharedWeightErrors(
		h_weight_error,
		d_weight_error,
		h_shared_weights,
		d_shared_weights,
		source_layer_size,
		sink_layer_size,
		n_shared_weights,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 2> expected_weight_error(source_layer_size, sink_layer_size);
	expected_weight_error.setValues({ {3, 3}, {3, 4} });

	for (int source_iter = 0; source_iter < source_layer_size; ++source_iter) {
		for (int sink_iter = 0; sink_iter < sink_layer_size; ++sink_iter) {
			//std::cout << "[Weight Error] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << " = " << weight_error(source_iter, sink_iter) << std::endl;
			assert(weight_error(source_iter, sink_iter) == expected_weight_error(source_iter, sink_iter));
		}
	}

	assert(cudaFreeHost(h_shared_weights) == cudaSuccess);
	assert(cudaFree(d_shared_weights) == cudaSuccess);
	assert(cudaFreeHost(h_weight_error) == cudaSuccess);
	assert(cudaFree(d_weight_error) == cudaSuccess);
}

void test_weightUpdateGpuDevice(){
	const int device_id = 0;
	ModelKernalGpu<float> kernal;

	std::shared_ptr<SolverTensorOp<float, Eigen::GpuDevice>> solver_function = std::make_shared<SGDTensorOp<float, Eigen::GpuDevice>>(SGDTensorOp<float, Eigen::GpuDevice>());
	const int source_layer_size = 2;
	const int sink_layer_size = 1;
  const int iter = 0;

	float* h_solver_params = new float[source_layer_size * sink_layer_size * 3];
	float* d_solver_params = new float[source_layer_size * sink_layer_size * 3];
	float* h_weight;
	float* d_weight;
	float* h_weight_error;
	float* d_weight_error;

	assert(cudaSetDevice(device_id) == cudaSuccess); // is this needed?

	// allocate memory
	std::size_t solver_bytes = source_layer_size * sink_layer_size * 3 * sizeof(float);
	std::size_t weight_bytes = source_layer_size * sink_layer_size * sizeof(float);
	assert(cudaHostAlloc((void**)(&h_solver_params), solver_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_solver_params), solver_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weight), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weight), weight_bytes) == cudaSuccess);
	assert(cudaHostAlloc((void**)(&h_weight_error), weight_bytes, cudaHostAllocDefault) == cudaSuccess);
	assert(cudaMalloc((void**)(&d_weight_error), weight_bytes) == cudaSuccess);

	Eigen::TensorMap<Eigen::Tensor<float, 3>> solver_params(h_solver_params, source_layer_size, sink_layer_size, 3);
	solver_params.setValues({ {{0.01, 0.99, 0.0}},
		{{0.01, 0.99, 0.0}} });
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight(h_weight, source_layer_size, sink_layer_size);
	weight.setConstant(1);
	Eigen::TensorMap<Eigen::Tensor<float, 2>> weight_error(h_weight_error, source_layer_size, sink_layer_size);
	weight_error.setValues({ {-0.2},	{-20} });

	// Set up the device
	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	bool success = kernal.executeWeightUpdate(
		h_weight,
		d_weight,
		h_solver_params,
		d_solver_params,
		h_weight_error,
		d_weight_error,
		solver_function,
		source_layer_size,
		sink_layer_size,
    iter,
		device,
		true,
		true);

	// Synchronize the stream
	cudaError_t err = cudaStreamQuery(stream);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 2> expected_weights(source_layer_size, sink_layer_size);
	expected_weights.setValues({ {1.002}, {1.2} });

	Eigen::Tensor<float, 3> expected_params(source_layer_size, sink_layer_size, 3);
	expected_params.setValues({ {{0.01, 0.99, -0.002}},
		{{0.01, 0.99, -0.2}} });

	for (int source_iter = 0; source_iter < source_layer_size; ++source_iter) {
		for (int sink_iter = 0; sink_iter < sink_layer_size; ++sink_iter) {
			//std::cout << "[Weight] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << " = " << weight(source_iter, sink_iter) << std::endl;
			assert(assert_close(weight(source_iter, sink_iter),expected_weights(source_iter, sink_iter)));
			for (int param_iter = 0; param_iter < 2; ++param_iter) { // [NOTE: should be `param_iter < 3`]
				//std::cout << "[Params] Source iter: " << source_iter << ", Sink Iter: " << sink_iter << ", Param Iter: " << param_iter << " = " << solver_params(source_iter, sink_iter, param_iter) << std::endl;
				assert(assert_close(solver_params(source_iter, sink_iter, param_iter),expected_params(source_iter, sink_iter, param_iter))); // Not sure why the last param does not pass...
			}
		}
	}

	assert(cudaFreeHost(h_solver_params) == cudaSuccess);
	assert(cudaFree(d_solver_params) == cudaSuccess);
	assert(cudaFreeHost(h_weight) == cudaSuccess);
	assert(cudaFree(d_weight) == cudaSuccess);
	assert(cudaFreeHost(h_weight_error) == cudaSuccess);
	assert(cudaFree(d_weight_error) == cudaSuccess);

}

int main(int argc, char** argv)
{
	test_constructorGpuDevice();
	test_destructorGpuDevice();
	test_nodeActivationGpuDevice();
	test_nodeDerivativeGpuDevice();
	test_forwardPropogationGpuDevice();
	test_backwardPropogationGpuDevice();
	test_modelErrorGpuDevice();
  test_modelMetricGpuDevice();
	test_weightErrorGpuDevice();
	test_sharedWeightErrorGpuDevice();
	test_weightUpdateGpuDevice();
	return 0;
}
#endif